#include "hip/hip_runtime.h"
#include "ntt.cuh"
#include "timer.h"

namespace troy {namespace utils {

    static const size_t NTT_KERNEL_THREAD_COUNT = 256;

    NTTTables::NTTTables(size_t coeff_count_power, const Modulus& modulus) {

        size_t coeff_count = static_cast<size_t>(1) << coeff_count_power;
        
        // We defer parameter checking to try_minimal_primitive_root(...)

        uint64_t root = 0;
        if (!utils::try_minimal_primitive_root(
            static_cast<uint64_t>(2 * coeff_count),
            modulus,
            root
        )) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to find primitive root.");
        }

        uint64_t inv_root = 0;
        if (!try_invert_uint64_mod(root, modulus, inv_root)) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to invert.");
        }

        // Populate tables with powers of root in specific orders.
        
        Array<MultiplyUint64Operand> root_powers(coeff_count, false);
        MultiplyUint64Operand root_operand(root, modulus);
        uint64_t power = root;
        for (size_t i = 1; i < coeff_count; i++) {
            root_powers[static_cast<size_t>(utils::reverse_bits_uint64(
                static_cast<uint64_t>(i),
                coeff_count_power
            ))] = MultiplyUint64Operand(power, modulus);
            power = utils::multiply_uint64operand_mod(power, root_operand, modulus);
        }
        root_powers[0] = MultiplyUint64Operand(1, modulus);

        Array<MultiplyUint64Operand> inv_root_powers(coeff_count, false);
        root_operand = MultiplyUint64Operand(inv_root, modulus);
        power = inv_root;
        for (size_t i = 1; i < coeff_count; i++) {
            inv_root_powers[static_cast<size_t>(utils::reverse_bits_uint64(
                static_cast<uint64_t>(i - 1),
                coeff_count_power
            )) + 1] = MultiplyUint64Operand(power, modulus);
            power = utils::multiply_uint64operand_mod(power, root_operand, modulus);
        }
        inv_root_powers[0] = MultiplyUint64Operand(1, modulus);

        uint64_t degree_uint64 = static_cast<uint64_t>(coeff_count);
        uint64_t inv_degree_modulo = 0;
        if (!utils::try_invert_uint64_mod(degree_uint64, modulus, inv_degree_modulo)) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to invert degree.");
        }
        MultiplyUint64Operand inv_degree_modulo_operand(inv_degree_modulo, modulus);

        this->root_ = root;
        this->coeff_count_ = coeff_count;
        this->coeff_count_power_ = coeff_count_power;
        this->modulus_ = modulus;
        this->inv_degree_modulo_ = inv_degree_modulo_operand;
        this->root_powers_ = std::move(root_powers);
        this->inv_root_powers_ = std::move(inv_root_powers);
        this->device = false;

    }

    void host_ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t degree = static_cast<size_t>(1) << log_degree; 
        for (size_t j = 0; j < tables.size(); j++) {
            const Modulus& modulus = tables[j].modulus();
            MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
            for (size_t k = 0; k < pcount; k++) {
                for (size_t i = 0; i < degree; i++) {
                    size_t x_index = ((k * tables.size() + j) << log_degree) + i;
                    operand[x_index] = multiply_uint64operand_mod_lazy(operand[x_index], scalar, modulus);
                }
            }
        }
    }

    __global__ void kernel_ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t degree = static_cast<size_t>(1) << log_degree;
        size_t total = pcount * tables.size() * degree;
        if (global_index < total) {
            // size_t k = global_index / (tables.size() * degree);
            size_t j = (global_index / degree) % tables.size();
            // size_t i = global_index % degree;
            const Modulus& modulus = tables[j].modulus();
            MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
            operand[global_index] = multiply_uint64operand_mod_lazy(operand[global_index], scalar, modulus);
        }
    }

    void ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        bool device = operand.on_device();
        // same device
        if (device != tables.on_device()) {
            throw std::invalid_argument("[ntt_multiply_inv_degree] Operand and tables must be on the same device.");
        }
        if (device) {
            size_t total = (pcount * tables.size()) << log_degree;
            size_t block_count = ceil_div<size_t>(total, KERNEL_THREAD_COUNT);
            kernel_ntt_multiply_inv_degree<<<block_count, KERNEL_THREAD_COUNT>>>(operand, pcount, log_degree, tables);
        } else {
            host_ntt_multiply_inv_degree(operand, pcount, log_degree, tables);
        }
    }

    void host_ntt_transfer_to_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = m + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t x = operand[x_index];
                    uint64_t y = operand[y_index];
                    uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
                    uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
                    x = u + v;
                    y = u + two_times_modulus - v;
                    operand[x_index] = x;
                    operand[y_index] = y;
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_to_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }
        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;
        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = m + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t x = operand[x_index];
        uint64_t y = operand[y_index];
        uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
        uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
        x = u + v;
        y = u + two_times_modulus - v;
        operand[x_index] = x;
        operand[y_index] = y;
    }

    void ntt_transfer_to_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        if (device) {
            size_t total = pcount * tables.size() * (1 << (log_degree - 1));
            size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
            kernel_ntt_transfer_to_rev_layer<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
        } else {
            host_ntt_transfer_to_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
        }
    }

    void ntt_transfer_to_rev(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        // same device
        if (device != tables.on_device()) {
            throw std::invalid_argument("[ntt_transfer_to_rev] Operand and tables must be on the same device.");
        }
        for (size_t layer = 0; layer < log_degree; layer++) {
            ntt_transfer_to_rev_layer(
                layer, operand,
                pcount, log_degree,
                tables, use_inv_root_powers 
            );
        }
    }

    void host_ntt_transfer_from_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t u = operand[x_index];
                    uint64_t v = operand[y_index];
                    operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
                    operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_from_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }
        
        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t u = operand[x_index];
        uint64_t v = operand[y_index];
        operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
        operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);
    }

    void ntt_transfer_from_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        if (device) {
            size_t total = (pcount * tables.size()) << (log_degree - 1);
            size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
            kernel_ntt_transfer_from_rev_layer<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
        } else {
            host_ntt_transfer_from_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
        }
    }

    void ntt_transfer_from_rev(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        // same device
        if (device != tables.on_device()) {
            throw std::invalid_argument("[ntt_transfer_from_rev] Operand and tables must be on the same device.");
        }
        size_t n = static_cast<size_t>(1) << log_degree;
        size_t m = n >> 1;
        size_t layer = 0;
        for (; m >= 1; m >>= 1) {
            ntt_transfer_from_rev_layer(
                layer, operand,
                pcount, log_degree,
                tables, use_inv_root_powers
            );
            layer ++;
        }
        ntt_multiply_inv_degree(
            operand, pcount, log_degree, tables
        );
    }

    void host_ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t degree = static_cast<size_t>(1) << log_degree; 
        for (size_t j = 0; j < tables.size(); j++) {
            uint64_t modulus = tables[j].modulus().value();
            uint64_t two_times_modulus = modulus << 1;
            for (size_t k = 0; k < pcount; k++) {
                for (size_t i = 0; i < degree; i++) {
                    size_t x_index = ((k * tables.size() + j) << log_degree) + i;
                    uint64_t x = operand[x_index];
                    if (x >= two_times_modulus) x -= two_times_modulus;
                    if (x >= modulus) x -= modulus;
                    operand[x_index] = x;
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t degree = static_cast<size_t>(1) << log_degree;
        size_t total = pcount * tables.size() * degree;
        if (global_index < total) {
            // size_t k = global_index / (tables.size() * degree);
            size_t j = (global_index / degree) % tables.size();
            // size_t i = global_index % degree;
            uint64_t x = operand[global_index];
            uint64_t modulus = tables[j].modulus().value();
            uint64_t two_times_modulus = modulus << 1;
            if (x >= two_times_modulus) x -= two_times_modulus;
            if (x >= modulus) x -= modulus;
            operand[global_index] = x;
        }
    }

    void ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        bool device = operand.on_device();
        // same device=
        if (device != tables.on_device()) {
            throw std::invalid_argument("[ntt_transfer_last_reduce] Operand and tables must be on the same device.");
        }
        if (device) {
            size_t total = (pcount * tables.size()) << log_degree;
            size_t block_count = ceil_div<size_t>(total, KERNEL_THREAD_COUNT);
            kernel_ntt_transfer_last_reduce<<<block_count, KERNEL_THREAD_COUNT>>>(operand, pcount, log_degree, tables);
        } else {
            host_ntt_transfer_last_reduce(operand, pcount, log_degree, tables);
        }
    }

}}