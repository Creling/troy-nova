#include "hip/hip_runtime.h"
#include "ntt.h"
#include "timer.h"
#include <cassert>

namespace troy {namespace utils {

    static const size_t NTT_KERNEL_THREAD_COUNT = 256;
    static const size_t NTT_KERNEL_THREAD_COUNT_LOG2 = 8;

    NTTTables::NTTTables(size_t coeff_count_power, const Modulus& modulus) {

        size_t coeff_count = static_cast<size_t>(1) << coeff_count_power;
        
        // We defer parameter checking to try_minimal_primitive_root(...)

        uint64_t root = 0;
        if (!utils::try_minimal_primitive_root(
            static_cast<uint64_t>(2 * coeff_count),
            modulus,
            root
        )) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to find primitive root.");
        }

        uint64_t inv_root = 0;
        if (!try_invert_uint64_mod(root, modulus, inv_root)) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to invert.");
        }

        // Populate tables with powers of root in specific orders.
        
        Array<MultiplyUint64Operand> root_powers(coeff_count, false, nullptr);
        MultiplyUint64Operand root_operand(root, modulus);
        uint64_t power = root;
        for (size_t i = 1; i < coeff_count; i++) {
            root_powers[static_cast<size_t>(utils::reverse_bits_uint64(
                static_cast<uint64_t>(i),
                coeff_count_power
            ))] = MultiplyUint64Operand(power, modulus);
            power = utils::multiply_uint64operand_mod(power, root_operand, modulus);
        }
        root_powers[0] = MultiplyUint64Operand(1, modulus);

        Array<MultiplyUint64Operand> inv_root_powers(coeff_count, false, nullptr);
        root_operand = MultiplyUint64Operand(inv_root, modulus);
        power = inv_root;
        for (size_t i = 1; i < coeff_count; i++) {
            inv_root_powers[static_cast<size_t>(utils::reverse_bits_uint64(
                static_cast<uint64_t>(i - 1),
                coeff_count_power
            )) + 1] = MultiplyUint64Operand(power, modulus);
            power = utils::multiply_uint64operand_mod(power, root_operand, modulus);
        }
        inv_root_powers[0] = MultiplyUint64Operand(1, modulus);

        uint64_t degree_uint64 = static_cast<uint64_t>(coeff_count);
        uint64_t inv_degree_modulo = 0;
        if (!utils::try_invert_uint64_mod(degree_uint64, modulus, inv_degree_modulo)) {
            throw std::invalid_argument("[NTTTables::NTTTables] Invalid modulus, unable to invert degree.");
        }
        MultiplyUint64Operand inv_degree_modulo_operand(inv_degree_modulo, modulus);

        this->root_ = root;
        this->coeff_count_ = coeff_count;
        this->coeff_count_power_ = coeff_count_power;
        this->modulus_ = modulus;
        this->inv_degree_modulo_ = inv_degree_modulo_operand;
        this->root_powers_ = std::move(root_powers);
        this->inv_root_powers_ = std::move(inv_root_powers);
        this->device = false;

    }

    void host_ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t degree = static_cast<size_t>(1) << log_degree; 
        for (size_t j = 0; j < tables.size(); j++) {
            const Modulus& modulus = tables[j].modulus();
            MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
            for (size_t k = 0; k < pcount; k++) {
                for (size_t i = 0; i < degree; i++) {
                    size_t x_index = ((k * tables.size() + j) << log_degree) + i;
                    operand[x_index] = multiply_uint64operand_mod_lazy(operand[x_index], scalar, modulus);
                }
            }
        }
    }

    __global__ void kernel_ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t degree = static_cast<size_t>(1) << log_degree;
        size_t total = pcount * tables.size() * degree;
        if (global_index < total) {
            // size_t k = global_index / (tables.size() * degree);
            size_t j = (global_index / degree) % tables.size();
            // size_t i = global_index % degree;
            const Modulus& modulus = tables[j].modulus();
            MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
            operand[global_index] = multiply_uint64operand_mod_lazy(operand[global_index], scalar, modulus);
        }
    }

    void ntt_multiply_inv_degree(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        bool device = operand.on_device();
        // same device
        if (!device_compatible(operand, tables)) {
            throw std::invalid_argument("[ntt_multiply_inv_degree] Operand and tables must be on the same device.");
        }
        if (device) {
            size_t total = (pcount * tables.size()) << log_degree;
            size_t block_count = ceil_div<size_t>(total, KERNEL_THREAD_COUNT);
            utils::set_device(operand.device_index());
            kernel_ntt_multiply_inv_degree<<<block_count, KERNEL_THREAD_COUNT>>>(operand, pcount, log_degree, tables);
        } else {
            host_ntt_multiply_inv_degree(operand, pcount, log_degree, tables);
        }
    }

    void host_ntt_transfer_to_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = m + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t x = operand[x_index];
                    uint64_t y = operand[y_index];
                    uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
                    uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
                    x = u + v;
                    y = u + two_times_modulus - v;
                    operand[x_index] = x;
                    operand[y_index] = y;
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_to_rev_layer1(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = m + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t x = operand[x_index];
        uint64_t y = operand[y_index];
        uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
        uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
        x = u + v;
        y = u + two_times_modulus - v;

        operand[x_index] = x;
        operand[y_index] = y;
    }

    /* // This old version does not use shared memory
    __global__ void kernel_ntt_transfer_to_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;

        size_t block_idx = static_cast<size_t>(blockIdx.x) % (gridDim.x / (pcount * coeff_modulus_size));
        size_t gap_power = log_degree - layer_lower - 1;
        size_t gap = 1 << gap_power;
        size_t E = min(static_cast<size_t>(blockDim.x), gap); // elements in gap
        size_t C = blockDim.x / E; // gaps crossed
        size_t stride = gap / E;

        size_t component_global_offset = (k * coeff_modulus_size + j) << log_degree;
        size_t coefficient_offset = block_idx % stride + (block_idx / stride) * C * 2 * gap;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;

        for (size_t dl = 0; dl < layer_upper - layer_lower; dl++) {

            size_t layer = layer_lower + dl;

            size_t x_index = threadIdx.x / E * 2 * gap + threadIdx.x % E * stride + coefficient_offset;
            
            size_t m = 1 << layer;

            size_t i = ((x_index >> (gap_power + 1)) << gap_power) + (x_index & (gap - 1));
            size_t rid = m + (i / gap);

            x_index += component_global_offset;
            size_t y_index = x_index + gap;
            
            MultiplyUint64Operand r = use_inv_root_powers ?
                tables[j].inv_root_powers()[rid] :
                tables[j].root_powers()[rid];
            uint64_t x = operand[x_index];
            uint64_t y = operand[y_index];
            uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
            uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
            x = u + v;
            y = u + two_times_modulus - v;

            operand[x_index] = x;
            operand[y_index] = y;

            __syncthreads();

            E >>= 1;
            gap >>= 1;
            gap_power -= 1;

        }
    }
    */

    __global__ void kernel_ntt_transfer_to_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        unsigned int global_index = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int coeff_modulus_size = tables.size();

        unsigned int k = global_index / (coeff_modulus_size << (log_degree - 1));
        unsigned int j = (global_index >> (log_degree - 1)) % coeff_modulus_size;
        
        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        unsigned int block_idx = blockIdx.x % (gridDim.x / (pcount * coeff_modulus_size));
        unsigned int gap_power = log_degree - layer_lower - 1;
        unsigned int E_power = min(static_cast<unsigned int>(get_power_of_two(blockDim.x)), gap_power); // elements in gap
        unsigned int E_mask = (1 << E_power) - 1;
        unsigned int stride_power = gap_power - E_power;
        unsigned int stride_mask = (1 << stride_power) - 1;

        unsigned int coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << (gap_power + 1));
        unsigned int global_offset = (k * coeff_modulus_size + j) << log_degree;

        __shared__ uint64_t sdata[NTT_KERNEL_THREAD_COUNT * 2];
        unsigned int from_x_index = 
            ((threadIdx.x >> E_power) << (gap_power + 1))
            + ((threadIdx.x & E_mask) << stride_power) 
            + coefficient_offset 
            + global_offset;
        unsigned int from_y_index = from_x_index + (1 << gap_power);
        unsigned int to_x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask);
        unsigned int to_y_index = to_x_index + (1 << E_power);
        sdata[to_x_index] = operand[from_x_index];
        sdata[to_y_index] = operand[from_y_index];
        __syncthreads();

        coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << gap_power);

        for (unsigned int layer = layer_lower; layer < layer_upper; layer++) {

            unsigned int rid = (1 << layer) + (threadIdx.x >> E_power) + ((((threadIdx.x & E_mask) << stride_power) + coefficient_offset) >> gap_power);
            const MultiplyUint64Operand& r = r_ptr[rid];

            unsigned int x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask); // wrt shared data
            unsigned int y_index = x_index + E_mask + 1;
            
            uint64_t& x = sdata[x_index];
            uint64_t& y = sdata[y_index];
            uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
            uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
            x = u + v;
            y = u + two_times_modulus - v;

            __syncthreads();

            E_power -= 1;
            E_mask = (1 << E_power) - 1;
            gap_power -= 1;
        }
        
        operand[from_x_index] = sdata[to_x_index];
        operand[from_y_index] = sdata[to_y_index];
    }

    void ntt_transfer_to_rev(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        // same device
        if (!device_compatible(operand, tables)) {
            throw std::invalid_argument("[ntt_transfer_to_rev] Operand and tables must be on the same device.");
        }
        if (!device) {
            for (size_t layer = 0; layer < log_degree; layer++) {
                host_ntt_transfer_to_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
            }
        } else {
            if (log_degree <= NTT_KERNEL_THREAD_COUNT_LOG2) {
                size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                size_t thread_count = 1 << (log_degree - 1);
                size_t block_count = ceil_div<size_t>(total, thread_count);
                assert(block_count == total / thread_count);
                kernel_ntt_transfer_to_rev_layers<<<block_count, thread_count>>>(
                    0, log_degree, operand, pcount, log_degree, tables, use_inv_root_powers
                );
            } else {
                for (size_t layer_lower = 0; layer_lower < log_degree; layer_lower += NTT_KERNEL_THREAD_COUNT_LOG2) {
                    size_t layer_upper = std::min(layer_lower + NTT_KERNEL_THREAD_COUNT_LOG2, log_degree);
                    size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                    size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
                    assert(block_count == total / NTT_KERNEL_THREAD_COUNT);
                    kernel_ntt_transfer_to_rev_layers<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(
                        layer_lower, layer_upper, operand, pcount, log_degree, tables, use_inv_root_powers
                    );
                }
            }
        }
    }

    void host_ntt_transfer_from_rev_layer(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t u = operand[x_index];
                    uint64_t v = operand[y_index];
                    operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
                    operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_from_rev_layer1(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }
        
        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t u = operand[x_index];
        uint64_t v = operand[y_index];
        operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
        operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

    }

    /* // This old version does not use shared memory
    __global__ void kernel_ntt_transfer_from_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;

        size_t block_idx = static_cast<size_t>(blockIdx.x) % (gridDim.x / (pcount * coeff_modulus_size));
        size_t gap_power = layer_upper - 1;
        size_t gap = 1 << gap_power;
        size_t E = min(static_cast<size_t>(blockDim.x), gap); // elements in gap
        size_t C = blockDim.x / E; // gaps crossed
        size_t stride = gap / E;

        size_t component_global_offset = (k * coeff_modulus_size + j) << log_degree;
        size_t coefficient_offset = block_idx % stride + (block_idx / stride) * C * 2 * gap;

        gap >>= (layer_upper - layer_lower - 1);
        gap_power -= (layer_upper - layer_lower - 1);
        E >>= (layer_upper - layer_lower - 1);

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        for (size_t layer = layer_lower; layer < layer_upper; layer++) {

            size_t x_index = threadIdx.x / E * 2 * gap + threadIdx.x % E * stride + coefficient_offset;
            
            size_t m = 1 << (log_degree - layer - 1);

            size_t i = ((x_index >> (gap_power + 1)) << gap_power) + (x_index & (gap - 1));
            size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);

            x_index += component_global_offset;
            size_t y_index = x_index + gap;

            const MultiplyUint64Operand& r = r_ptr[rid];
            
            uint64_t u = operand[x_index];
            uint64_t v = operand[y_index];
            operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
            operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

            __syncthreads();

            E <<= 1;
            gap <<= 1;
            gap_power += 1;

        }
    }
    */

    __global__ void kernel_ntt_transfer_from_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        unsigned int global_index = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int i_upperbound = 1 << (log_degree - 1);
        unsigned int coeff_modulus_size = tables.size();

        unsigned int k = global_index / (coeff_modulus_size * i_upperbound);
        unsigned int j = (global_index / i_upperbound) % coeff_modulus_size;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        unsigned int block_idx = blockIdx.x % (gridDim.x / (pcount * coeff_modulus_size));
        unsigned int gap_power = layer_upper - 1;
        unsigned int E_power = min(static_cast<unsigned int>(get_power_of_two(blockDim.x)), gap_power); // elements in gap
        unsigned int E_mask = (1 << E_power) - 1;
        unsigned int stride_power = gap_power - E_power;
        unsigned int stride_mask = (1 << stride_power) - 1;

        unsigned int global_offset = (k * coeff_modulus_size + j) << log_degree;
        unsigned int coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << (gap_power + 1));

        __shared__ uint64_t sdata[NTT_KERNEL_THREAD_COUNT * 2];
        unsigned int from_x_index = 
            ((threadIdx.x >> E_power) << (gap_power + 1))
            + ((threadIdx.x & E_mask) << stride_power) 
            + coefficient_offset 
            + global_offset;
        unsigned int from_y_index = from_x_index + (1 << gap_power);
        unsigned int to_x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask);
        unsigned int to_y_index = to_x_index + (1 << E_power);
        sdata[to_x_index] = operand[from_x_index];
        sdata[to_y_index] = operand[from_y_index];
        __syncthreads();
        
        coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << gap_power);

        gap_power -= (layer_upper - layer_lower - 1);
        E_power -= (layer_upper - layer_lower - 1);
        E_mask = (1 << E_power) - 1;

        for (unsigned int layer = layer_lower; layer < layer_upper; layer++) {

            unsigned int rid = (1 << log_degree) - (1 << (log_degree - layer)) + 1
                + (threadIdx.x >> E_power) + ((((threadIdx.x & E_mask) << stride_power) + coefficient_offset) >> gap_power);

            unsigned int x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask); // wrt shared data
            unsigned int y_index = x_index + E_mask + 1;

            const MultiplyUint64Operand& r = r_ptr[rid];
            
            uint64_t u = sdata[x_index];
            uint64_t v = sdata[y_index];
            sdata[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
            sdata[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

            __syncthreads();

            E_power += 1;
            E_mask = (1 << E_power) - 1;
            gap_power += 1;

        }
        
        operand[from_x_index] = sdata[to_x_index];
        operand[from_y_index] = sdata[to_y_index];
    }

    void ntt_transfer_from_rev(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        bool device = operand.on_device();
        // same device
        if (!device_compatible(operand, tables)) {
            throw std::invalid_argument("[ntt_transfer_from_rev] Operand and tables must be on the same device.");
        }
        if (!device) {
            for (size_t layer = 0; layer < log_degree; layer++) {
                host_ntt_transfer_from_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers);
            }
        } else {
            if (log_degree <= NTT_KERNEL_THREAD_COUNT_LOG2) {
                size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                size_t thread_count = 1 << (log_degree - 1);
                size_t block_count = ceil_div<size_t>(total, thread_count);
                assert(block_count == total / thread_count);
                kernel_ntt_transfer_from_rev_layers<<<block_count, thread_count>>>(
                    0, log_degree, operand, pcount, log_degree, tables, use_inv_root_powers
                );
            } else {
                for (size_t layer_lower = 0; layer_lower < log_degree; layer_lower += NTT_KERNEL_THREAD_COUNT_LOG2) {
                    size_t layer_upper = std::min(layer_lower + NTT_KERNEL_THREAD_COUNT_LOG2, log_degree);
                    size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                    size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
                    assert(block_count == total / NTT_KERNEL_THREAD_COUNT);
                    kernel_ntt_transfer_from_rev_layers<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(
                        layer_lower, layer_upper, operand, pcount, log_degree, tables, use_inv_root_powers
                    );
                }
            }
        }
        ntt_multiply_inv_degree(
            operand, pcount, log_degree, tables
        );
    }

    void host_ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t degree = static_cast<size_t>(1) << log_degree; 
        for (size_t j = 0; j < tables.size(); j++) {
            uint64_t modulus = tables[j].modulus().value();
            uint64_t two_times_modulus = modulus << 1;
            for (size_t k = 0; k < pcount; k++) {
                for (size_t i = 0; i < degree; i++) {
                    size_t x_index = ((k * tables.size() + j) << log_degree) + i;
                    uint64_t x = operand[x_index];
                    if (x >= two_times_modulus) x -= two_times_modulus;
                    if (x >= modulus) x -= modulus;
                    operand[x_index] = x;
                }
            }
        }
    }

    __global__ void kernel_ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t degree = static_cast<size_t>(1) << log_degree;
        size_t total = pcount * tables.size() * degree;
        if (global_index < total) {
            // size_t k = global_index / (tables.size() * degree);
            size_t j = (global_index / degree) % tables.size();
            // size_t i = global_index % degree;
            uint64_t x = operand[global_index];
            uint64_t modulus = tables[j].modulus().value();
            uint64_t two_times_modulus = modulus << 1;
            if (x >= two_times_modulus) x -= two_times_modulus;
            if (x >= modulus) x -= modulus;
            operand[global_index] = x;
        }
    }

    void ntt_transfer_last_reduce(Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables) {
        bool device = operand.on_device();
        // same device=
        if (!device_compatible(operand, tables)) {
            throw std::invalid_argument("[ntt_transfer_last_reduce] Operand and tables must be on the same device.");
        }
        if (device) {
            size_t total = (pcount * tables.size()) << log_degree;
            size_t block_count = ceil_div<size_t>(total, KERNEL_THREAD_COUNT);
            kernel_ntt_transfer_last_reduce<<<block_count, KERNEL_THREAD_COUNT>>>(operand, pcount, log_degree, tables);
        } else {
            host_ntt_transfer_last_reduce(operand, pcount, log_degree, tables);
        }
    }

}}