#include "hip/hip_runtime.h"
#include "ntt_grouped.h"
#include <cassert>

namespace troy::utils::fgk::ntt_grouped {

    static constexpr size_t NTT_KERNEL_THREAD_COUNT = 256;
    static constexpr size_t NTT_KERNEL_THREAD_COUNT_LOG2 = 8;

    void host_ntt_transfer_to_rev_layer(size_t layer, ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = m + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t x = operand[x_index];
                    uint64_t y = operand[y_index];
                    uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
                    uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
                    x = u + v;
                    y = u + two_times_modulus - v;
                    result[x_index] = x;
                    result[y_index] = y;
                }
            }
        }
        if (layer == log_degree - 1) {
            size_t n = 1 << log_degree;
            for (size_t j = 0; j < coeff_modulus_size; j++) {
                const Modulus& modulus = tables[j].modulus();
                uint64_t mv = modulus.value();
                uint64_t tmv = modulus.value() << 1;
                for (size_t i = 0; i < n; i++) {
                    for (size_t k = 0; k < pcount; k++) {
                        size_t index = ((k * coeff_modulus_size + j) << log_degree) + i;
                        if (result[index] >= tmv) result[index] -= tmv;
                        if (result[index] >= mv) result[index] -= mv;
                    }
                }
            }
        }
    }

    /*
    __global__ void kernel_ntt_transfer_to_rev_layer1(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        size_t m = 1 << layer;
        size_t gap_power = log_degree - layer - 1;
        size_t gap = 1 << gap_power;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = m + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t x = operand[x_index];
        uint64_t y = operand[y_index];
        uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
        uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
        x = u + v;
        y = u + two_times_modulus - v;

        operand[x_index] = x;
        operand[y_index] = y;
    }
    */

    /* // This old version does not use shared memory
    __global__ void kernel_ntt_transfer_to_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;

        size_t block_idx = static_cast<size_t>(blockIdx.x) % (gridDim.x / (pcount * coeff_modulus_size));
        size_t gap_power = log_degree - layer_lower - 1;
        size_t gap = 1 << gap_power;
        size_t E = min(static_cast<size_t>(blockDim.x), gap); // elements in gap
        size_t C = blockDim.x / E; // gaps crossed
        size_t stride = gap / E;

        size_t component_global_offset = (k * coeff_modulus_size + j) << log_degree;
        size_t coefficient_offset = block_idx % stride + (block_idx / stride) * C * 2 * gap;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;

        for (size_t dl = 0; dl < layer_upper - layer_lower; dl++) {

            size_t layer = layer_lower + dl;

            size_t x_index = threadIdx.x / E * 2 * gap + threadIdx.x % E * stride + coefficient_offset;
            
            size_t m = 1 << layer;

            size_t i = ((x_index >> (gap_power + 1)) << gap_power) + (x_index & (gap - 1));
            size_t rid = m + (i / gap);

            x_index += component_global_offset;
            size_t y_index = x_index + gap;
            
            MultiplyUint64Operand r = use_inv_root_powers ?
                tables[j].inv_root_powers()[rid] :
                tables[j].root_powers()[rid];
            uint64_t x = operand[x_index];
            uint64_t y = operand[y_index];
            uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
            uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
            x = u + v;
            y = u + two_times_modulus - v;

            operand[x_index] = x;
            operand[y_index] = y;

            __syncthreads();

            E >>= 1;
            gap >>= 1;
            gap_power -= 1;

        }
    }
    */

    __global__ void kernel_ntt_transfer_to_rev_layers(size_t layer_lower, size_t layer_upper, ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        unsigned int global_index = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int coeff_modulus_size = tables.size();

        unsigned int k = global_index / (coeff_modulus_size << (log_degree - 1));
        unsigned int j = (global_index >> (log_degree - 1)) % coeff_modulus_size;
        
        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        unsigned int block_idx = blockIdx.x % (gridDim.x / (pcount * coeff_modulus_size));
        unsigned int gap_power = log_degree - layer_lower - 1;
        unsigned int E_power = min(static_cast<unsigned int>(get_power_of_two(blockDim.x)), gap_power); // elements in gap
        unsigned int E_mask = (1 << E_power) - 1;
        unsigned int stride_power = gap_power - E_power;
        unsigned int stride_mask = (1 << stride_power) - 1;

        unsigned int coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << (gap_power + 1));
        unsigned int global_offset = (k * coeff_modulus_size + j) << log_degree;

        __shared__ uint64_t sdata[NTT_KERNEL_THREAD_COUNT * 2];
        unsigned int from_x_index = 
            ((threadIdx.x >> E_power) << (gap_power + 1))
            + ((threadIdx.x & E_mask) << stride_power) 
            + coefficient_offset 
            + global_offset;
        unsigned int from_y_index = from_x_index + (1 << gap_power);
        unsigned int to_x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask);
        unsigned int to_y_index = to_x_index + (1 << E_power);
        sdata[to_x_index] = operand[from_x_index];
        sdata[to_y_index] = operand[from_y_index];
        __syncthreads();

        coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << gap_power);

        for (unsigned int layer = layer_lower; layer < layer_upper; layer++) {

            unsigned int rid = (1 << layer) + (threadIdx.x >> E_power) + ((((threadIdx.x & E_mask) << stride_power) + coefficient_offset) >> gap_power);
            const MultiplyUint64Operand& r = r_ptr[rid];

            unsigned int x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask); // wrt shared data
            unsigned int y_index = x_index + E_mask + 1;
            
            uint64_t& x = sdata[x_index];
            uint64_t& y = sdata[y_index];
            uint64_t u = (x >= two_times_modulus) ? (x - two_times_modulus) : x;
            uint64_t v = utils::multiply_uint64operand_mod_lazy(y, r, modulus);
            x = u + v;
            y = u + two_times_modulus - v;

            __syncthreads();

            E_power -= 1;
            E_mask = (1 << E_power) - 1;
            gap_power -= 1;
        }
        
        uint64_t mv = modulus.value();
        if (sdata[to_x_index] >= two_times_modulus) sdata[to_x_index] -= two_times_modulus;
        if (sdata[to_x_index] >= mv) sdata[to_x_index] -= mv;
        if (sdata[to_y_index] >= two_times_modulus) sdata[to_y_index] -= two_times_modulus;
        if (sdata[to_y_index] >= mv) sdata[to_y_index] -= mv;

        result[from_x_index] = sdata[to_x_index];
        result[from_y_index] = sdata[to_y_index];
    }

    void ntt(ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        bool device = operand.on_device();
        // same device
        if (!device_compatible(operand, tables, result)) {
            throw std::invalid_argument("[ntt_transfer_to_rev] Operand and tables must be on the same device.");
        }
        if (!device) {
            for (size_t layer = 0; layer < log_degree; layer++) {
                host_ntt_transfer_to_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers, result);
                operand = result.as_const();
            }
        } else {
            if (log_degree <= NTT_KERNEL_THREAD_COUNT_LOG2) {
                size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                size_t thread_count = 1 << (log_degree - 1);
                size_t block_count = ceil_div<size_t>(total, thread_count);
                assert(block_count == total / thread_count);
                utils::set_device(operand.device_index());
                kernel_ntt_transfer_to_rev_layers<<<block_count, thread_count>>>(
                    0, log_degree, operand, pcount, log_degree, tables, use_inv_root_powers, result
                );
                utils::stream_sync();
            } else {
                for (size_t layer_lower = 0; layer_lower < log_degree; layer_lower += NTT_KERNEL_THREAD_COUNT_LOG2) {
                    size_t layer_upper = std::min(layer_lower + NTT_KERNEL_THREAD_COUNT_LOG2, log_degree);
                    size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                    size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
                    assert(block_count == total / NTT_KERNEL_THREAD_COUNT);
                    utils::set_device(operand.device_index());
                    kernel_ntt_transfer_to_rev_layers<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(
                        layer_lower, layer_upper, operand, pcount, log_degree, tables, use_inv_root_powers, result
                    );
                    utils::stream_sync();
                    operand = result.as_const();
                }
            }
        }
    }

    void host_ntt_transfer_from_rev_layer(size_t layer, ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        for (size_t j = 0; j < coeff_modulus_size; j++) {
            const Modulus& modulus = tables[j].modulus();
            uint64_t two_times_modulus = modulus.value() << 1;
            for (size_t i = 0; i < i_upperbound; i++) {
                size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
                size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
                MultiplyUint64Operand r = use_inv_root_powers ?
                    tables[j].inv_root_powers()[rid] :
                    tables[j].root_powers()[rid];
                for (size_t k = 0; k < pcount; k++) {
                    size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
                    size_t y_index = x_index + gap;
                    uint64_t u = operand[x_index];
                    uint64_t v = operand[y_index];
                    result[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
                    result[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);
                }
            }
        }
        if (layer == log_degree - 1) {
            size_t n = 1 << log_degree;
            for (size_t j = 0; j < coeff_modulus_size; j++) {
                const Modulus& modulus = tables[j].modulus();
                MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
                uint64_t mv = modulus.value();
                uint64_t tmv = modulus.value() << 1;
                for (size_t i = 0; i < n; i++) {
                    for (size_t k = 0; k < pcount; k++) {
                        size_t index = ((k * coeff_modulus_size + j) << log_degree) + i;
                        if (result[index] >= tmv) result[index] -= tmv;
                        if (result[index] >= mv) result[index] -= mv;
                        result[index] = multiply_uint64operand_mod_lazy(result[index], scalar, modulus);
                    }
                }
            }
        }
        
    }

    /*
    __global__ void kernel_ntt_transfer_from_rev_layer1(size_t layer, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }
        
        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;
        size_t i = global_index % i_upperbound;

        size_t m = 1 << (log_degree - layer - 1);
        size_t gap_power = layer;
        size_t gap = 1 << gap_power;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);
        size_t coeff_index = ((i >> gap_power) << (gap_power + 1)) + (i & (gap - 1));
        MultiplyUint64Operand r = use_inv_root_powers ?
            tables[j].inv_root_powers()[rid] :
            tables[j].root_powers()[rid];
        
        size_t x_index = ((k * coeff_modulus_size + j) << log_degree) + coeff_index;
        size_t y_index = x_index + gap;
        uint64_t u = operand[x_index];
        uint64_t v = operand[y_index];
        operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
        operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

    }
    */

    /* // This old version does not use shared memory
    __global__ void kernel_ntt_transfer_from_rev_layers(size_t layer_lower, size_t layer_upper, Slice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        size_t i_upperbound = 1 << (log_degree - 1);
        size_t coeff_modulus_size = tables.size();
        if (global_index >= (pcount * coeff_modulus_size * i_upperbound)) {
            return;
        }

        size_t k = global_index / (coeff_modulus_size * i_upperbound);
        size_t j = (global_index / i_upperbound) % coeff_modulus_size;

        size_t block_idx = static_cast<size_t>(blockIdx.x) % (gridDim.x / (pcount * coeff_modulus_size));
        size_t gap_power = layer_upper - 1;
        size_t gap = 1 << gap_power;
        size_t E = min(static_cast<size_t>(blockDim.x), gap); // elements in gap
        size_t C = blockDim.x / E; // gaps crossed
        size_t stride = gap / E;

        size_t component_global_offset = (k * coeff_modulus_size + j) << log_degree;
        size_t coefficient_offset = block_idx % stride + (block_idx / stride) * C * 2 * gap;

        gap >>= (layer_upper - layer_lower - 1);
        gap_power -= (layer_upper - layer_lower - 1);
        E >>= (layer_upper - layer_lower - 1);

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        for (size_t layer = layer_lower; layer < layer_upper; layer++) {

            size_t x_index = threadIdx.x / E * 2 * gap + threadIdx.x % E * stride + coefficient_offset;
            
            size_t m = 1 << (log_degree - layer - 1);

            size_t i = ((x_index >> (gap_power + 1)) << gap_power) + (x_index & (gap - 1));
            size_t rid = (1 << log_degree) - (m << 1) + 1 + (i >> gap_power);

            x_index += component_global_offset;
            size_t y_index = x_index + gap;

            const MultiplyUint64Operand& r = r_ptr[rid];
            
            uint64_t u = operand[x_index];
            uint64_t v = operand[y_index];
            operand[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
            operand[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

            __syncthreads();

            E <<= 1;
            gap <<= 1;
            gap_power += 1;

        }
    }
    */

    __global__ void kernel_ntt_transfer_from_rev_layers(size_t layer_lower, size_t layer_upper, ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        unsigned int global_index = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int i_upperbound = 1 << (log_degree - 1);
        unsigned int coeff_modulus_size = tables.size();

        unsigned int k = global_index / (coeff_modulus_size * i_upperbound);
        unsigned int j = (global_index / i_upperbound) % coeff_modulus_size;

        const Modulus& modulus = tables[j].modulus();
        uint64_t two_times_modulus = modulus.value() << 1;
        const MultiplyUint64Operand* r_ptr = use_inv_root_powers ?
            tables[j].inv_root_powers().raw_pointer() :
            tables[j].root_powers().raw_pointer();

        unsigned int block_idx = blockIdx.x % (gridDim.x / (pcount * coeff_modulus_size));
        unsigned int gap_power = layer_upper - 1;
        unsigned int E_power = min(static_cast<unsigned int>(get_power_of_two(blockDim.x)), gap_power); // elements in gap
        unsigned int E_mask = (1 << E_power) - 1;
        unsigned int stride_power = gap_power - E_power;
        unsigned int stride_mask = (1 << stride_power) - 1;

        unsigned int global_offset = (k * coeff_modulus_size + j) << log_degree;
        unsigned int coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << (gap_power + 1));

        __shared__ uint64_t sdata[NTT_KERNEL_THREAD_COUNT * 2];
        unsigned int from_x_index = 
            ((threadIdx.x >> E_power) << (gap_power + 1))
            + ((threadIdx.x & E_mask) << stride_power) 
            + coefficient_offset 
            + global_offset;
        unsigned int from_y_index = from_x_index + (1 << gap_power);
        unsigned int to_x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask);
        unsigned int to_y_index = to_x_index + (1 << E_power);
        sdata[to_x_index] = operand[from_x_index];
        sdata[to_y_index] = operand[from_y_index];
        __syncthreads();
        
        coefficient_offset = (block_idx & stride_mask) + (((block_idx >> stride_power) * (blockDim.x >> E_power)) << gap_power);

        gap_power -= (layer_upper - layer_lower - 1);
        E_power -= (layer_upper - layer_lower - 1);
        E_mask = (1 << E_power) - 1;

        for (unsigned int layer = layer_lower; layer < layer_upper; layer++) {

            unsigned int rid = (1 << log_degree) - (1 << (log_degree - layer)) + 1
                + (threadIdx.x >> E_power) + ((((threadIdx.x & E_mask) << stride_power) + coefficient_offset) >> gap_power);

            unsigned int x_index = ((threadIdx.x & (~E_mask)) << 1) + (threadIdx.x & E_mask); // wrt shared data
            unsigned int y_index = x_index + E_mask + 1;

            const MultiplyUint64Operand& r = r_ptr[rid];
            
            uint64_t u = sdata[x_index];
            uint64_t v = sdata[y_index];
            sdata[x_index] = (u + v > two_times_modulus) ? (u + v - two_times_modulus) : (u + v);
            sdata[y_index] = utils::multiply_uint64operand_mod_lazy(u + two_times_modulus - v, r, modulus);

            __syncthreads();

            E_power += 1;
            E_mask = (1 << E_power) - 1;
            gap_power += 1;

        }
        
        uint64_t mv = modulus.value();
        if (sdata[to_x_index] >= two_times_modulus) sdata[to_x_index] -= two_times_modulus;
        if (sdata[to_x_index] >= mv) sdata[to_x_index] -= mv;
        if (sdata[to_y_index] >= two_times_modulus) sdata[to_y_index] -= two_times_modulus;
        if (sdata[to_y_index] >= mv) sdata[to_y_index] -= mv;

        if (layer_upper == log_degree) {
            const Modulus& modulus = tables[j].modulus();
            MultiplyUint64Operand scalar = tables[j].inv_degree_modulo();
            sdata[to_x_index] = multiply_uint64operand_mod_lazy(sdata[to_x_index], scalar, modulus);
            sdata[to_y_index] = multiply_uint64operand_mod_lazy(sdata[to_y_index], scalar, modulus);
        }

        result[from_x_index] = sdata[to_x_index];
        result[from_y_index] = sdata[to_y_index];
    }

    void intt(ConstSlice<uint64_t> operand, size_t pcount, size_t log_degree, ConstSlice<NTTTables> tables, bool use_inv_root_powers, Slice<uint64_t> result) {
        bool device = operand.on_device();
        // same device
        if (!device_compatible(operand, tables, result)) {
            throw std::invalid_argument("[ntt_transfer_from_rev] Operand and tables must be on the same device.");
        }
        if (!device) {
            for (size_t layer = 0; layer < log_degree; layer++) {
                host_ntt_transfer_from_rev_layer(layer, operand, pcount, log_degree, tables, use_inv_root_powers, result);
                operand = result.as_const();
            }
        } else {
            if (log_degree <= NTT_KERNEL_THREAD_COUNT_LOG2) {
                size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                size_t thread_count = 1 << (log_degree - 1);
                size_t block_count = ceil_div<size_t>(total, thread_count);
                assert(block_count == total / thread_count);
                utils::set_device(operand.device_index());
                kernel_ntt_transfer_from_rev_layers<<<block_count, thread_count>>>(
                    0, log_degree, operand, pcount, log_degree, tables, use_inv_root_powers, result
                );
                utils::stream_sync();
            } else {
                for (size_t layer_lower = 0; layer_lower < log_degree; layer_lower += NTT_KERNEL_THREAD_COUNT_LOG2) {
                    size_t layer_upper = std::min(layer_lower + NTT_KERNEL_THREAD_COUNT_LOG2, log_degree);
                    size_t total = pcount * tables.size() * (1 << (log_degree - 1));
                    size_t block_count = ceil_div<size_t>(total, NTT_KERNEL_THREAD_COUNT);
                    assert(block_count == total / NTT_KERNEL_THREAD_COUNT);
                    utils::set_device(operand.device_index());
                    kernel_ntt_transfer_from_rev_layers<<<block_count, NTT_KERNEL_THREAD_COUNT>>>(
                        layer_lower, layer_upper, operand, pcount, log_degree, tables, use_inv_root_powers, result
                    );
                    utils::stream_sync();
                    operand = result.as_const();
                }
            }
        }
    }

}