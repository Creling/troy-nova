#include "hip/hip_runtime.h"
#include "bfv_ring2k.h"

#include <sstream>
#include <stdexcept>
#include <cstdint>
#include <unordered_map>

namespace troy::linear {

    using utils::Array;
    using utils::MultiplyUint64Operand;
    using utils::ConstSlice;
    using utils::Slice;
    using uint128_t = __uint128_t;
    using utils::KERNEL_THREAD_COUNT;
    using utils::ceil_div;

    static void custom_assert(bool condition, const char* message = "") {
        if (!condition) {
            throw std::invalid_argument(message);
        }
    }

    template <typename T>
    static constexpr typename std::enable_if<is_compatible_ring2k<T>::value, size_t>::type
    type_bits() {
        return sizeof(T) * 8;
    }

    template <typename T>
    static constexpr typename std::enable_if<is_compatible_ring2k<T>::value, T>::type
    inverse_ring2k(T x) {
        if ((x & 1) == 0) throw std::invalid_argument("[bfv_ring2k::inverse_ring2k] x must be odd");
        T inv = 1; T p = x;
        for (size_t i = 1; i < type_bits<T>(); i++) {
            inv *= p; p *= p;
        }
        return inv;
    }

    template <typename T>
    __host__ __device__ static constexpr typename std::enable_if<is_compatible_ring2k<T>::value, uint64_t>::type
    general_reduce(T x, const Modulus& modulus) {
        if constexpr (std::is_same<T, uint32_t>::value) {
            return modulus.reduce(static_cast<uint64_t>(x));
        } else if constexpr (std::is_same<T, uint64_t>::value) {
            return modulus.reduce(x);
        } else if constexpr (std::is_same<T, uint128_t>::value) {
            return modulus.reduce_uint128(x);
        }
    }

    static inline uint128_t assemble_from_limbs(ConstSlice<uint64_t> limbs) {
        if (limbs.size() == 0) return 0;
        if (limbs.size() == 1) return static_cast<uint128_t>(limbs[0]);
        return (
            (static_cast<uint128_t>(limbs[1]) << 64) | 
            (static_cast<uint128_t>(limbs[0]))
        );
    }

    __host__ __device__ static inline uint128_t uint128_from_uint64s(uint64_t low, uint64_t high) {
        return (static_cast<uint128_t>(high) << 64) | static_cast<uint128_t>(low);
    }

    __host__ __device__ static inline void set_uint64s_with_uint128(uint64_t* target, uint128_t source) {
        target[0] = static_cast<uint64_t>(source);
        target[1] = static_cast<uint64_t>(source >> 64);
    }

    template <typename T>
    static constexpr typename std::enable_if<is_compatible_ring2k<T>::value, T>::type
    modulo_mask(size_t bit_length) {
        if (bit_length > type_bits<T>()) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::modulo_mask] bit_length must be less than or equal to type_bits<T>()");
        }
        if (bit_length == type_bits<T>()) {
            return static_cast<T>(-1);
        } else {
            return (static_cast<T>(1) << bit_length) - 1;
        }
    }

    template <typename T>
    static typename std::enable_if<is_compatible_ring2k<T>::value, T>::type
    modulo_from_limbs(ConstSlice<uint64_t> limbs, size_t mod_bit_length) {
        return static_cast<T>(assemble_from_limbs(limbs) & modulo_mask<T>(mod_bit_length));
    }

    template <typename T>
    PolynomialEncoderRNSHelper<T>::PolynomialEncoderRNSHelper(ContextDataPointer context_data, size_t t_bit_length) {
        if (t_bit_length <= type_bits<T>() / 2) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] t_bit_length must be greater than type_bits<T>() / 2");
        }
        EncryptionParameters parms = context_data->parms();
        if (parms.scheme() != SchemeType::BFV) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] scheme must be BFV");
        }
        if (context_data->parms().on_device()) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] context_data must be on host. Please turn context to device after the encoder is created.");
        }
        
        this->parms_id_ = context_data->parms_id();
        this->t_bit_length_ = t_bit_length;
        size_t log_Q = context_data->total_coeff_modulus_bit_count();

        size_t poly_degree = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t num_modulus = coeff_modulus.size();
        gamma_host_ = utils::get_prime(poly_degree, utils::HE_INTERNAL_MOD_BIT_COUNT);
        Modulus* gamma_space = reinterpret_cast<Modulus*>(std::malloc(sizeof(Modulus)));
        *gamma_space = gamma_host_;
        gamma_ = utils::Box<Modulus>(gamma_space, false, nullptr);
        for (size_t i = 0; i < coeff_modulus.size(); i++) {
            if (coeff_modulus[i].value() == gamma_host_.value()) {
                throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] gamma is in coeff_modulus");
            }
        }
        mod_t_mask_ = modulo_mask<T>(t_bit_length);
        t_half_ = static_cast<T>(1) << (t_bit_length - 1);

        ConstSlice<uint64_t> Q = context_data->total_coeff_modulus();
        Array<uint64_t> Q_div_t(num_modulus, false, nullptr); Q_div_t.set_zero();
        if (log_Q > t_bit_length) {
            utils::right_shift_uint(Q, t_bit_length, num_modulus, Q_div_t.reference());
        } else {
            Q_div_t.set_zero();
        }

        Q_mod_t_ = modulo_from_limbs<T>(Q, t_bit_length);

        const utils::RNSTool& rns_tool = context_data->rns_tool();
        rns_tool.base_q().decompose_single(Q_div_t.reference());
        Q_div_t_mod_qi_ = Array<MultiplyUint64Operand>(num_modulus, false, nullptr);
        for (size_t i = 0; i < num_modulus; i++) {
            Q_div_t_mod_qi_[i] = MultiplyUint64Operand(Q_div_t[i], coeff_modulus[i]);
        }

        const utils::RNSBase& base_Q = rns_tool.base_q();
        utils::RNSBase base_gamma = utils::RNSBase(ConstSlice(&gamma_host_, 1, false, nullptr));
        base_Q_to_gamma_ = std::move(utils::BaseConverter(base_Q, base_gamma));

        punctured_q_mod_t_ = Array<T>(num_modulus, false, nullptr);
        for (size_t i = 0; i < num_modulus; i++) {
            punctured_q_mod_t_[i] = modulo_from_limbs<T>(base_Q.punctured_product().const_slice(i * num_modulus, (i + 1) * num_modulus), t_bit_length);
        }

        if (t_bit_length <= 64) {
            neg_inv_Q_mod_t_ = (-inverse_ring2k<T>(static_cast<T>(base_Q.base_product()[0]))) & mod_t_mask_;
            inv_gamma_mod_t_ = inverse_ring2k<T>(static_cast<T>(gamma_host_.value())) & mod_t_mask_;
        } else {
            T base_Q_128 = static_cast<T>(assemble_from_limbs(base_Q.base_product()));
            T base_gamma_128 = static_cast<T>(gamma_host_.value());
            neg_inv_Q_mod_t_ = (-inverse_ring2k<T>(base_Q_128)) & mod_t_mask_;
            inv_gamma_mod_t_ = inverse_ring2k<T>(base_gamma_128) & mod_t_mask_;
            if (((-(base_Q_128 * neg_inv_Q_mod_t_)) & mod_t_mask_) != 1) {
                throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] -(base_Q * neg_inv_Q_mod_t) != 1");
            }
            if (((base_gamma_128 * inv_gamma_mod_t_) & mod_t_mask_) != 1) {
                throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] base_gamma * inv_gamma_mod_t != 1");
            }
        };

        {
            uint64_t Q_mod_gamma = utils::modulo_uint(base_Q.base_product(), gamma_host_);
            uint64_t inv = 0;
            bool success = utils::try_invert_uint64_mod(Q_mod_gamma, gamma_host_, inv);
            if (!success) {
                throw std::invalid_argument("[PolynomialEncoderRNSHelper::PolynomialEncoderRNSHelper] failed to invert Q_mod_gamma");
            }
            neg_inv_Q_mod_gamma_ = utils::Box(
                reinterpret_cast<MultiplyUint64Operand*>(std::malloc(sizeof(MultiplyUint64Operand))),
                false, nullptr
            );
            *neg_inv_Q_mod_gamma_ = MultiplyUint64Operand(utils::negate_uint64_mod(inv, gamma_host_), gamma_host_);
        }

        gamma_t_mod_Q_ = Array<MultiplyUint64Operand>(num_modulus, false, nullptr);
        uint64_t t0[2]; set_uint64s_with_uint128(t0, static_cast<uint128_t>(1) << (t_bit_length / 2));
        uint64_t t1[2]; set_uint64s_with_uint128(t1, static_cast<uint128_t>(1) << (t_bit_length - t_bit_length / 2));

        for (size_t i = 0; i < num_modulus; i++) {
            const Modulus& prime = coeff_modulus[i];
            uint64_t t = prime.reduce_uint128_limbs(ConstSlice<uint64_t>(t0, 2, false, nullptr)); 
            t = utils::multiply_uint64_mod(t, prime.reduce_uint128_limbs(ConstSlice<uint64_t>(t1, 2, false, nullptr)), prime);
            uint64_t g = prime.reduce(gamma_host_.value());
            gamma_t_mod_Q_[i] = MultiplyUint64Operand(utils::multiply_uint64_mod(g, t, prime), prime);
        }
    }

    template <typename T>
    void PolynomialEncoderRNSHelper<T>::to_device_inplace(MemoryPoolHandle pool) {
        gamma_.to_device_inplace(pool);
        punctured_q_mod_t_.to_device_inplace(pool);
        gamma_t_mod_Q_.to_device_inplace(pool);
        base_Q_to_gamma_.to_device_inplace(pool);
        Q_div_t_mod_qi_.to_device_inplace(pool);
        neg_inv_Q_mod_gamma_.to_device_inplace(pool);
    }

    template <typename T>
    __global__ static void kernel_scale_up_component(
        ConstSlice<T> source,
        ConstSlice<Modulus> modulus, ConstSlice<MultiplyUint64Operand> Q_div_t_mod_qi,
        uint128_t Q_mod_t, uint128_t t_half, uint32_t base_mod_bitlen,
        size_t mod_idx, 
        Slice<uint64_t> out
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x; // global_index
        if (i < source.size()) {
        // u = (Q mod t)*x mod qi
        T x = source[i];
        uint64_t x64 = modulus[mod_idx].reduce(x);
        uint64_t u = utils::multiply_uint64operand_mod(x64, Q_div_t_mod_qi[mod_idx], modulus[mod_idx]);
        // uint128_t can conver uint32_t/uint64_t mult here
        T v = ((Q_mod_t * x + t_half) >> base_mod_bitlen);
            out[i] = modulus[mod_idx].reduce(u + v);
        }
    }

    /// This function obtains one RNS component of the scaled-up polynomial.
    template <typename T>
    void PolynomialEncoderRNSHelper<T>::scale_up_component(utils::ConstSlice<T> source, const HeContext& context, size_t modulus_index, utils::Slice<uint64_t> destination) const {
        // This implementation is only for uint32 and uint64.
        // Uint128 is implemented with a specialized version.
        static_assert(std::is_same<T, uint32_t>::value || std::is_same<T, uint64_t>::value, "[PolynomialEncoderRNSHelper::scale_up_component] T must be uint32_t or uint64_t");
        ContextDataPointer context_data = context.get_context_data(this->parms_id_).value();
        custom_assert(source.on_device() == destination.on_device(), "[PolynomialEncoderRNSHelper::scale_up_component] source and destination are not in the same device");
        const EncryptionParameters& parms = context_data->parms();
        custom_assert(modulus_index < parms.coeff_modulus().size(), "[PolynomialEncoderRNSHelper::scale_up_component] modulus_index is out of range");
        custom_assert(destination.size() >= source.size());
        if (!source.on_device()) {
            const Modulus& modulus = parms.coeff_modulus()[modulus_index];
            for (size_t i = 0; i < source.size(); i++) {
                uint64_t x64 = modulus.reduce(source[i]);
                uint64_t u = utils::multiply_uint64operand_mod(x64, Q_div_t_mod_qi_[modulus_index], modulus);
                // uint128_t can conver uint32_t/uint64_t mult here
                uint64_t v = static_cast<uint64_t>(((static_cast<uint128_t>(Q_mod_t_) * static_cast<uint128_t>(source[i])) + static_cast<uint128_t>(t_half_)) >> t_bit_length_);
                destination[i] = modulus.reduce(u + v);
            }
        } else {
            size_t block_count = ceil_div(source.size(), KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_scale_up_component<T><<<block_count, KERNEL_THREAD_COUNT>>>(
                source, context_data->parms().coeff_modulus(), 
                Q_div_t_mod_qi_.const_reference(), Q_mod_t_, 
                t_half_, t_bit_length_, modulus_index, destination
            );
        }
    }

    __global__ static void kernel_scale_up_component_uint128(
        ConstSlice<uint128_t> source,
        ConstSlice<Modulus> modulus, ConstSlice<MultiplyUint64Operand> Q_div_t_mod_qi,
        uint128_t Q_mod_t, uint128_t t_half, uint32_t base_mod_bitlen,
        size_t mod_idx, 
        Slice<uint64_t> out
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x; // global_index
        if (i < source.size()) {
            uint128_t x = source[i];
            uint64_t x64 = modulus[mod_idx].reduce_uint128(x);
            uint64_t u = troy::utils::multiply_uint64operand_mod(x64, Q_div_t_mod_qi[mod_idx], modulus[mod_idx]);
            
            // ensure 8-byte alignment
            uint64_t Q_mod_t_arr[2]; set_uint64s_with_uint128(Q_mod_t_arr, Q_mod_t);
            uint64_t t_half_arr[2]; set_uint64s_with_uint128(t_half_arr, t_half);
            uint64_t x_arr[2]; set_uint64s_with_uint128(x_arr, x);

            ConstSlice<uint64_t> Q_mod_t(Q_mod_t_arr, 2, true, nullptr);
            ConstSlice<uint64_t> xlimbs(x_arr, 2, true, nullptr);
            ConstSlice<uint64_t> t_half(t_half_arr, 2, true, nullptr);

            // Compute round(x * Q_mod_t / t) for 2^64 < x, t <= 2^128
            // round(x * Q_mod_t / t) = floor((x * Q_mod_t + t_half) / t)
            // We need 4 limbs to store the product x * Q_mod_t
            uint64_t mul_limbs[4]; Slice<uint64_t> mul_limbs_slice(mul_limbs, 4, true, nullptr);
            uint64_t add_limbs[4]; Slice<uint64_t> add_limbs_slice(add_limbs, 4, true, nullptr);
            uint64_t rs_limbs[3]; Slice<uint64_t> rs_limbs_slice(rs_limbs, 3, true, nullptr);
            utils::multiply_uint(Q_mod_t, xlimbs, mul_limbs_slice);
            utils::add_uint_carry(mul_limbs_slice.as_const(), t_half,
                    0, add_limbs_slice);
            // NOTE(juhou) base_mod_bitlen_ > 64, we can direct drop the LSB here.
            utils::right_shift_uint192(add_limbs_slice.const_slice(1, 4), base_mod_bitlen - 64,
                                rs_limbs_slice);
            out[i] = modulus[mod_idx].reduce_uint128(u + uint128_from_uint64s(rs_limbs[0], rs_limbs[1]));
        }
    }

    template <>
    void PolynomialEncoderRNSHelper<uint128_t>::scale_up_component(utils::ConstSlice<uint128_t> source, const HeContext& context, size_t modulus_index, utils::Slice<uint64_t> destination) const {
        ContextDataPointer context_data = context.get_context_data(this->parms_id_).value();
        custom_assert(source.on_device() == destination.on_device(), "[PolynomialEncoderRNSHelper::scale_up_component] source and destination are not in the same device");
        const EncryptionParameters& parms = context_data->parms();
        custom_assert(modulus_index < parms.coeff_modulus().size(), "[PolynomialEncoderRNSHelper::scale_up_component] modulus_index is out of range");
        custom_assert(destination.size() >= source.size());
        if (!source.on_device()) {
            const Modulus& modulus = parms.coeff_modulus()[modulus_index];
            ConstSlice<uint64_t> t_half = ConstSlice<uint64_t>(reinterpret_cast<const uint64_t*>(&this->t_half_), 2, false, nullptr);
            ConstSlice<uint64_t> q_mod_t = ConstSlice<uint64_t>(reinterpret_cast<const uint64_t*>(&this->Q_mod_t_), 2, false, nullptr);
            for (size_t i = 0; i < source.size(); i++) {
                uint128_t x = source[i];
                uint64_t x64 = modulus.reduce_uint128(x);
                uint64_t u = utils::multiply_uint64operand_mod(x64, Q_div_t_mod_qi_[modulus_index], modulus);
                
                uint64_t mul_limbs[4] = {0, 0, 0, 0}; Slice<uint64_t> mul_limbs_slice = Slice<uint64_t>(mul_limbs, 4, false, nullptr);
                uint64_t add_limbs[4] = {0, 0, 0, 0}; Slice<uint64_t> add_limbs_slice = Slice<uint64_t>(add_limbs, 4, false, nullptr);
                uint64_t rs_limbs[3] = {0, 0, 0}; Slice<uint64_t> rs_limbs_slice = Slice<uint64_t>(rs_limbs, 3, false, nullptr);
                uint64_t x_limbs[2] = {static_cast<uint64_t>(x), static_cast<uint64_t>(x >> 64)}; ConstSlice<uint64_t> x_limbs_slice = ConstSlice<uint64_t>(x_limbs, 2, false, nullptr);
                utils::multiply_uint(q_mod_t, x_limbs_slice, mul_limbs_slice);
                utils::add_uint_carry(mul_limbs_slice.as_const(), t_half, 0, add_limbs_slice);
                utils::right_shift_uint192(add_limbs_slice.const_slice(1, 4), t_bit_length_ - 64, rs_limbs_slice);
                destination[i] = modulus.reduce_uint128(static_cast<uint128_t>(u) + assemble_from_limbs(rs_limbs_slice.as_const()));
            }
        } else {
            size_t block_count = ceil_div(source.size(), KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_scale_up_component_uint128<<<block_count, KERNEL_THREAD_COUNT>>>(
                source, context_data->parms().coeff_modulus(), 
                Q_div_t_mod_qi_.const_reference(), Q_mod_t_, 
                t_half_, t_bit_length_, modulus_index, destination
            );
        }
    }

    template <typename T>
    __global__ static void kernel_centralize_at_component(
        ConstSlice<T> source,
        utils::ConstPointer<troy::Modulus> mod_qj,
        uint128_t t_half, uint128_t mod_t_mask,
        Slice<uint64_t> out
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x; // global_index
        if (i < source.size()) {
            T x = source[i];
            auto x128 = static_cast<uint128_t>(x);
            if (x128 > t_half) {
                uint64_t u = general_reduce(-x128 & mod_t_mask, *mod_qj);
                out[i] = troy::utils::negate_uint64_mod(u, *mod_qj);
            } else {
                out[i] = general_reduce(x, *mod_qj);
            }
        }
    }

    template <typename T>
    void PolynomialEncoderRNSHelper<T>::centralize_at_component(utils::ConstSlice<T> source, const HeContext& context, size_t modulus_index, utils::Slice<uint64_t> destination) const {
        custom_assert(destination.size() >= source.size());
        custom_assert(source.on_device() == destination.on_device(), "[PolynomialEncoderRNSHelper::centralize_at_component] source and destination are not in the same device");
        ContextDataPointer context_data = context.get_context_data(this->parms_id_).value();
        const EncryptionParameters& parms = context_data->parms();
        custom_assert(modulus_index < parms.coeff_modulus().size(), "modulus_index is out of range");
        if (!source.on_device()) { 
            const Modulus& modulus = parms.coeff_modulus()[modulus_index];
            for (size_t i = 0; i < source.size(); i++) {
                T x = source[i];
                if (x > t_half_) {
                    uint64_t u = general_reduce((-x) & mod_t_mask_, modulus);
                    destination[i] = utils::negate_uint64_mod(u, modulus);
                } else {
                    destination[i] = general_reduce(x, modulus);
                }
            }
        } else {
            size_t block_count = ceil_div(source.size(), KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_centralize_at_component<T><<<block_count, KERNEL_THREAD_COUNT>>>(
                source, parms.coeff_modulus().at(modulus_index), t_half_, mod_t_mask_, destination
            );
        }
    }

    template <typename T>
    __global__ static void kernel_scale_down(
        size_t num_modulus, size_t coeff_count,
        ConstSlice<uint64_t> tmp,
        ConstSlice<T> punctured_base_mod_t,
        uint128_t neg_inv_Q_mod_t,
        uint128_t inv_gamma_mod_t,
        uint128_t mod_t_mask,
        ConstSlice<uint64_t> base_on_gamma,
        utils::ConstPointer<troy::Modulus> gamma,
        Slice<T> out
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x; // global_index
        if (i >= coeff_count) return;

        T base_on_t = 0;
        // sum_i (x * (Q/qi)^{-1} mod qi) * (Q/qi) mod t
        for (size_t l = 0; l < num_modulus; ++l) {
            const T factor = punctured_base_mod_t[l];
            base_on_t += tmp[l * coeff_count + i] * factor;
        }

        // 3-2 Then multiply with -Q^{-1} mod t
        base_on_t = (base_on_t * neg_inv_Q_mod_t) & mod_t_mask;
        
        // clang-format off
        // 4 Correct sign: (base_on_t - [base_on_gamma]_gamma) * gamma^{-1} mod t
        // NOTE(juhou):
        // `base_on_gamma` and `base_on_t` together gives
        // `gamma*(x + t*r) + round(gamma*v/q) - e` mod gamma*t for some unknown v and e.
        // (Key point): Taking `base_on_gamma` along equals to
        //    `round(gamma*v/q) - e mod gamma`
        // When gamma > v, e, we can have the centered remainder
        // [round(gamma*v/q) - e mod gamma]_gamma = round(gamma*v/q) - e.
        // As a result, `base_on_t - [base_on_gamma]_gamma mod t` will cancel out the
        // last term and gives `gamma*(x + t*r) mod t`.
        // Finally, multiply with `gamma^{-1} mod t` gives `x mod t`.
        // clang-format on
        uint64_t gamma_div_2 = gamma->value() >> 1;
        uint64_t on_gamma = base_on_gamma[i];
        // [0, gamma) -> [-gamma/2, gamma/2]
        if (on_gamma > gamma_div_2) {
            out[i] = ((base_on_t + gamma->value() - on_gamma) * inv_gamma_mod_t) & mod_t_mask;
        } else {
            out[i] = ((base_on_t - on_gamma) * inv_gamma_mod_t) & mod_t_mask;
        }
    }


    template <typename T>
    void PolynomialEncoderRNSHelper<T>::scale_down(const Plaintext& input, const HeContext& context, utils::Slice<T> destination, MemoryPoolHandle pool) const {
        // Ref: Bajard et al. "A Full RNS Variant of FV like Somewhat Homomorphic
        // Encryption Schemes" (Section 3.2 & 3.3)
        // NOTE(juhou): Basically the same code in seal/util/rns.cpp instead we
        // use the plain modulus `t` as 2^k here.
        ParmsID parms_id = input.parms_id();
        if (parms_id == parms_id_zero) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::scale_down] input is not valid");
        }
        if (input.on_device() != this->on_device() || input.on_device() != destination.on_device()) {
            throw std::invalid_argument("[PolynomialEncoderRNSHelper::scale_down] self, input, destination are not in the same device");
        }
        ContextDataPointer context_data = context.get_context_data(parms_id).value();
        const EncryptionParameters& parms = context_data->parms();
        custom_assert(parms.poly_modulus_degree() == destination.size());
        size_t num_modulus = parms.coeff_modulus().size();
        size_t coeff_count = destination.size();
        custom_assert(input.coeff_modulus_size() == num_modulus);
        custom_assert(input.poly_modulus_degree() == coeff_count);
        custom_assert(input.data().size() == num_modulus * coeff_count);
        const utils::RNSBase &base_Q = context_data->rns_tool().base_q();
        ConstSlice<troy::Modulus> coeff_modulus = parms.coeff_modulus();
        custom_assert(input.on_device() == destination.on_device(), "[PolynomialEncoderRNSHelper::scale_down] input and destination are not in the same device");
        bool device = input.on_device();

        Array<uint64_t> tmp(input.data().size(), device, pool);
        tmp.set_zero();

        // 1. multiply with gamma*t
        troy::utils::multiply_uint64operand_p(
            input.const_reference(), gamma_t_mod_Q_.const_reference(), 
            coeff_count, coeff_modulus, tmp.reference()
        );

        // 2-1 FastBase convert from baseQ to {gamma}
        Array<uint64_t> base_on_gamma(coeff_count, device, pool);
        base_Q_to_gamma_.fast_convert_array(tmp.const_reference(), base_on_gamma.reference(), pool);
        // 2-2 Then multiply with -Q^{-1} mod gamma
        troy::utils::multiply_uint64operand_inplace(
            base_on_gamma.reference(), neg_inv_Q_mod_gamma_.as_const_pointer(),
            gamma_.as_const_pointer()
        );

        // 3-1 FastBase convert from baseQ to {t}
        // NOTE: overwrite the `tmp` (tmp is gamma*t*x mod Q)
        ConstSlice<troy::utils::MultiplyUint64Operand> inv_punctured = base_Q.inv_punctured_product_mod_base();
        utils::multiply_uint64operand_inplace_p(
            tmp.reference(), inv_punctured, coeff_count, coeff_modulus
        );

        if (!device) {
            for (size_t i = 0; i < coeff_count; i++) {
                T base_on_t = 0;
                // sum_i (x * (Q/qi)^{-1} mod qi) * (Q/qi) mod t
                for (size_t l = 0; l < num_modulus; ++l) {
                    const T factor = punctured_q_mod_t_[l];
                    base_on_t += tmp[l * coeff_count + i] * factor;
                }

                // 3-2 Then multiply with -Q^{-1} mod t
                base_on_t = (base_on_t * neg_inv_Q_mod_t_) & mod_t_mask_;

                // clang-format off
                // 4 Correct sign: (base_on_t - [base_on_gamma]_gamma) * gamma^{-1} mod t
                // NOTE(juhou):
                // `base_on_gamma` and `base_on_t` together gives
                // `gamma*(x + t*r) + round(gamma*v/q) - e` mod gamma*t for some unknown v and e.
                // (Key point): Taking `base_on_gamma` along equals to
                //    `round(gamma*v/q) - e mod gamma`
                // When gamma > v, e, we can have the centered remainder
                // [round(gamma*v/q) - e mod gamma]_gamma = round(gamma*v/q) - e.
                // As a result, `base_on_t - [base_on_gamma]_gamma mod t` will cancel out the
                // last term and gives `gamma*(x + t*r) mod t`.
                // Finally, multiply with `gamma^{-1} mod t` gives `x mod t`.
                // clang-format on
                uint64_t gamma_div_2 = gamma_->value() >> 1;
                uint64_t on_gamma = base_on_gamma[i];
                // [0, gamma) -> [-gamma/2, gamma/2]
                if (on_gamma > gamma_div_2) {
                    destination[i] = ((base_on_t + gamma_->value() - on_gamma) * inv_gamma_mod_t_) & mod_t_mask_;
                } else {
                    destination[i] = ((base_on_t - on_gamma) * inv_gamma_mod_t_) & mod_t_mask_;
                }
            }
        } else {
            size_t block_count = ceil_div(coeff_count, KERNEL_THREAD_COUNT);
            utils::set_device(destination.device_index());
            kernel_scale_down<T><<<block_count, KERNEL_THREAD_COUNT>>>(
                num_modulus, coeff_count, tmp.const_reference(),
                punctured_q_mod_t_.const_reference(),
                neg_inv_Q_mod_t_, inv_gamma_mod_t_, mod_t_mask_,
                base_on_gamma.const_reference(), gamma_.as_const_pointer(),
                destination
            );
        }
    }

    template <typename T>
    PolynomialEncoderRing2k<T>::PolynomialEncoderRing2k(HeContextPointer context, size_t t_bit_length) {
        context_ = context;
        t_bit_length_ = t_bit_length;
        std::optional<ContextDataPointer> context_data = context->key_context_data();
        std::unordered_map<ParmsID, std::shared_ptr<PolynomialEncoderRNSHelper<T>>, std::TroyHashParmsID> helpers;
        while (context_data.has_value()) {
            ContextDataPointer c = context_data.value();
            ParmsID parms_id = c->parms_id();
            helpers[parms_id] = std::make_shared<PolynomialEncoderRNSHelper<T>>(c, t_bit_length);
            context_data = c->next_context_data();
        }
        helpers_ = helpers;
    }
    
    // Instantiate the template class implementations
    
    template class PolynomialEncoderRNSHelper<uint32_t>;
    template class PolynomialEncoderRNSHelper<uint64_t>;
    template class PolynomialEncoderRNSHelper<uint128_t>;

    template class PolynomialEncoderRing2k<uint32_t>;
    template class PolynomialEncoderRing2k<uint64_t>;
    template class PolynomialEncoderRing2k<uint128_t>;
    

}