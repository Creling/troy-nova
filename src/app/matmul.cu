#include "hip/hip_runtime.h"
#include "matmul.h"

namespace troy { namespace linear {

    using uint128_t = __uint128_t;

    void MatmulHelper::determine_block() {
        size_t b_best = 0, i_best = 0, o_best = 0;
        size_t c_best = 2147483647;
        if (!pack_lwe) {
            for (size_t b = batch_size; b >= 1; b--) {
                size_t bc = ceil_div(batch_size, b);
                if (b >= slot_count) continue;
                if (bc * 2 > c_best) continue;
                for (size_t i = 1; i < slot_count / b; i++) {
                    size_t o = slot_count / b / i;
                    if (o > output_dims) o = output_dims;
                    if (i > input_dims) continue;
                    if (o < 1) continue;
                    size_t c = 0;
                    if (objective == MatmulObjective::EncryptLeft) {
                        c = bc * (ceil_div(input_dims, i) + ceil_div(output_dims, o));
                    } else if (objective == MatmulObjective::EncryptRight) {
                        c = (bc + ceil_div(input_dims, i)) * ceil_div(output_dims, o);
                    } else if (objective == MatmulObjective::Crossed) {
                        c = bc * input_dims + (bc + ceil_div(input_dims, i)) * ceil_div(output_dims, o);
                    } else {
                        throw std::runtime_error("[MatmulHelper::determine_block] Invalid objective");
                    }
                    if (c >= c_best) continue;
                    b_best = b; i_best = i; o_best = o; c_best = c;
                }
            }
        } else {
            double sqrtn = std::pow(slot_count, 0.33);
            size_t i = 1; while (i * 2 < sqrtn) {i *= 2;}
            if (i > input_dims) {
                i = 1; while (i < input_dims) i *= 2;
            }
            
            for (size_t b = 1; b <= batch_size; b++) {
                size_t bc = ceil_div(batch_size, b);
                if (b > slot_count) {continue;}
                size_t o = slot_count / b / i;
                if (o > output_dims) {o = output_dims;}
                if (o < 1) {continue;}
                size_t c = 0;
                if (objective == MatmulObjective::EncryptLeft) {
                    c = bc * ceil_div(input_dims, i);
                    c += ceil_div(bc * ceil_div(output_dims, o), i);
                } else if (objective == MatmulObjective::EncryptRight) {
                    c = ceil_div(output_dims, o) * ceil_div(input_dims, i);
                    c += ceil_div(bc * ceil_div(output_dims, o), i);
                } else if (objective == MatmulObjective::Crossed) {
                    c = bc * ceil_div(input_dims, i);
                    c += ceil_div(output_dims, o) * ceil_div(input_dims, i);
                    c += ceil_div(bc * ceil_div(output_dims, o), i);
                } else {
                    throw std::runtime_error("MatmulHelper: invalid objective");
                }
                if (c >= c_best) {continue;}
                b_best = b; i_best = i; o_best = o; c_best = c;
            }

        }
        batch_block = b_best;
        input_block = i_best;
        output_block = o_best;
        // printf("block (%zu, %zu, %zu) -> (%zu, %zu, %zu)\n", batch_size, input_dims, output_dims, batch_block, input_block, output_block);
    }
    
    template <typename E, typename T>
    Plaintext MatmulHelper::encode_weights_small(
        const E& encoder, const T* weights,
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const {
        std::vector<T> vec(input_block * output_block, 0);
        for (size_t j = lj; j < uj; j++) {
            for (size_t i = li; i < ui; i++) {
                size_t r = (j-lj) * input_block + input_block - (i-li) - 1;
                assert(r < slot_count);
                vec[r] = weights[i * output_dims + j];
            }
        }
        if (for_cipher) {
            return encoder.encode_for_cipher(vec, pool);
        } else {
            return encoder.encode_for_plain(vec, pool);
        }
    }

    template Plaintext MatmulHelper::encode_weights_small<BatchEncoderAdapter, uint64_t>(
        const BatchEncoderAdapter& encoder, const uint64_t* weights, 
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const;
    template Plaintext MatmulHelper::encode_weights_small<CKKSEncoderAdapter, double>(
        const CKKSEncoderAdapter& encoder, const double* weights, 
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const;
    template Plaintext MatmulHelper::encode_weights_small<PolynomialEncoderRing2kAdapter<uint32_t>, uint32_t>(
        const PolynomialEncoderRing2kAdapter<uint32_t>& encoder, const uint32_t* weights, 
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const;
    template Plaintext MatmulHelper::encode_weights_small<PolynomialEncoderRing2kAdapter<uint64_t>, uint64_t>(
        const PolynomialEncoderRing2kAdapter<uint64_t>& encoder, const uint64_t* weights, 
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const;
    template Plaintext MatmulHelper::encode_weights_small<PolynomialEncoderRing2kAdapter<uint128_t>, uint128_t>(
        const PolynomialEncoderRing2kAdapter<uint128_t>& encoder, const uint128_t* weights, 
        size_t li, size_t ui, size_t lj, size_t uj, bool for_cipher
    ) const;

    template <typename E, typename T>
    Plain2d MatmulHelper::encode_weights(const E& encoder, const T* weights, bool for_cipher) const {
        size_t height = input_dims, width = output_dims;
        size_t h = input_block, w = output_block;
        Plain2d encoded_weights;
        encoded_weights.data().clear();
        encoded_weights.data().reserve(ceil_div(height, h));
        for (size_t li = 0; li < height; li += h) {
            size_t ui = (li + h > height) ? height : (li + h);
            std::vector<Plaintext> encoded_row; encoded_row.reserve(ceil_div(width, w));
            for (size_t lj = 0; lj < width; lj += w) {
                size_t uj = (lj + w > width) ? width : (lj + w);
                encoded_row.push_back(
                    this->encode_weights_small(encoder, weights, li, ui, lj, uj, for_cipher)
                );
            }
            encoded_weights.data().push_back(std::move(encoded_row));
        }
        return encoded_weights;
    }

    template Plain2d MatmulHelper::encode_weights<BatchEncoderAdapter, uint64_t>(
        const BatchEncoderAdapter& encoder, const uint64_t* weights, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights<CKKSEncoderAdapter, double>(
        const CKKSEncoderAdapter& encoder, const double* weights, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights<PolynomialEncoderRing2kAdapter<uint32_t>, uint32_t>(
        const PolynomialEncoderRing2kAdapter<uint32_t>& encoder, const uint32_t* weights, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights<PolynomialEncoderRing2kAdapter<uint64_t>, uint64_t>(
        const PolynomialEncoderRing2kAdapter<uint64_t>& encoder, const uint64_t* weights, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights<PolynomialEncoderRing2kAdapter<uint128_t>, uint128_t>(
        const PolynomialEncoderRing2kAdapter<uint128_t>& encoder, const uint128_t* weights, bool for_cipher
    ) const;
    
    Plain2d MatmulHelper::encode_weights_uint64s(const BatchEncoder& encoder, const uint64_t* weights) const {
        BatchEncoderAdapter adapter(encoder);
        return encode_weights(adapter, weights, false);
    }
    Plain2d MatmulHelper::encode_weights_doubles(const CKKSEncoder& encoder, const double* weights, std::optional<ParmsID> parms_id, double scale) const {
        CKKSEncoderAdapter adapter(encoder, parms_id, scale);
        return encode_weights(adapter, weights, false);
    }
    template <typename T>
    Plain2d MatmulHelper::encode_weights_ring2k(const PolynomialEncoderRing2k<T>& encoder, const T* weights, std::optional<ParmsID> parms_id, bool for_cipher) const {
        PolynomialEncoderRing2kAdapter<T> adapter(encoder, parms_id);
        return encode_weights(adapter, weights, for_cipher);
    }
    template Plain2d MatmulHelper::encode_weights_ring2k<uint32_t>(
        const PolynomialEncoderRing2k<uint32_t>& encoder, const uint32_t* weights, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights_ring2k<uint64_t>(
        const PolynomialEncoderRing2k<uint64_t>& encoder, const uint64_t* weights, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_weights_ring2k<uint128_t>(
        const PolynomialEncoderRing2k<uint128_t>& encoder, const uint128_t* weights, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;

    template <typename E, typename T>
    Plain2d MatmulHelper::encode_inputs(const E& encoder, const T* inputs, bool for_cipher) const {
        size_t vecsize = input_block;
        Plain2d ret;
        ret.data().reserve(batch_size);
        for (size_t li = 0; li < batch_size; li += batch_block) {
            size_t ui = (li + batch_block > batch_size) ? batch_size : li + batch_block;
            std::vector<Plaintext> encoded_row;
            encoded_row.reserve(ceil_div(input_dims, vecsize));
            for (size_t lj = 0; lj < input_dims; lj += vecsize) {
                size_t uj = (lj + vecsize > input_dims) ? input_dims : lj + vecsize;
                std::vector<T> vec(slot_count, 0);
                for (size_t i = li; i < ui; i++)
                    for (size_t j = lj; j < uj; j++)
                        vec[(i - li) * input_block * output_block + (j - lj)] = inputs[i * input_dims + j];
                Plaintext encoded = for_cipher ? encoder.encode_for_cipher(vec, pool) : encoder.encode_for_plain(vec, pool);
                encoded_row.push_back(std::move(encoded));
            }
            ret.data().push_back(std::move(encoded_row));
        }
        return ret;
    }

    template Plain2d MatmulHelper::encode_inputs<BatchEncoderAdapter, uint64_t>(
        const BatchEncoderAdapter& encoder, const uint64_t* inputs, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs<CKKSEncoderAdapter, double>(
        const CKKSEncoderAdapter& encoder, const double* inputs, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs<PolynomialEncoderRing2kAdapter<uint32_t>, uint32_t>(
        const PolynomialEncoderRing2kAdapter<uint32_t>& encoder, const uint32_t* inputs, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs<PolynomialEncoderRing2kAdapter<uint64_t>, uint64_t>(
        const PolynomialEncoderRing2kAdapter<uint64_t>& encoder, const uint64_t* inputs, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs<PolynomialEncoderRing2kAdapter<uint128_t>, uint128_t>(
        const PolynomialEncoderRing2kAdapter<uint128_t>& encoder, const uint128_t* inputs, bool for_cipher
    ) const;

    Plain2d MatmulHelper::encode_inputs_uint64s(const BatchEncoder& encoder, const uint64_t* inputs) const {
        BatchEncoderAdapter adapter(encoder);
        return encode_inputs(adapter, inputs, true);
    }
    Plain2d MatmulHelper::encode_inputs_doubles(const CKKSEncoder& encoder, const double* inputs, std::optional<ParmsID> parms_id, double scale) const {
        CKKSEncoderAdapter adapter(encoder, parms_id, scale);
        return encode_inputs(adapter, inputs, true);
    }
    template <typename T>
    Plain2d MatmulHelper::encode_inputs_ring2k(const PolynomialEncoderRing2k<T>& encoder, const T* inputs, std::optional<ParmsID> parms_id, bool for_cipher) const {
        PolynomialEncoderRing2kAdapter<T> adapter(encoder, parms_id);
        return encode_inputs(adapter, inputs, for_cipher);
    }
    template Plain2d MatmulHelper::encode_inputs_ring2k<uint32_t>(
        const PolynomialEncoderRing2k<uint32_t>& encoder, const uint32_t* inputs, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs_ring2k<uint64_t>(
        const PolynomialEncoderRing2k<uint64_t>& encoder, const uint64_t* inputs, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;
    template Plain2d MatmulHelper::encode_inputs_ring2k<uint128_t>(
        const PolynomialEncoderRing2k<uint128_t>& encoder, const uint128_t* inputs, std::optional<ParmsID> parms_id, bool for_cipher
    ) const;

    Cipher2d MatmulHelper::encrypt_inputs_uint64s(const Encryptor& encryptor, const BatchEncoder& encoder, const uint64_t* inputs) const {
        Plain2d plain = encode_inputs_uint64s(encoder, inputs);
        return plain.encrypt_symmetric(encryptor, pool);
    }
    Cipher2d MatmulHelper::encrypt_inputs_doubles(const Encryptor& encryptor, const CKKSEncoder& encoder, const double* inputs, std::optional<ParmsID> parms_id, double scale) const {
        Plain2d plain = encode_inputs_doubles(encoder, inputs, parms_id, scale);
        return plain.encrypt_symmetric(encryptor, pool);
    }
    template <typename T>
    Cipher2d MatmulHelper::encrypt_inputs_ring2k(const Encryptor& encryptor, const PolynomialEncoderRing2k<T>& encoder, const T* inputs, std::optional<ParmsID> parms_id) const {
        Plain2d plain = encode_inputs_ring2k(encoder, inputs, parms_id, true);
        return plain.encrypt_symmetric(encryptor, pool);
    }
    template Cipher2d MatmulHelper::encrypt_inputs_ring2k<uint32_t>(
        const Encryptor& encryptor, const PolynomialEncoderRing2k<uint32_t>& encoder, const uint32_t* inputs, std::optional<ParmsID> parms_id
    ) const;
    template Cipher2d MatmulHelper::encrypt_inputs_ring2k<uint64_t>(
        const Encryptor& encryptor, const PolynomialEncoderRing2k<uint64_t>& encoder, const uint64_t* inputs, std::optional<ParmsID> parms_id
    ) const;
    template Cipher2d MatmulHelper::encrypt_inputs_ring2k<uint128_t>(
        const Encryptor& encryptor, const PolynomialEncoderRing2k<uint128_t>& encoder, const uint128_t* inputs, std::optional<ParmsID> parms_id
    ) const;

    Cipher2d MatmulHelper::matmul(const Evaluator& evaluator, const Cipher2d& a, const Plain2d& w) const {
        Cipher2d ret; ret.data().reserve(ceil_div(batch_size, batch_block));
        size_t outputVectorCount = ceil_div(output_dims, output_block);
        if (a.data().size() != ceil_div(batch_size, batch_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul] Input batch_size incorrect.");
        }
        if (w.data().size() != ceil_div(input_dims, input_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul] Weight input dimension incorrect.");
        }
        for (size_t b = 0; b < ceil_div(batch_size, batch_block); b++) {
            std::vector<Ciphertext> outVecs(outputVectorCount);
            for (size_t i = 0; i < w.data().size(); i++) {
                for (size_t j = 0; j < w[i].size(); j++) {
                    Ciphertext prod;
                    evaluator.multiply_plain(a[b][i], w[i][j], prod, pool);
                    if (i==0) outVecs[j] = std::move(prod);
                    else {
                        evaluator.add_inplace(outVecs[j], prod, pool);
                    }
                }
            }
            ret.data().push_back(std::move(outVecs));
        }
        return ret;
    }

    Cipher2d MatmulHelper::matmul_cipher(const Evaluator& evaluator, const Cipher2d& a, const Cipher2d& w) const {
        Cipher2d ret; ret.data().reserve(ceil_div(batch_size, batch_block));
        size_t outputVectorCount = ceil_div(output_dims, output_block);
        if (a.data().size() != ceil_div(batch_size, batch_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul_cipher] Input batch_size incorrect.");
        }
        if (w.data().size() != ceil_div(input_dims, input_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul_cipher] Weight input dimension incorrect.");
        }
        for (size_t b = 0; b < ceil_div(batch_size, batch_block); b++) {
            std::vector<Ciphertext> outVecs(outputVectorCount);
            for (size_t i = 0; i < w.data().size(); i++) {
                for (size_t j = 0; j < w[i].size(); j++) {
                    Ciphertext prod;
                    evaluator.multiply(a[b][i], w[i][j], prod, pool);
                    if (i==0) outVecs[j] = std::move(prod);
                    else {
                        evaluator.add_inplace(outVecs[j], prod, pool);
                    }
                }
            }
            ret.data().push_back(std::move(outVecs));
        }
        return ret;
    }

    Cipher2d MatmulHelper::matmul_reverse(const Evaluator& evaluator, const Plain2d& a, const Cipher2d& w) const {
        Cipher2d ret; ret.data().reserve(ceil_div(batch_size, batch_block));
        size_t outputVectorCount = ceil_div(output_dims, output_block);
        if (a.data().size() != ceil_div(batch_size, batch_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul_reverse] Input batch_size incorrect.");
        }
        if (w.data().size() != ceil_div(input_dims, input_block)) {
            throw std::invalid_argument("[MatmulHelper::matmul_reverse] Weight input dimension incorrect.");
        }
        for (size_t b = 0; b < ceil_div(batch_size, batch_block); b++) {
            std::vector<Ciphertext> outVecs(outputVectorCount);
            for (size_t i = 0; i < w.data().size(); i++) {
                for (size_t j = 0; j < w[i].size(); j++) {
                    Ciphertext prod;
                    evaluator.multiply_plain(w[i][j], a[b][i], prod, pool);
                    if (i==0) outVecs[j] = std::move(prod);
                    else {
                        evaluator.add_inplace(outVecs[j], prod, pool);
                    }
                }
            }
            ret.data().push_back(std::move(outVecs));
        }
        return ret;
    }

    template <typename E, typename T>
    Plain2d MatmulHelper::encode_outputs(const E& encoder, const T* outputs) const {
        size_t vecsize = output_block;
        if (!this->pack_lwe) {
            Plain2d ret; ret.data().reserve(batch_size);
            for (size_t li = 0; li < batch_size; li += batch_block) {
                size_t ui = (li + batch_block > batch_size) ? batch_size : (li + batch_block);
                std::vector<Plaintext> encoded_row;
                encoded_row.reserve(ceil_div(output_dims, vecsize));
                for (size_t lj = 0; lj < output_dims; lj += vecsize) {
                    size_t uj = (lj + vecsize > output_dims) ? output_dims : (lj + vecsize);
                    std::vector<T> buffer(slot_count, 0);
                    for (size_t i = li; i < ui; i++)
                        for (size_t j = lj; j < uj; j++) 
                            buffer[(i - li) * input_block * output_block + (j - lj) * input_block + input_block - 1] = outputs[i * output_dims + j];
                    Plaintext pt = encoder.encode_for_cipher(buffer, pool);
                    encoded_row.push_back(std::move(pt));
                }
                ret.data().push_back(std::move(encoded_row));
            }
            return ret;
        } else {
            Plain2d plain2d; plain2d.data().reserve(batch_size);
            plain2d.data().push_back(std::vector<Plaintext>());
            size_t batch_blockCount = ceil_div(this->batch_size, this->batch_block);
            size_t output_blockCount = ceil_div(this->output_dims, this->output_block);
            auto ret = std::vector<std::vector<T>>(ceil_div(batch_blockCount * output_blockCount, this->input_block), std::vector<T>(this->slot_count, 0)); 
            size_t li = 0; size_t di = 0; while (li < this->batch_size) {
                size_t ui = std::min(this->batch_size, li + this->batch_block);
                size_t lj = 0; size_t dj = 0; while (lj < this->output_dims) {
                    size_t uj = std::min(this->output_dims, lj + vecsize);
                    size_t cipherId = di * ceil_div(this->output_dims, this->output_block) + dj;
                    size_t packedId = cipherId / this->input_block;
                    size_t packedOffset = cipherId % this->input_block;
                    for (size_t i = li; i < ui; i++) {
                        for (size_t j = lj; j < uj; j++) {
                            ret[packedId][(i - li) * this->input_block * this->output_block + (j - lj) * this->input_block + packedOffset] 
                                = outputs[i * this->output_dims + j];
                        }
                    }
                    dj += 1;
                    lj += vecsize; 
                }
                di += 1;
                li += this->batch_block;
            }
            plain2d.data()[0].reserve(ret.size());
            for (size_t i = 0; i < ret.size(); i++) {
                Plaintext pt = encoder.encode_for_cipher(ret[i], pool);
                plain2d.data()[0].push_back(std::move(pt));
            }
            return plain2d;
        }
    }

    template Plain2d MatmulHelper::encode_outputs<BatchEncoderAdapter, uint64_t>(
        const BatchEncoderAdapter& encoder, const uint64_t* outputs
    ) const;
    template Plain2d MatmulHelper::encode_outputs<CKKSEncoderAdapter, double>(
        const CKKSEncoderAdapter& encoder, const double* outputs
    ) const;
    template Plain2d MatmulHelper::encode_outputs<PolynomialEncoderRing2kAdapter<uint32_t>, uint32_t>(
        const PolynomialEncoderRing2kAdapter<uint32_t>& encoder, const uint32_t* outputs
    ) const;
    template Plain2d MatmulHelper::encode_outputs<PolynomialEncoderRing2kAdapter<uint64_t>, uint64_t>(
        const PolynomialEncoderRing2kAdapter<uint64_t>& encoder, const uint64_t* outputs
    ) const;
    template Plain2d MatmulHelper::encode_outputs<PolynomialEncoderRing2kAdapter<uint128_t>, uint128_t>(
        const PolynomialEncoderRing2kAdapter<uint128_t>& encoder, const uint128_t* outputs
    ) const;

    Plain2d MatmulHelper::encode_outputs_uint64s(const BatchEncoder& encoder, const uint64_t* outputs) const {
        BatchEncoderAdapter adapter(encoder);
        return encode_outputs(adapter, outputs);
    }
    Plain2d MatmulHelper::encode_outputs_doubles(const CKKSEncoder& encoder, const double* outputs, std::optional<ParmsID> parms_id, double scale) const {
        CKKSEncoderAdapter adapter(encoder, parms_id, scale);
        return encode_outputs(adapter, outputs);
    }
    template <typename T>
    Plain2d MatmulHelper::encode_outputs_ring2k(const PolynomialEncoderRing2k<T>& encoder, const T* outputs, std::optional<ParmsID> parms_id) const {
        PolynomialEncoderRing2kAdapter<T> adapter(encoder, parms_id);
        return encode_outputs(adapter, outputs);
    }
    template Plain2d MatmulHelper::encode_outputs_ring2k<uint32_t>(
        const PolynomialEncoderRing2k<uint32_t>& encoder, const uint32_t* outputs, std::optional<ParmsID> parms_id
    ) const;
    template Plain2d MatmulHelper::encode_outputs_ring2k<uint64_t>(
        const PolynomialEncoderRing2k<uint64_t>& encoder, const uint64_t* outputs, std::optional<ParmsID> parms_id
    ) const;
    template Plain2d MatmulHelper::encode_outputs_ring2k<uint128_t>(
        const PolynomialEncoderRing2k<uint128_t>& encoder, const uint128_t* outputs, std::optional<ParmsID> parms_id
    ) const;

    template <typename E, typename T>
    std::vector<T> MatmulHelper::decrypt_outputs(const E& encoder, const Decryptor& decryptor, const Cipher2d& outputs) const {
        std::vector<T> dec(batch_size * output_dims);
        size_t vecsize = output_block;
        Plaintext pt;
        if (!this->pack_lwe) {
            size_t di = 0;
            for (size_t li = 0; li < batch_size; li += batch_block) {
                size_t ui = (li + batch_block > batch_size) ? batch_size : (li + batch_block);
                size_t dj = 0;
                for (size_t lj = 0; lj < output_dims; lj += vecsize) {
                    size_t uj = (lj + vecsize > output_dims) ? output_dims : (lj + vecsize);
                    std::vector<T> buffer = encoder.decrypt_outputs(decryptor, outputs[di][dj], pool);
                    for (size_t i = li; i < ui; i++)
                        for (size_t j = lj; j < uj; j++) 
                            dec[i * output_dims + j] = buffer[(i - li) * input_block * output_block + (j - lj) * input_block + input_block - 1];
                    dj += 1;
                }
                di += 1;
            }
        } else {
            std::vector<std::vector<T>> buffer;
            for (size_t i = 0; i < outputs.data()[0].size(); i++) {
                buffer.push_back(encoder.decrypt_outputs(decryptor, outputs[0][i], pool));
            }
            size_t li = 0; size_t di = 0; while (li < this->batch_size) {
                size_t ui = std::min(this->batch_size, li + this->batch_block);
                size_t lj = 0; size_t dj = 0; while (lj < this->output_dims) {
                    size_t uj = std::min(this->output_dims, lj + vecsize);
                    size_t cipherId = di * ceil_div(this->output_dims, this->output_block) + dj;
                    size_t packedId = cipherId / this->input_block;
                    size_t packedOffset = cipherId % this->input_block;
                    for (size_t i = li; i < ui; i++) {
                        for (size_t j = lj; j < uj; j++) {
                            dec[i * output_dims + j] = buffer[packedId][(i - li) * input_block * output_block + (j - lj) * input_block + packedOffset];
                        }
                    }
                    dj += 1;
                    lj += vecsize; 
                }
                di += 1;
                li += this->batch_block;
            }
        }
        return dec;
    }

    template std::vector<uint64_t> MatmulHelper::decrypt_outputs<BatchEncoderAdapter, uint64_t>(
        const BatchEncoderAdapter& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<double> MatmulHelper::decrypt_outputs<CKKSEncoderAdapter, double>(
        const CKKSEncoderAdapter& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<uint32_t> MatmulHelper::decrypt_outputs<PolynomialEncoderRing2kAdapter<uint32_t>, uint32_t>(
        const PolynomialEncoderRing2kAdapter<uint32_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<uint64_t> MatmulHelper::decrypt_outputs<PolynomialEncoderRing2kAdapter<uint64_t>, uint64_t>(
        const PolynomialEncoderRing2kAdapter<uint64_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<uint128_t> MatmulHelper::decrypt_outputs<PolynomialEncoderRing2kAdapter<uint128_t>, uint128_t>(
        const PolynomialEncoderRing2kAdapter<uint128_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;

    std::vector<uint64_t> MatmulHelper::decrypt_outputs_uint64s(const BatchEncoder& encoder, const Decryptor& decryptor, const Cipher2d& outputs) const {
        BatchEncoderAdapter adapter(encoder);
        return decrypt_outputs<BatchEncoderAdapter, uint64_t>(adapter, decryptor, outputs);
    }
    std::vector<double> MatmulHelper::decrypt_outputs_doubles(const CKKSEncoder& encoder, const Decryptor& decryptor, const Cipher2d& outputs) const {
        CKKSEncoderAdapter adapter(encoder, std::nullopt, 0);
        return decrypt_outputs<CKKSEncoderAdapter, double>(adapter, decryptor, outputs);
    }
    template <typename T>
    std::vector<T> MatmulHelper::decrypt_outputs_ring2k(const PolynomialEncoderRing2k<T>& encoder, const Decryptor& decryptor, const Cipher2d& outputs) const {
        PolynomialEncoderRing2kAdapter<T> adapter(encoder, std::nullopt);
        return decrypt_outputs<PolynomialEncoderRing2kAdapter<T>, T>(adapter, decryptor, outputs);
    }
    template std::vector<uint32_t> MatmulHelper::decrypt_outputs_ring2k<uint32_t>(
        const PolynomialEncoderRing2k<uint32_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<uint64_t> MatmulHelper::decrypt_outputs_ring2k<uint64_t>(
        const PolynomialEncoderRing2k<uint64_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;
    template std::vector<uint128_t> MatmulHelper::decrypt_outputs_ring2k<uint128_t>(
        const PolynomialEncoderRing2k<uint128_t>& encoder, const Decryptor& decryptor, const Cipher2d& outputs
    ) const;

    Cipher2d MatmulHelper::pack_outputs(const Evaluator& evaluator, const GaloisKeys& autoKey, const Cipher2d& cipher) const {
        if (!this->pack_lwe) {
            throw std::invalid_argument("[MatmulHelper::packOutputs] PackLwe not enabled");
        }
        if (cipher.data().size() == 0 || cipher.data()[0].size() == 0) {
            Cipher2d ret; ret.data().push_back(std::vector<Ciphertext>());
            return ret;
        }
        size_t packSlots = this->input_block;
        size_t totalCount = cipher.data().size() * cipher.data()[0].size();
        std::vector<Ciphertext> output; output.reserve(ceil_div(totalCount, packSlots));
        Ciphertext current; bool currentSet = false;
        size_t currentSlot = 0;

        bool is_ntt = cipher.data()[0][0].is_ntt_form();
        
        size_t field_trace_logn = 0;
        size_t field_trace_n = 1;
        while (field_trace_n != slot_count / packSlots) {
            field_trace_logn += 1;
            field_trace_n *= 2;
        }

        Ciphertext buffer = cipher.data()[0][0].clone(pool);
        Ciphertext shifted = buffer.clone(pool);
        for (size_t i = 0; i < cipher.data().size(); i++) {
            for (size_t j = 0; j < cipher.data()[0].size(); j++) {
                size_t shift = packSlots - 1;
                Ciphertext ciphertext = cipher.data()[i][j].clone(pool);
                if (is_ntt) evaluator.transform_from_ntt_inplace(ciphertext);
                if (shift != 0) {
                    evaluator.negacyclic_shift(ciphertext, 2 * slot_count - shift, buffer, pool);
                } else {
                    buffer = ciphertext.clone(pool);
                }
                
                evaluator.divide_by_poly_modulus_degree_inplace(buffer, slot_count / packSlots);
                if (is_ntt) evaluator.transform_to_ntt_inplace(buffer);
                
                evaluator.field_trace_inplace(buffer, autoKey, field_trace_logn, pool);
                if (is_ntt) evaluator.transform_from_ntt_inplace(buffer);
                
                shift = currentSlot;
                if (shift != 0) {
                    evaluator.negacyclic_shift(buffer, shift, shifted, pool);
                } else {
                    shifted = buffer.clone(pool);
                }

                if (currentSet == false) {
                    current = shifted.clone(pool);
                    currentSet = true;
                } else {
                    evaluator.add_inplace(current, shifted, pool);
                }

                currentSlot += 1;
                if (currentSlot == packSlots) {
                    currentSlot = 0; currentSet = false;
                    output.push_back(std::move(current));
                }
            }
        }
        if (currentSet) {
            output.push_back(std::move(current));
        }
        if (is_ntt) for (Ciphertext& c : output) {
            evaluator.transform_to_ntt_inplace(c);
        }
        Cipher2d ret; ret.data().push_back(output);
        return ret;
    }

    void MatmulHelper::serialize_encoded_weights(const Plain2d& w, std::ostream& stream, CompressionMode mode) const {
        size_t rows = w.data().size();
        size_t cols = w[0].size();
        if (rows == 0) throw std::invalid_argument("[MatmulHelper::serialize_encoded_weights] No rows in weight matrix.");
        if (cols == 0) throw std::invalid_argument("[MatmulHelper::serialize_encoded_weights] No columns in weight matrix.");
        for (size_t i=0; i<rows; i++) {
            if (w[i].size() != cols) throw std::invalid_argument("[MatmulHelper::serialize_encoded_weights] Weight matrix is not rectangular.");
        }
        serialize::save_object(stream, rows);
        serialize::save_object(stream, cols);
        for (size_t i = 0; i < rows; i++) {
            for (size_t j = 0; j < cols; j++) {
                w[i][j].save(stream, mode);
            }
        }
    }

    Plain2d MatmulHelper::deserialize_encoded_weights(std::istream& stream) const {
        size_t rows, cols;
        serialize::load_object(stream, rows);
        serialize::load_object(stream, cols);
        Plain2d ret; ret.data().reserve(rows);
        for (size_t i = 0; i < rows; i++) {
            std::vector<Plaintext> row; row.reserve(cols);
            for (size_t j = 0; j < cols; j++) {
                Plaintext pt;
                pt.load(stream, pool);
                row.push_back(std::move(pt));
            }
            ret.data().push_back(std::move(row));
        }
        return ret;
    }

    void MatmulHelper::serialize_outputs(const Evaluator &evaluator, const Cipher2d& x, std::ostream& stream, CompressionMode mode) const {
        HeContextPointer context = evaluator.context();
        if (!this->pack_lwe) {
            size_t vecsize = output_block;
            Plaintext pt;
            size_t di = 0;
            for (size_t li = 0; li < batch_size; li += batch_block) {
                size_t ui = (li + batch_block > batch_size) ? batch_size : (li + batch_block);
                size_t dj = 0;
                for (size_t lj = 0; lj < output_dims; lj += vecsize) {
                    size_t uj = (lj + vecsize > output_dims) ? output_dims : (lj + vecsize);
                    std::vector<size_t> required((ui - li) * (uj - lj)); size_t rid = 0;
                    for (size_t i = li; i < ui; i++)
                        for (size_t j = lj; j < uj; j++) 
                            required[rid++] = (i - li) * input_block * output_block + (j - lj) * input_block + input_block - 1;
                    x[di][dj].save_terms(stream, context, required, pool, mode);
                    dj += 1;
                }
                di += 1;
            }
        } else {
            size_t count = ceil_div(batch_size, batch_block) * ceil_div(output_dims, output_block);
            count = ceil_div(count, input_block);
            if (count != x.data()[0].size()) {
                throw std::invalid_argument("[MatmulHelper::serialize_outputs] Output ciphertext count incorrect");
            }
            for (size_t i = 0; i < x.data()[0].size(); i++) {
                x[0][i].save(stream, context, mode);
            }
        }
    }

    Cipher2d MatmulHelper::deserialize_outputs(const Evaluator &evaluator, std::istream& stream) const {
        HeContextPointer context = evaluator.context();
        if (!this->pack_lwe) {
            size_t vecsize = output_block;
            Plaintext pt;
            Cipher2d ret; ret.data().reserve(ceil_div(batch_size, batch_block));
            for (size_t li = 0; li < batch_size; li += batch_block) {
                size_t ui = (li + batch_block > batch_size) ? batch_size : (li + batch_block);
                std::vector<Ciphertext> row; row.reserve(ceil_div(output_dims, vecsize));
                for (size_t lj = 0; lj < output_dims; lj += vecsize) {
                    size_t uj = (lj + vecsize > output_dims) ? output_dims : (lj + vecsize);
                    std::vector<size_t> required((ui - li) * (uj - lj)); size_t rid = 0;
                    for (size_t i = li; i < ui; i++)
                        for (size_t j = lj; j < uj; j++) 
                            required[rid++] = (i - li) * input_block * output_block + (j - lj) * input_block + input_block - 1;
                    Ciphertext c;
                    c.load_terms(stream, context, required, pool);
                    row.push_back(std::move(c));
                }
                ret.data().push_back(std::move(row));
            }
            return ret;
        } else {
            size_t count = ceil_div(batch_size, batch_block) * ceil_div(output_dims, output_block);
            count = ceil_div(count, input_block);
            Cipher2d ret; ret.data().push_back(std::vector<Ciphertext>());
            ret[0].reserve(count);
            for (size_t i = 0; i < count; i++) {
                Ciphertext c; c.load(stream, context, pool);
                ret[0].push_back(std::move(c));
            }
            return ret;
        }
    }

}}