#include "hip/hip_runtime.h"
#include "evaluator.cuh"
#include "utils/polynomial_buffer.cuh"

namespace troy {

    using utils::Slice;
    using utils::ConstSlice;
    using utils::Array;
    using utils::NTTTables;
    using utils::Pointer;
    using utils::ConstPointer;
    using utils::RNSTool;
    using utils::Buffer;
    using utils::MultiplyUint64Operand;
    using utils::GaloisTool;

    template <typename C>
    inline static void check_no_seed(const char* prompt, const C& c) {
        if (c.contains_seed()) {
            throw std::invalid_argument(std::string(prompt) + " Argument contains seed.");
        }
    }

    inline void check_ciphertext(const char* prompt, const Ciphertext& ciphertext) {
        check_no_seed(prompt, ciphertext);
    }

    template <typename C1, typename C2>
    inline static void check_same_parms_id(const char* prompt, const C1& a, const C2& b) {
        if (a.parms_id() != b.parms_id()) {
            throw std::invalid_argument(std::string(prompt) + " Arguments have different parms ID.");
        }
    }

    inline static void check_same_scale(const char* prompt, const Ciphertext& a, const Ciphertext& b) {
        if (!utils::are_close_double(a.scale(), b.scale())) {
            throw std::invalid_argument(std::string(prompt) + " Arguments have different scales.");
        }
    }

    inline static void check_same_ntt_form(const char* prompt, const Ciphertext& a, const Ciphertext& b) {
        if (a.is_ntt_form() != b.is_ntt_form()) {
            throw std::invalid_argument(std::string(prompt) + " Arguments have different NTT form.");
        }
    }

    inline static void check_is_ntt_form(const char* prompt, const Ciphertext& a) {
        if (!a.is_ntt_form()) {
            throw std::invalid_argument(std::string(prompt) + " Argument is not in NTT form.");
        }
    }

    inline static void check_is_not_ntt_form(const char* prompt, const Ciphertext& a) {
        if (a.is_ntt_form()) {
            throw std::invalid_argument(std::string(prompt) + " Argument is in NTT form.");
        }
    }

    static void balance_correction_factors(
        uint64_t factor1, uint64_t factor2, const Modulus& plain_modulus,
        uint64_t& prod, uint64_t& e1, uint64_t& e2
    ) {
        uint64_t t = plain_modulus.value();
        uint64_t half_t = t >> 1;
        auto sum_abs = [half_t, t](uint64_t x, uint64_t y) -> uint64_t {
            int64_t x_bal = x > half_t ? static_cast<int64_t>(x - t) : static_cast<int64_t>(x);
            int64_t y_bal = y > half_t ? static_cast<int64_t>(y - t) : static_cast<int64_t>(y);
            return static_cast<uint64_t>(std::abs(x_bal) + std::abs(y_bal));
        };
        uint64_t ratio = 1;
        if (!utils::try_invert_uint64_mod(factor1, plain_modulus, ratio)) {
            throw std::logic_error("[balance_correction_factors] Failed to invert factor1.");
        }
        ratio = utils::multiply_uint64_mod(ratio, factor2, plain_modulus);
        e1 = ratio;
        e2 = 1;
        uint64_t sum = sum_abs(factor1, factor2);
        
        // Extended Euclidean
        int64_t prev_a = static_cast<int64_t>(plain_modulus.value());
        int64_t prev_b = static_cast<int64_t>(0);
        int64_t a = static_cast<int64_t>(ratio);
        int64_t b = static_cast<int64_t>(1);
        while (a != 0) {
            int64_t q = prev_a / a;
            int64_t temp = prev_a % a;
            prev_a = a;
            a = temp;
            temp = prev_b - q * b;
            prev_b = b;
            b = temp;
            uint64_t a_mod = plain_modulus.reduce(static_cast<uint64_t>(std::abs(a)));
            if (a < 0) {a_mod = utils::negate_uint64_mod(a_mod, plain_modulus);}
            uint64_t b_mod = plain_modulus.reduce(static_cast<uint64_t>(std::abs(b)));
            if (b < 0) {b_mod = utils::negate_uint64_mod(b_mod, plain_modulus);}
            if ((a_mod != 0) && (utils::gcd(a_mod, t) == 1)) {
                uint64_t new_sum = sum_abs(a_mod, b_mod);
                if (new_sum < sum) {
                    e1 = a_mod;
                    e2 = b_mod;
                    sum = new_sum;
                }
            }
        }
        prod = utils::multiply_uint64_mod(e1, factor1, plain_modulus);
    }

    static bool is_scale_within_bounds(double scale, ContextDataPointer context_data) {
        SchemeType scheme = context_data->parms().scheme();
        int scale_bit_count_bound = -1;
        switch (scheme) {
            case SchemeType::BFV: case SchemeType::BGV: {
                scale_bit_count_bound = static_cast<int>(context_data->parms().plain_modulus_host().bit_count());
                break;
            }
            case SchemeType::CKKS: {
                scale_bit_count_bound = static_cast<int>(context_data->total_coeff_modulus_bit_count());
                break;
            }
        }
        // std::cerr << static_cast<int>(std::log2(scale)) << " " << scale_bit_count_bound << std::endl;
        return !(scale <= 0.0 || static_cast<int>(std::log2(scale)) >= scale_bit_count_bound);
    }

    ContextDataPointer Evaluator::get_context_data(const char* prompt, const ParmsID& encrypted) const {
        auto context_data_ptr = context_->get_context_data(encrypted);
        if (!context_data_ptr.has_value()) {
            throw std::invalid_argument(std::string(prompt) + " Context data not found parms id.");
        }
        return context_data_ptr.value();
    }

    void Evaluator::negate_inplace(Ciphertext& encrypted) const {
        check_ciphertext("[Evaluator::negate_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negate_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t poly_count = encrypted.polynomial_count();
        size_t poly_degree = parms.poly_modulus_degree();
        utils::negate_inplace_ps(encrypted.data().reference(), poly_count, poly_degree, coeff_modulus);
    }

    void Evaluator::translate_inplace(Ciphertext& encrypted1, const Ciphertext& encrypted2, bool subtract) const {
        check_ciphertext("[Evaluator::translate_inplace]", encrypted1);
        check_ciphertext("[Evaluator::translate_inplace]", encrypted2);
        check_same_parms_id("[Evaluator::translate_inplace]", encrypted1, encrypted2);
        check_same_scale("[Evaluator::translate_inplace]", encrypted1, encrypted2);
        check_same_ntt_form("[Evaluator::translate_inplace]", encrypted1, encrypted2);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::translate_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t enc1_size = encrypted1.polynomial_count();
        size_t enc2_size = encrypted2.polynomial_count();
        size_t max_size = std::max(enc1_size, enc2_size);
        size_t min_size = std::min(enc1_size, enc2_size);
        size_t coeff_count = parms.poly_modulus_degree();

        if (encrypted1.correction_factor() != 1 || encrypted2.correction_factor() != 1) {
            // Balance correction factors and multiply by scalars before addition in BGV
            uint64_t f0, f1, f2;
            const Modulus& plain_modulus = parms.plain_modulus_host();
            balance_correction_factors(
                encrypted1.correction_factor(), encrypted2.correction_factor(),
                plain_modulus, f0, f1, f2
            );
            utils::multiply_scalar_inplace_ps(encrypted1.data().reference(), f1, enc1_size, coeff_count, coeff_modulus);
            Ciphertext encrypted2_copy = encrypted2;
            utils::multiply_scalar_inplace_ps(encrypted2_copy.data().reference(), f2, enc2_size, coeff_count, coeff_modulus); 
            // Set new correction factor
            encrypted1.correction_factor() = f0;
            encrypted2_copy.correction_factor() = f0;
            this->translate_inplace(encrypted1, encrypted2_copy, subtract);
        } else {
            // Prepare destination
            encrypted1.resize(this->context(), context_data->parms_id(), max_size);
            if (!subtract) {
                utils::add_inplace_ps(encrypted1.data().reference(), encrypted2.data().const_reference(), min_size, coeff_count, coeff_modulus);
            } else {
                utils::sub_inplace_ps(encrypted1.data().reference(), encrypted2.data().const_reference(), min_size, coeff_count, coeff_modulus);
            }
            // Copy the remainding polys of the array with larger count into encrypted1
            if (enc1_size < enc2_size) {
                if (!subtract) {
                    encrypted1.polys(enc1_size, enc2_size).copy_from_slice(encrypted2.polys(enc1_size, enc2_size));
                } else {
                    utils::negate_ps(encrypted2.polys(enc1_size, enc2_size), enc2_size - enc1_size, coeff_count, coeff_modulus, encrypted1.polys(enc1_size, enc2_size));
                }
            }
        }
    }

    void Evaluator::bfv_multiply_inplace(Ciphertext& encrypted1, const Ciphertext& encrypted2) const {
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted1);
        check_is_not_ntt_form("[Evaluator::bfv_multiply_inplace]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        ConstSlice<Modulus> base_Bsk_m_tilde = rns_tool.base_Bsk_m_tilde().base();
        size_t base_Bsk_m_tilde_size = base_Bsk_m_tilde.size();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted1.on_device();
        encrypted1.resize(this->context(), context_data->parms_id(), dest_size);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_q(encrypted1_size, base_q_size, coeff_count, device);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted1_Bsk(encrypted1_size, base_Bsk_size, coeff_count, device);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        encrypted1_q.copy_from_slice(encrypted1.const_polys(0, encrypted1_size));
        // Lazy reduction
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted1_q.reference(), encrypted1_size, coeff_count, base_q_ntt_tables);
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        Buffer<uint64_t> temp(base_Bsk_m_tilde_size, coeff_count, device);
        for (size_t i = 0; i < encrypted1_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde(encrypted1.const_poly(i), temp.reference());
            // (2) Reduce q-overflows in with Montgomery reduction, switching base to Bsk
            rns_tool.sm_mrq(temp.const_reference(), encrypted1_Bsk.poly(i));
        }
        // Transform to NTT form in base Bsk
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted1_Bsk.reference(), encrypted1_size, coeff_count, base_Bsk_ntt_tables);

        // Repeat for encrypted2
        Buffer<uint64_t> encrypted2_q(encrypted2_size, base_q_size, coeff_count, device);
        Buffer<uint64_t> encrypted2_Bsk(encrypted2_size, base_Bsk_size, coeff_count, device);
        encrypted2_q.copy_from_slice(encrypted2.polys(0, encrypted2_size));
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted2_q.reference(), encrypted2_size, coeff_count, base_q_ntt_tables);
        for (size_t i = 0; i < encrypted2_size; i++) {
            rns_tool.fast_b_conv_m_tilde(encrypted2.poly(i), temp.reference());
            rns_tool.sm_mrq(temp.const_reference(), encrypted2_Bsk.poly(i));
        }
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted2_Bsk.reference(), encrypted2_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device);

        // Perform BEHZ step (4): dyadic multiplication on arbitrary size ciphertexts
        Buffer<uint64_t> temp1(base_q_size, coeff_count, device);
        Buffer<uint64_t> temp2(base_Bsk_size, coeff_count, device);
        for (size_t i = 0; i < dest_size; i++) {
            // We iterate over relevant components of encrypted1 and encrypted2 in increasing order for
            // encrypted1 and reversed (decreasing) order for encrypted2. The bounds for the indices of
            // the relevant terms are obtained as follows.
            size_t curr_encrypted1_last = std::min(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;

            // Perform the BEHZ ciphertext product both for base q and base Bsk
            for (size_t j = 0; j < steps; j++) {
                utils::dyadic_product_p(
                    encrypted1_q.const_poly(curr_encrypted1_first + j),
                    encrypted2_q.const_poly(curr_encrypted2_first - j),
                    coeff_count,
                    base_q,
                    temp1.reference()
                );
                utils::add_inplace_p(
                    temp_dest_q.poly(i),
                    temp1.const_reference(),
                    coeff_count,
                    base_q
                );
            }
            for (size_t j = 0; j < steps; j++) {
                utils::dyadic_product_p(
                    encrypted1_Bsk.const_poly(curr_encrypted1_first + j),
                    encrypted2_Bsk.const_poly(curr_encrypted2_first - j),
                    coeff_count,
                    base_Bsk,
                    temp2.reference()
                );
                utils::add_inplace_p(
                    temp_dest_Bsk.poly(i),
                    temp2.const_reference(),
                    coeff_count,
                    base_Bsk
                );
            }
        }

        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::inverse_ntt_negacyclic_harvey_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::inverse_ntt_negacyclic_harvey_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        Buffer<uint64_t> temp_q_Bsk(base_q_size + base_Bsk_size, coeff_count, device);
        Buffer<uint64_t> temp_Bsk(base_Bsk_size, coeff_count, device);
        uint64_t plain_modulus_value = parms.plain_modulus_host().value();
        for (size_t i = 0; i < dest_size; i++) {
            // Bring together the base q and base Bsk components into a single allocation
            // Step (6): multiply base q components by t (plain_modulus)
            utils::multiply_scalar_p(
                temp_dest_q.const_slice(i*coeff_count*base_q_size, (i+1)*coeff_count*base_q_size),
                plain_modulus_value,
                coeff_count,
                base_q,
                temp_q_Bsk.components(0, base_q_size)
            );
            utils::multiply_scalar_p(
                temp_dest_Bsk.const_slice(i*coeff_count*base_Bsk_size, (i+1)*coeff_count*base_Bsk_size),
                plain_modulus_value,
                coeff_count,
                base_Bsk,
                temp_q_Bsk.components(base_q_size, base_q_size + base_Bsk_size)
            );
            // Step (7): divide by q and floor, producing a result in base Bsk
            rns_tool.fast_floor(temp_q_Bsk.const_reference(), temp_Bsk.reference());
            // Step (8): use Shenoy-Kumaresan method to convert the result to base q and write to encrypted1
            rns_tool.fast_b_conv_sk(temp_Bsk.const_reference(), encrypted1.poly(i));
        }
    }
    
    void Evaluator::ckks_multiply_inplace(Ciphertext& encrypted1, const Ciphertext& encrypted2) const {
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted1);
        check_is_ntt_form("[Evaluator::ckks_multiply_inplace]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;

        encrypted1.resize(this->context(), context_data->parms_id(), dest_size);
        bool device = encrypted1.on_device();
        Buffer<uint64_t> temp(dest_size, coeff_modulus_size, coeff_count, device);

        Buffer<uint64_t> prod(coeff_modulus_size, coeff_count, device);
        for (size_t i = 0; i < dest_size; i++) {
            // We iterate over relevant components of encrypted1 and encrypted2 in increasing order for
            // encrypted1 and reversed (decreasing) order for encrypted2. The bounds for the indices of
            // the relevant terms are obtained as follows.
            size_t curr_encrypted1_last = std::min(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            // let curr_encrypted2_last = i - curr_encrypted1_last;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;

            for (size_t j = 0; j < steps; j++) {
                utils::dyadic_product_p(
                    encrypted1.const_poly(curr_encrypted1_first + j),
                    encrypted2.const_poly(curr_encrypted2_first - j),
                    coeff_count,
                    coeff_modulus,
                    prod.reference()
                );
                utils::add_inplace_p(
                    temp.poly(i),
                    prod.const_reference(),
                    coeff_count,
                    coeff_modulus
                );
            }
        }

        encrypted1.polys(0, dest_size).copy_from_slice(temp.const_reference());
        encrypted1.scale() = encrypted1.scale() * encrypted2.scale();
        if (!is_scale_within_bounds(encrypted1.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply_inplace] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_multiply_inplace(Ciphertext& encrypted1, const Ciphertext& encrypted2) const {
        check_is_not_ntt_form("[Evaluator::bgv_multiply_inplace]", encrypted1);
        check_is_not_ntt_form("[Evaluator::bgv_multiply_inplace]", encrypted2);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_multiply_inplace]", encrypted1.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted1_size = encrypted1.polynomial_count();
        size_t encrypted2_size = encrypted2.polynomial_count();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        
        // Determine destination.size()
        size_t dest_size = encrypted1_size + encrypted2_size - 1;

        encrypted1.resize(this->context(), context_data->parms_id(), dest_size);
        bool device = encrypted1.on_device();

        utils::ntt_negacyclic_harvey_ps(
            encrypted1.polys(0, encrypted1_size), 
            encrypted1_size, coeff_count, ntt_tables
        );
        Ciphertext encrypted2_copy = encrypted2;
        utils::ntt_negacyclic_harvey_ps(
            encrypted2_copy.polys(0, encrypted2_size), 
            encrypted2_size, coeff_count, ntt_tables
        );
        Buffer<uint64_t> temp(dest_size, coeff_modulus_size, coeff_count, device);

        Buffer<uint64_t> prod(coeff_modulus_size, coeff_count, device);
        for (size_t i = 0; i < dest_size; i++) {
            // We iterate over relevant components of encrypted1 and encrypted2 in increasing order for
            // encrypted1 and reversed (decreasing) order for encrypted2. The bounds for the indices of
            // the relevant terms are obtained as follows.
            size_t curr_encrypted1_last = std::min(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            // let curr_encrypted2_last = i - curr_encrypted1_last;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;

            for (size_t j = 0; j < steps; j++) {
                utils::dyadic_product_p(
                    encrypted1.const_poly(curr_encrypted1_first + j),
                    encrypted2_copy.const_poly(curr_encrypted2_first - j),
                    coeff_count,
                    coeff_modulus,
                    prod.reference()
                );
                utils::add_inplace_p(
                    temp.poly(i),
                    prod.const_reference(),
                    coeff_count,
                    coeff_modulus
                );
            }
        }
        
        encrypted1.polys(0, dest_size).copy_from_slice(temp.const_reference());
        utils::inverse_ntt_negacyclic_harvey_ps(
            encrypted1.polys(0, dest_size), 
            dest_size, coeff_count, ntt_tables
        );
        encrypted1.correction_factor() = utils::multiply_uint64_mod(
            encrypted1.correction_factor(),
            encrypted2.correction_factor(),
            parms.plain_modulus_host()
        );
    }

    void Evaluator::multiply_inplace(Ciphertext& encrypted1, const Ciphertext& encrypted2) const {
        check_no_seed("[Evaluator::multiply_inplace]", encrypted1);
        check_no_seed("[Evaluator::multiply_inplace]", encrypted2);
        check_same_parms_id("[Evaluator::multiply_inplace]", encrypted1, encrypted2);
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_multiply_inplace(encrypted1, encrypted2);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_multiply_inplace(encrypted1, encrypted2);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_multiply_inplace(encrypted1, encrypted2);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::multiply_inplace] Scheme not implemented.");
            }
        }
    }

    void Evaluator::bfv_square_inplace(Ciphertext& encrypted) const {
        check_is_not_ntt_form("[Evaluator::bfv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bfv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> base_q = parms.coeff_modulus();
        size_t base_q_size = base_q.size();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->bfv_multiply_inplace(encrypted, encrypted);
            return;
        }
        
        const RNSTool& rns_tool = context_data->rns_tool();
        ConstSlice<Modulus> base_Bsk = rns_tool.base_Bsk().base();
        size_t base_Bsk_size = base_Bsk.size();
        ConstSlice<Modulus> base_Bsk_m_tilde = rns_tool.base_Bsk_m_tilde().base();
        size_t base_Bsk_m_tilde_size = base_Bsk_m_tilde.size();
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;
        ConstSlice<NTTTables> base_q_ntt_tables = context_data->small_ntt_tables();
        ConstSlice<NTTTables> base_Bsk_ntt_tables = rns_tool.base_Bsk_ntt_tables();
        
        // Microsoft SEAL uses BEHZ-style RNS multiplication. This process is somewhat complex and consists of the
        // following steps:
        //
        // (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
        // (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
        // (3) Transform the data to NTT form
        // (4) Compute the ciphertext polynomial product using dyadic multiplication
        // (5) Transform the data back from NTT form
        // (6) Multiply the result by t (plain_modulus)
        // (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
        // (8) Use Shenoy-Kumaresan method to convert the result to base q

        bool device = encrypted.on_device();
        encrypted.resize(this->context(), context_data->parms_id(), dest_size);
        // Allocate space for a base q output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_q(encrypted_size, base_q_size, coeff_count, device);
        // Allocate space for a base Bsk output of behz_extend_base_convertToNtt for encrypted1
        Buffer<uint64_t> encrypted_Bsk(encrypted_size, base_Bsk_size, coeff_count, device);

        // Perform BEHZ steps (1)-(3) for encrypted1
        // Make copy of input polynomial (in base q) and convert to NTT form
        encrypted_q.copy_from_slice(encrypted.const_polys(0, encrypted_size));
        // Lazy reduction
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted_q.reference(), encrypted_size, coeff_count, base_q_ntt_tables);
        // Allocate temporary space for a polynomial in the Bsk U {m_tilde} base
        Buffer<uint64_t> temp(base_Bsk_m_tilde_size, coeff_count, device);
        for (size_t i = 0; i < encrypted_size; i++) {
            // (1) Convert from base q to base Bsk U {m_tilde}
            rns_tool.fast_b_conv_m_tilde(encrypted.const_poly(i), temp.reference());
            // (2) Reduce q-overflows in with Montgomery reduction, switching base to Bsk
            rns_tool.sm_mrq(temp.const_reference(), encrypted_Bsk.poly(i));
        }
        // Transform to NTT form in base Bsk
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted_Bsk.reference(), encrypted_size, coeff_count, base_Bsk_ntt_tables);

        // Allocate temporary space for the output of step (4)
        // We allocate space separately for the base q and the base Bsk components
        Buffer<uint64_t> temp_dest_q(dest_size, base_q_size, coeff_count, device);
        Buffer<uint64_t> temp_dest_Bsk(dest_size, base_Bsk_size, coeff_count, device);

        // Perform the BEHZ ciphertext square both for base q and base Bsk

        // Compute c0^2
        Slice<uint64_t> eq0 = encrypted_q.poly(0);
        Slice<uint64_t> eq1 = encrypted_q.poly(1);
        utils::dyadic_product_p(eq0.as_const(), eq0.as_const(), coeff_count, base_q, temp_dest_q.poly(0));
        // Compute 2*c0*c1
        utils::dyadic_product_p(eq0.as_const(), eq1.as_const(), coeff_count, base_q, temp_dest_q.poly(1));
        utils::add_inplace_p(temp_dest_q.poly(1), temp_dest_q.const_poly(1), coeff_count, base_q);
        // Compute c1^2
        utils::dyadic_product_p(eq1.as_const(), eq1.as_const(), coeff_count, base_q, temp_dest_q.poly(2));

        Slice<uint64_t> eb0 = encrypted_Bsk.poly(0);
        Slice<uint64_t> eb1 = encrypted_Bsk.poly(1);
        utils::dyadic_product_p(eb0.as_const(), eb0.as_const(), coeff_count, base_Bsk, temp_dest_Bsk.poly(0));
        utils::dyadic_product_p(eb0.as_const(), eb1.as_const(), coeff_count, base_Bsk, temp_dest_Bsk.poly(1));
        utils::add_inplace_p(temp_dest_Bsk.poly(1), temp_dest_Bsk.const_poly(1), coeff_count, base_Bsk);
        utils::dyadic_product_p(eb1.as_const(), eb1.as_const(), coeff_count, base_Bsk, temp_dest_Bsk.poly(2));
        
        // Perform BEHZ step (5): transform data from NTT form
        // Lazy reduction here. The following multiplyPolyScalarCoeffmod will correct the value back to [0, p)
        utils::inverse_ntt_negacyclic_harvey_ps(temp_dest_q.reference(), dest_size, coeff_count, base_q_ntt_tables);
        utils::inverse_ntt_negacyclic_harvey_ps(temp_dest_Bsk.reference(), dest_size, coeff_count, base_Bsk_ntt_tables);

        // Perform BEHZ steps (6)-(8)
        Buffer<uint64_t> temp_q_Bsk(base_q_size + base_Bsk_size, coeff_count, device);
        Buffer<uint64_t> temp_Bsk(base_Bsk_size, coeff_count, device);
        uint64_t plain_modulus_value = parms.plain_modulus_host().value();
        for (size_t i = 0; i < dest_size; i++) {
            // Bring together the base q and base Bsk components into a single allocation
            // Step (6): multiply base q components by t (plain_modulus)
            utils::multiply_scalar_p(
                temp_dest_q.const_slice(i*coeff_count*base_q_size, (i+1)*coeff_count*base_q_size),
                plain_modulus_value,
                coeff_count,
                base_q,
                temp_q_Bsk.components(0, base_q_size)
            );
            utils::multiply_scalar_p(
                temp_dest_Bsk.const_slice(i*coeff_count*base_Bsk_size, (i+1)*coeff_count*base_Bsk_size),
                plain_modulus_value,
                coeff_count,
                base_Bsk,
                temp_q_Bsk.components(base_q_size, base_q_size + base_Bsk_size)
            );
            // Step (7): divide by q and floor, producing a result in base Bsk
            rns_tool.fast_floor(temp_q_Bsk.const_reference(), temp_Bsk.reference());
            // Step (8): use Shenoy-Kumaresan method to convert the result to base q and write to encrypted1
            rns_tool.fast_b_conv_sk(temp_Bsk.const_reference(), encrypted.poly(i));
        }
    }

    void Evaluator::ckks_square_inplace(Ciphertext& encrypted) const {
        check_is_ntt_form("[Evaluator::ckks_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::ckks_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->ckks_multiply_inplace(encrypted, encrypted);
            return;
        }
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;

        encrypted.resize(this->context(), context_data->parms_id(), dest_size);
        bool device = encrypted.on_device();
        
        Slice<uint64_t> c0 = encrypted.poly(0);
        Slice<uint64_t> c1 = encrypted.poly(1);
        Slice<uint64_t> c2 = encrypted.poly(2);
        
        utils::dyadic_product_p(c1.as_const(), c1.as_const(), coeff_count, coeff_modulus, c2);
        utils::dyadic_product_p(c0.as_const(), c1.as_const(), coeff_count, coeff_modulus, c1);
        utils::add_inplace_p(   c1,            c1.as_const(), coeff_count, coeff_modulus);
        utils::dyadic_product_p(c0.as_const(), c0.as_const(), coeff_count, coeff_modulus, c0);

        encrypted.scale() = encrypted.scale() * encrypted.scale();
        if (!is_scale_within_bounds(encrypted.scale(), context_data)) {
            throw std::invalid_argument("[Evaluator::ckks_multiply_inplace] Scale out of bounds");
        }
    }
    
    void Evaluator::bgv_square_inplace(Ciphertext& encrypted) const {
        check_is_not_ntt_form("[Evaluator::bgv_square_inplace]", encrypted);
        
        // Extract encryption parameters.
        ContextDataPointer context_data = this->get_context_data("[Evaluator::bgv_square_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.polynomial_count();

        if (encrypted_size != 2) {
            this->bgv_multiply_inplace(encrypted, encrypted);
            return;
        }
        
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        
        // Determine destination.size()
        size_t dest_size = 2 * encrypted_size - 1;

        encrypted.resize(this->context(), context_data->parms_id(), dest_size);
        bool device = encrypted.on_device();

        utils::ntt_negacyclic_harvey_ps(
            encrypted.polys(0, encrypted_size), 
            encrypted_size, coeff_count, ntt_tables
        );
        Buffer<uint64_t> temp(dest_size, coeff_modulus_size, coeff_count, device);

        ConstSlice<uint64_t> eq0 = encrypted.const_poly(0);
        ConstSlice<uint64_t> eq1 = encrypted.const_poly(1);
        Slice<uint64_t> tq0 = temp.poly(0);
        Slice<uint64_t> tq1 = temp.poly(1);
        Slice<uint64_t> tq2 = temp.poly(2);
        
        utils::dyadic_product_p(eq0, eq0, coeff_count, coeff_modulus, tq0);
        // Compute 2*c0*c1
        utils::dyadic_product_p(eq0, eq1, coeff_count, coeff_modulus, tq1);
        utils::add_inplace_p(tq1, tq1.as_const(), coeff_count, coeff_modulus);
        // Compute c1^2
        utils::dyadic_product_p(eq1, eq1, coeff_count, coeff_modulus, tq2);

        encrypted.polys(0, dest_size).copy_from_slice(temp.const_reference());
        utils::inverse_ntt_negacyclic_harvey_ps(
            encrypted.polys(0, dest_size), 
            dest_size, coeff_count, ntt_tables
        );
        encrypted.correction_factor() = utils::multiply_uint64_mod(
            encrypted.correction_factor(),
            encrypted.correction_factor(),
            parms.plain_modulus_host()
        );
    }

    void Evaluator::square_inplace(Ciphertext& encrypted) const {
        check_no_seed("[Evaluator::square_inplace]", encrypted);
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: {
                this->bfv_square_inplace(encrypted);
                break;
            }
            case SchemeType::CKKS: {
                this->ckks_square_inplace(encrypted);
                break;
            }
            case SchemeType::BGV: {
                this->bgv_square_inplace(encrypted);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::square_inplace] Scheme not implemented.");
            }
        }
    }

    __global__ static void kernel_ski_util1(
        Slice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        ConstSlice<uint64_t> key_vector_j,
        size_t key_poly_coeff_size,
        ConstSlice<uint64_t> t_operand,
        size_t key_index,
        ConstPointer<Modulus> key_modulus
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * key_component_count) return;
        size_t i = global_index % coeff_count;
        size_t k = global_index / coeff_count;
        uint64_t qword[2] {0, 0}; Slice<uint64_t> qword_slice(qword, 2, true);
        utils::multiply_uint64_uint64(t_operand[i], key_vector_j[k * key_poly_coeff_size + key_index * coeff_count + i], qword_slice);
        size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
        Slice<uint64_t> accumulator_l = t_poly_lazy.slice(accumulator_l_offset, accumulator_l_offset + 2);
        utils::add_uint128_inplace(qword_slice, accumulator_l.as_const());
        accumulator_l[0] = key_modulus->reduce_uint128(qword_slice.as_const());
        accumulator_l[1] = 0;
    }

    static void ski_util1(
        Slice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        ConstSlice<uint64_t> key_vector_j,
        size_t key_poly_coeff_size,
        ConstSlice<uint64_t> t_operand,
        size_t key_index,
        ConstPointer<Modulus> key_modulus
    ) {
        bool device = t_poly_lazy.on_device();
        if (!device) {
            uint64_t qword[2] {0, 0}; Slice<uint64_t> qword_slice(qword, 2, false);
            for (size_t i = 0; i < coeff_count; i++) {
                for (size_t k = 0; k < key_component_count; k++) {
                    utils::multiply_uint64_uint64(t_operand[i], key_vector_j[k * key_poly_coeff_size + key_index * coeff_count + i], qword_slice);
                    size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
                    Slice<uint64_t> accumulator_l = t_poly_lazy.slice(accumulator_l_offset, accumulator_l_offset + 2);
                    utils::add_uint128_inplace(qword_slice, accumulator_l.as_const());
                    accumulator_l[0] = key_modulus->reduce_uint128(qword_slice.as_const());
                    accumulator_l[1] = 0;
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * key_component_count, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util1<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_poly_lazy, coeff_count, key_component_count, 
                key_vector_j, key_poly_coeff_size, t_operand, key_index, key_modulus
            );
        }
    }
    
    __global__ static void kernel_ski_util2(
        Slice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        ConstSlice<uint64_t> key_vector_j,
        size_t key_poly_coeff_size,
        ConstSlice<uint64_t> t_operand,
        size_t key_index,
        ConstPointer<Modulus> key_modulus
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * key_component_count) return;
        size_t i = global_index % coeff_count;
        size_t k = global_index / coeff_count;
        uint64_t qword[2] {0, 0}; Slice<uint64_t> qword_slice(qword, 2, true);
        utils::multiply_uint64_uint64(t_operand[i], key_vector_j[k * key_poly_coeff_size + key_index * coeff_count + i], qword_slice);
        size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
        Slice<uint64_t> accumulator_l = t_poly_lazy.slice(accumulator_l_offset, accumulator_l_offset + 2);
        utils::add_uint128_inplace(qword_slice, accumulator_l.as_const());
        accumulator_l[0] = qword_slice[0];
        accumulator_l[1] = qword_slice[1];
    }

    static void ski_util2(
        Slice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        ConstSlice<uint64_t> key_vector_j,
        size_t key_poly_coeff_size,
        ConstSlice<uint64_t> t_operand,
        size_t key_index,
        ConstPointer<Modulus> key_modulus
    ) {
        bool device = t_poly_lazy.on_device();
        if (!device) {
            uint64_t qword[2] {0, 0}; Slice<uint64_t> qword_slice(qword, 2, false);
            for (size_t i = 0; i < coeff_count; i++) {
                for (size_t k = 0; k < key_component_count; k++) {
                    utils::multiply_uint64_uint64(t_operand[i], key_vector_j[k * key_poly_coeff_size + key_index * coeff_count + i], qword_slice);
                    size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
                    Slice<uint64_t> accumulator_l = t_poly_lazy.slice(accumulator_l_offset, accumulator_l_offset + 2);
                    utils::add_uint128_inplace(qword_slice, accumulator_l.as_const());
                    accumulator_l[0] = qword_slice[0];
                    accumulator_l[1] = qword_slice[1];
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * key_component_count, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util2<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_poly_lazy, coeff_count, key_component_count, 
                key_vector_j, key_poly_coeff_size, t_operand, key_index, key_modulus
            );
        }
    }

    __global__ static void kernel_ski_util3(
        ConstSlice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        Slice<uint64_t> t_poly_prod_iter
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * key_component_count) return;
        size_t i = global_index % coeff_count;
        size_t k = global_index / coeff_count;
        size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
        t_poly_prod_iter[k * coeff_count * rns_modulus_size + i] = t_poly_lazy[accumulator_l_offset];
    }

    static void ski_util3(
        ConstSlice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        Slice<uint64_t> t_poly_prod_iter
    ) {
        bool device = t_poly_lazy.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_count; i++) {
                for (size_t k = 0; k < key_component_count; k++) {
                    size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
                    t_poly_prod_iter[k * coeff_count * rns_modulus_size + i] = t_poly_lazy[accumulator_l_offset];
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * key_component_count, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util3<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_poly_lazy, coeff_count, key_component_count, rns_modulus_size, t_poly_prod_iter
            );
        }
    }


    __global__ static void kernel_ski_util4(
        ConstSlice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        Slice<uint64_t> t_poly_prod_iter,
        ConstPointer<Modulus> key_modulus
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * key_component_count) return;
        size_t i = global_index % coeff_count;
        size_t k = global_index / coeff_count;
        size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
        t_poly_prod_iter[k * coeff_count * rns_modulus_size + i] = key_modulus->reduce_uint128(
            t_poly_lazy.const_slice(accumulator_l_offset, accumulator_l_offset + 2)
        );
    }

    static void ski_util4(
        ConstSlice<uint64_t> t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        Slice<uint64_t> t_poly_prod_iter,
        ConstPointer<Modulus> key_modulus
    ) {
        bool device = t_poly_lazy.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_count; i++) {
                for (size_t k = 0; k < key_component_count; k++) {
                    size_t accumulator_l_offset = k * coeff_count * 2 + 2 * i;
                    t_poly_prod_iter[k * coeff_count * rns_modulus_size + i] = key_modulus->reduce_uint128(
                        t_poly_lazy.const_slice(accumulator_l_offset, accumulator_l_offset + 2)
                    );
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * key_component_count, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util4<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_poly_lazy, coeff_count, key_component_count, 
                rns_modulus_size, t_poly_prod_iter, key_modulus
            );
        }
    }

    __global__ static void kernel_ski_util5(
        ConstSlice<uint64_t> t_last,
        Slice<uint64_t> t_poly_prod_i,
        size_t coeff_count,
        ConstPointer<Modulus> plain_modulus,
        ConstSlice<Modulus> key_modulus,
        size_t decomp_modulus_size,
        size_t rns_modulus_size,
        uint64_t qk_inv_qp,
        uint64_t qk,
        ConstSlice<MultiplyUint64Operand> modswitch_factors,
        Slice<uint64_t> encrypted_i
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * decomp_modulus_size) return;
        size_t i = global_index % coeff_count;
        size_t j = global_index / coeff_count;
        uint64_t k = utils::barrett_reduce_uint64(t_last[i], *plain_modulus);
        k = utils::negate_uint64_mod(k, *plain_modulus);
        if (qk_inv_qp != 1) 
            k = utils::multiply_uint64_mod(k, qk_inv_qp, *plain_modulus);
        uint64_t delta = 0; uint64_t c_mod_qi = 0;
        delta = utils::barrett_reduce_uint64(k, key_modulus[j]);
        delta = utils::multiply_uint64_mod(delta, qk, key_modulus[j]);
        c_mod_qi = utils::barrett_reduce_uint64(t_last[i], key_modulus[j]);
        const uint64_t Lqi = key_modulus[j].value() << 1;
        uint64_t& target = t_poly_prod_i[j * coeff_count + i];
        target = target + Lqi - (delta + c_mod_qi);
        target = utils::multiply_uint64operand_mod(target, modswitch_factors[j], key_modulus[j]);
        encrypted_i[j * coeff_count + i] = utils::add_uint64_mod(target, encrypted_i[j * coeff_count + i], key_modulus[j]);
    }

    static void ski_util5(
        ConstSlice<uint64_t> t_last,
        Slice<uint64_t> t_poly_prod_i,
        size_t coeff_count,
        ConstPointer<Modulus> plain_modulus,
        ConstSlice<Modulus> key_modulus,
        size_t decomp_modulus_size,
        size_t rns_modulus_size,
        uint64_t qk_inv_qp,
        uint64_t qk,
        ConstSlice<MultiplyUint64Operand> modswitch_factors,
        Slice<uint64_t> encrypted_i
    ) {
        bool device = t_last.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_count; i++) {
                uint64_t k = utils::barrett_reduce_uint64(t_last[i], *plain_modulus);
                k = utils::negate_uint64_mod(k, *plain_modulus);
                if (qk_inv_qp != 1) 
                    k = utils::multiply_uint64_mod(k, qk_inv_qp, *plain_modulus);
                uint64_t delta = 0; uint64_t c_mod_qi = 0;
                for (size_t j = 0; j < decomp_modulus_size; j++) {
                    delta = utils::barrett_reduce_uint64(k, key_modulus[j]);
                    delta = utils::multiply_uint64_mod(delta, qk, key_modulus[j]);
                    c_mod_qi = utils::barrett_reduce_uint64(t_last[i], key_modulus[j]);
                    const uint64_t Lqi = key_modulus[j].value() << 1;
                    uint64_t& target = t_poly_prod_i[j * coeff_count + i];
                    target = target + Lqi - (delta + c_mod_qi);
                    target = utils::multiply_uint64operand_mod(target, modswitch_factors[j], key_modulus[j]);
                    encrypted_i[j * coeff_count + i] = utils::add_uint64_mod(target, encrypted_i[j * coeff_count + i], key_modulus[j]);
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * decomp_modulus_size, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util5<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_last, t_poly_prod_i, coeff_count, plain_modulus, key_modulus, 
                decomp_modulus_size, rns_modulus_size, qk_inv_qp, qk, modswitch_factors, encrypted_i
            );
        }
    }

    __global__ static void kernel_ski_util6(
        Slice<uint64_t> t_last,
        size_t coeff_count,
        ConstPointer<Modulus> qk,
        ConstSlice<Modulus> key_modulus,
        size_t decomp_modulus_size,
        Slice<uint64_t> t_ntt
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * decomp_modulus_size) return;
        size_t i = global_index % coeff_count;
        size_t j = global_index / coeff_count;
        uint64_t qk_half = qk->value() >> 1;
        t_last[i] = utils::barrett_reduce_uint64(t_last[i] + qk_half, *qk);
        const Modulus& qi = key_modulus[j];
        if (qk->value() > qi.value()) {
            t_ntt[j * coeff_count + i] = utils::barrett_reduce_uint64(t_last[i], qi);
        } else {
            t_ntt[j * coeff_count + i] = t_last[i];
        }
        uint64_t fix = qi.value() - utils::barrett_reduce_uint64(qk_half, key_modulus[j]);
        t_ntt[j * coeff_count + i] += fix;
    }

    static void ski_util6(
        Slice<uint64_t> t_last,
        size_t coeff_count,
        ConstPointer<Modulus> qk,
        ConstSlice<Modulus> key_modulus,
        size_t decomp_modulus_size,
        Slice<uint64_t> t_ntt
    ) {
        bool device = t_last.on_device();
        if (!device) {
            uint64_t qk_half = qk->value() >> 1;
            for (size_t i = 0; i < coeff_count; i++) {
                t_last[i] = utils::barrett_reduce_uint64(t_last[i] + qk_half, *qk);
                for (size_t j = 0; j < decomp_modulus_size; j++) {
                    const Modulus& qi = key_modulus[j];
                    if (qk->value() > qi.value()) {
                        t_ntt[j * coeff_count + i] = utils::barrett_reduce_uint64(t_last[i], qi);
                    } else {
                        t_ntt[j * coeff_count + i] = t_last[i];
                    }
                    uint64_t fix = qi.value() - utils::barrett_reduce_uint64(qk_half, key_modulus[j]);
                    t_ntt[j * coeff_count + i] += fix;
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * decomp_modulus_size, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util6<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_last, coeff_count, qk, key_modulus, decomp_modulus_size, t_ntt
            );
        }
    }

    __global__ static void kernel_ski_util7(
        Slice<uint64_t> t_poly_prod_i,
        ConstSlice<uint64_t> t_ntt,
        size_t coeff_count, 
        Slice<uint64_t> encrypted_i,
        bool is_ckks,
        size_t decomp_modulus_size,
        ConstSlice<Modulus> key_modulus,
        ConstSlice<MultiplyUint64Operand> modswitch_factors
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= coeff_count * decomp_modulus_size) return;
        size_t i = global_index % coeff_count;
        size_t j = global_index / coeff_count;
        uint64_t& dest = t_poly_prod_i[j*coeff_count + i];
        uint64_t qi = key_modulus[j].value();
        dest += ((is_ckks) ? (qi << 2) : (qi << 1)) - t_ntt[j * coeff_count + i];
        dest = utils::multiply_uint64operand_mod(dest, modswitch_factors[j], key_modulus[j]);
        encrypted_i[j * coeff_count + i] = utils::add_uint64_mod(
            encrypted_i[j * coeff_count + i], dest, key_modulus[j]
        );
    }

    static void ski_util7(
        Slice<uint64_t> t_poly_prod_i,
        ConstSlice<uint64_t> t_ntt,
        size_t coeff_count, 
        Slice<uint64_t> encrypted_i,
        bool is_ckks,
        size_t decomp_modulus_size,
        ConstSlice<Modulus> key_modulus,
        ConstSlice<MultiplyUint64Operand> modswitch_factors
    ) {
        bool device = t_poly_prod_i.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_count; i++) {
                for (size_t j = 0; j < decomp_modulus_size; j++) {
                    uint64_t& dest = t_poly_prod_i[j*coeff_count + i];
                    uint64_t qi = key_modulus[j].value();
                    dest += ((is_ckks) ? (qi << 2) : (qi << 1)) - t_ntt[j * coeff_count + i];
                    dest = utils::multiply_uint64operand_mod(dest, modswitch_factors[j], key_modulus[j]);
                    encrypted_i[j * coeff_count + i] = utils::add_uint64_mod(
                        encrypted_i[j * coeff_count + i], dest, key_modulus[j]
                    );
                }
            }
        } else {
            size_t block_count = utils::ceil_div(coeff_count * decomp_modulus_size, utils::KERNEL_THREAD_COUNT);
            kernel_ski_util7<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                t_poly_prod_i, t_ntt, coeff_count, encrypted_i, is_ckks, 
                decomp_modulus_size, key_modulus, modswitch_factors
            );
        }
    }

    void Evaluator::switch_key_inplace_internal(Ciphertext& encrypted, utils::ConstSlice<uint64_t> target, const KSwitchKeys& kswitch_keys, size_t kswitch_keys_index) const {
        check_no_seed("[Evaluator::switch_key_inplace_internal]", encrypted);
        if (!this->context()->using_keyswitching()) {
            throw std::invalid_argument("[Evaluator::switch_key_inplace_internal] Keyswitching is not supported.");
        }
        if (kswitch_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::switch_key_inplace_internal] Keyswitching key has incorrect parms id.");
        }
        if (kswitch_keys_index >= kswitch_keys.data().size()) {
            throw std::out_of_range("[Evaluator::switch_key_inplace_internal] Key switch keys index out of range.");
        }

        ParmsID parms_id = encrypted.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::switch_key_inplace_internal]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        ContextDataPointer key_context_data = this->context()->key_context_data().value();
        const EncryptionParameters& key_parms = key_context_data->parms();
        SchemeType scheme = parms.scheme();
        bool is_ntt_form = encrypted.is_ntt_form();

        size_t coeff_count = parms.poly_modulus_degree();
        size_t decomp_modulus_size = parms.coeff_modulus().size();
        ConstSlice<Modulus> key_modulus = key_parms.coeff_modulus();
        Array<Modulus> key_modulus_host = Array<Modulus>::create_and_copy_from_slice(key_modulus);
        key_modulus_host.to_host_inplace();
        size_t key_modulus_size = key_modulus.size();
        size_t rns_modulus_size = decomp_modulus_size + 1;
        ConstSlice<NTTTables> key_ntt_tables = key_context_data->small_ntt_tables();
        ConstSlice<MultiplyUint64Operand> modswitch_factors = key_context_data->rns_tool().inv_q_last_mod_q();

        const std::vector<PublicKey>& key_vector = kswitch_keys.data()[kswitch_keys_index];
        size_t key_component_count = key_vector[0].as_ciphertext().polynomial_count();
        for (size_t i = 0; i < key_vector.size(); i++) {
            check_no_seed("[Evaluator::switch_key_inplace_internal]", key_vector[i].as_ciphertext());
        }

        if (target.size() != decomp_modulus_size * coeff_count) {
            throw std::invalid_argument("[Evaluator::switch_key_inplace_internal] Invalid target size.");
        }
        Array<uint64_t> target_copied = Array<uint64_t>::create_and_copy_from_slice(target);

        // If target is in NTT form; switch back to normal form
        if (is_ntt_form) {
            utils::inverse_ntt_negacyclic_harvey_p(
                target_copied.reference(), coeff_count, key_ntt_tables.const_slice(0, decomp_modulus_size)
            );
        }

        // Temporary result
        bool device = target.on_device();
        Array<uint64_t> poly_prod(key_component_count * coeff_count * rns_modulus_size, device);
        Array<uint64_t> poly_lazy(key_component_count * coeff_count * 2, device);
        Array<uint64_t> temp_ntt(coeff_count, device);

        for (size_t i = 0; i < rns_modulus_size; i++) {
            size_t key_index = (i == decomp_modulus_size ? key_modulus_size - 1 : i);

            // Product of two numbers is up to 60 + 60 = 120 bits, so we can sum up to 256 of them without reduction.
            size_t lazy_reduction_summand_bound = utils::HE_MULTIPLY_ACCUMULATE_USER_MOD_MAX;
            size_t lazy_reduction_counter = lazy_reduction_summand_bound;

            // Allocate memory for a lazy accumulator (128-bit coefficients)
            poly_lazy.set_zero();

            // Multiply with keys and perform lazy reduction on product's coefficients
            temp_ntt.set_zero();
            for (size_t j = 0; j < decomp_modulus_size; j++) {
                ConstSlice<uint64_t> temp_operand(nullptr, 0, device);
                if (is_ntt_form && (i == j)) {
                    temp_operand = target.const_slice(j * coeff_count, (j + 1) * coeff_count);
                } else {
                    if (key_modulus_host[j].value() <= key_modulus_host[key_index].value()) {
                        temp_ntt.copy_from_slice(target_copied.const_slice(j * coeff_count, (j + 1) * coeff_count));
                    } else {
                        utils::modulo(target_copied.const_slice(j * coeff_count, (j + 1) * coeff_count), key_modulus.at(key_index), temp_ntt.reference());
                    }
                    utils::ntt_negacyclic_harvey_lazy(temp_ntt.reference(), coeff_count, key_ntt_tables.at(key_index));
                    temp_operand = temp_ntt.const_reference();
                }
                
                // Multiply with keys and modular accumulate products in a lazy fashion
                size_t key_vector_poly_coeff_size = key_modulus_size * coeff_count;

                if (!lazy_reduction_counter) {
                    ski_util1(
                        poly_lazy.reference(), coeff_count, key_component_count,
                        key_vector[j].as_ciphertext().const_reference(),
                        key_vector_poly_coeff_size,
                        temp_operand, key_index, key_modulus.at(key_index)
                    );
                } else {
                    ski_util2(
                        poly_lazy.reference(), coeff_count, key_component_count,
                        key_vector[j].as_ciphertext().const_reference(),
                        key_vector_poly_coeff_size,
                        temp_operand, key_index, key_modulus.at(key_index)
                    );
                }

                lazy_reduction_counter -= 1;
                if (lazy_reduction_counter == 0) {
                    lazy_reduction_counter = lazy_reduction_summand_bound;
                }
            }
            
            Slice<uint64_t> t_poly_prod_iter = poly_prod.slice(i * coeff_count, poly_prod.size());

            if (lazy_reduction_counter == lazy_reduction_summand_bound) {
                ski_util3(
                    poly_lazy.const_reference(), coeff_count, key_component_count,
                    rns_modulus_size, t_poly_prod_iter
                );
            } else {
                ski_util4(
                    poly_lazy.const_reference(), coeff_count, key_component_count,
                    rns_modulus_size, t_poly_prod_iter,
                    key_modulus.at(key_index)
                );
            }
        } // i
        
        // Accumulated products are now stored in t_poly_prod

        temp_ntt = Array<uint64_t>(decomp_modulus_size * coeff_count, device);
        for (size_t i = 0; i < key_component_count; i++) {
            if (scheme == SchemeType::BGV) {
                // qk is the special prime
                uint64_t qk = key_modulus_host[key_modulus_size - 1].value();
                uint64_t qk_inv_qp = this->context()->key_context_data().value()->rns_tool().inv_q_last_mod_t();

                // Lazy reduction; this needs to be then reduced mod qi
                size_t t_last_offset = coeff_count * rns_modulus_size * i + decomp_modulus_size * coeff_count;
                Slice<uint64_t> t_last = poly_prod.slice(t_last_offset, t_last_offset + coeff_count);
                utils::inverse_ntt_negacyclic_harvey(t_last, coeff_count, key_ntt_tables.at(key_modulus_size - 1));
                utils::inverse_ntt_negacyclic_harvey_p(
                    poly_prod.slice(
                        i * coeff_count * rns_modulus_size, 
                        i * coeff_count * rns_modulus_size + decomp_modulus_size * coeff_count
                    ), 
                    coeff_count, 
                    key_ntt_tables.const_slice(0, decomp_modulus_size)
                );
                ConstPointer<Modulus> plain_modulus = parms.plain_modulus();

                ski_util5(
                    t_last.as_const(), poly_prod.slice(i * coeff_count * rns_modulus_size, poly_prod.size()),
                    coeff_count, plain_modulus, key_modulus,
                    decomp_modulus_size, rns_modulus_size, qk_inv_qp, qk,
                    modswitch_factors, encrypted.poly(i)
                );
            } else {
                // Lazy reduction; this needs to be then reduced mod qi
                size_t t_last_offset = coeff_count * rns_modulus_size * i + decomp_modulus_size * coeff_count;
                Slice<uint64_t> t_last = poly_prod.slice(t_last_offset, t_last_offset + coeff_count);
                temp_ntt.set_zero();
                utils::inverse_ntt_negacyclic_harvey(t_last, coeff_count, key_ntt_tables.at(key_modulus_size - 1));

                ski_util6(
                    t_last, coeff_count, key_modulus.at(key_modulus_size - 1),
                    key_modulus,
                    decomp_modulus_size,
                    temp_ntt.reference()
                );
            
                if (is_ntt_form) {
                    utils::ntt_negacyclic_harvey_lazy_p(temp_ntt.reference(), coeff_count, key_ntt_tables.const_slice(0, decomp_modulus_size));
                } else {
                    utils::inverse_ntt_negacyclic_harvey_p(
                        poly_prod.slice(
                            i * coeff_count * rns_modulus_size, 
                            i * coeff_count * rns_modulus_size + decomp_modulus_size * coeff_count
                        ), 
                        coeff_count, 
                        key_ntt_tables.const_slice(0, decomp_modulus_size)
                    );
                }

                ski_util7(
                    poly_prod.slice(i * coeff_count * rns_modulus_size, poly_prod.size()),
                    temp_ntt.const_reference(),
                    coeff_count, encrypted.poly(i),
                    scheme==SchemeType::CKKS, decomp_modulus_size, key_modulus,
                    modswitch_factors
                );
            }
            // printf("enc %ld: ", i); printDeviceArray(encrypted.data(i).get(), key_component_count * coeff_count);
        }
    }

    void Evaluator::apply_keyswitching_inplace(Ciphertext& encrypted, const KSwitchKeys& kswitch_keys) const {
        if (kswitch_keys.data().size() != 1) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Key switch keys size must be 1.");
        }
        if (encrypted.polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::apply_keyswitching_inplace] Ciphertext polynomial count must be 2.");
        }
        // due to the semantics of `switch_key_inplace_internal`, we should first get the c0 out
        // and then clear the original c0 in the encrypted.
        Array<uint64_t> target = Array<uint64_t>::create_and_copy_from_slice(encrypted.const_poly(1));
        encrypted.poly(1).set_zero();
        this->switch_key_inplace_internal(encrypted, target.const_reference(), kswitch_keys, 0);
    }

    void Evaluator::relinearize_inplace_internal(Ciphertext& encrypted, const RelinKeys& relin_keys, size_t destination_size) const {
        check_no_seed("[Evaluator::relinearize_inplace_internal]", encrypted);
        if (relin_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Relin keys has incorrect parms id.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::relinearize_inplace_internal]", encrypted.parms_id());
        size_t encrypted_size = encrypted.polynomial_count();
        if (encrypted_size < 2 || destination_size > encrypted_size) {
            throw std::invalid_argument("[Evaluator::relinearize_inplace_internal] Destination size must be at least 2 and less/equal to the size of the encrypted polynomial.");
        }
        if (destination_size == encrypted_size) {
            return;
        }
        size_t relins_needed = encrypted_size - destination_size;
        for (size_t i = 0; i < relins_needed; i++) {
            this->switch_key_inplace_internal(
                encrypted, encrypted.const_poly(encrypted_size - 1),
                relin_keys.as_kswitch_keys(), RelinKeys::get_index(encrypted_size - 1));
            encrypted_size -= 1;
        }
        encrypted.resize(this->context(), context_data->parms_id(), destination_size);
    }

    void Evaluator::mod_switch_scale_to_next_internal(const Ciphertext& encrypted, Ciphertext& destination) const {
        ParmsID parms_id = encrypted.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_scale_to_next_internal]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        SchemeType scheme = parms.scheme();
        switch (scheme) {
            case SchemeType::BFV: case SchemeType::BGV: {
                check_is_not_ntt_form("[Evaluator::mod_switch_scale_to_next_internal]", encrypted);
                break;
            }
            case SchemeType::CKKS: {
                check_is_ntt_form("[Evaluator::mod_switch_scale_to_next_internal]", encrypted);
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::mod_switch_scale_to_next_internal] Scheme not implemented.");
            }
        }
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_scale_to_next_internal] Next context data is not set.");
        }
        ContextDataPointer next_context_data = context_data->next_context_data().value();
        const EncryptionParameters& next_parms = next_context_data->parms();
        const RNSTool& rns_tool = context_data->rns_tool();
        
        size_t encrypted_size = encrypted.polynomial_count();
        size_t coeff_count = next_parms.poly_modulus_degree();
        size_t next_coeff_modulus_size = next_parms.coeff_modulus().size();

        Ciphertext encrypted_copy = encrypted.clone();
        switch (scheme) {
            case SchemeType::BFV: {
                for (size_t i = 0; i < encrypted_size; i++) {
                    rns_tool.divide_and_round_q_last_inplace(encrypted_copy.poly(i));
                }
                break;
            }
            case SchemeType::CKKS: {
                for (size_t i = 0; i < encrypted_size; i++) {
                    rns_tool.divide_and_round_q_last_ntt_inplace(encrypted_copy.poly(i), context_data->small_ntt_tables());
                }
                break;
            }
            case SchemeType::BGV: {
                for (size_t i = 0; i < encrypted_size; i++) {
                    rns_tool.mod_t_and_divide_q_last_inplace(encrypted_copy.poly(i));
                }
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::mod_switch_scale_to_next_internal] Scheme not implemented.");
            }
        }

        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace();
        else destination.to_host_inplace();

        destination.resize(this->context(), next_context_data->parms_id(), encrypted_size);
        for (size_t i = 0; i < encrypted_size; i++) {
            destination.poly(i).copy_from_slice(encrypted_copy.poly(i).const_slice(0, coeff_count * next_coeff_modulus_size));
        }

        destination.is_ntt_form() = encrypted.is_ntt_form();
        if (scheme == SchemeType::CKKS) {
            // take the last modulus
            size_t id = parms.coeff_modulus().size() - 1;
            Array<Modulus> modulus = Array<Modulus>::create_and_copy_from_slice(parms.coeff_modulus().const_slice(id, id+1));
            modulus.to_host_inplace();
            destination.scale() = encrypted.scale() / modulus[0].value();
        } else if (scheme == SchemeType::BGV) {
            destination.correction_factor() = utils::multiply_uint64_mod(
                encrypted.correction_factor(), rns_tool.inv_q_last_mod_t(), next_parms.plain_modulus_host()
            );
        }
    }

    void Evaluator::mod_switch_drop_to_next_internal(const Ciphertext& encrypted, Ciphertext& destination) const {
        ParmsID parms_id = encrypted.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_scale_to_next_internal]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        SchemeType scheme = parms.scheme();
        if (scheme == SchemeType::CKKS) {
            check_is_ntt_form("[Evaluator::mod_switch_drop_to_next_internal]", encrypted);
        }
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_internal] Next context data is not set.");
        }
        ContextDataPointer next_context_data = context_data->next_context_data().value();
        const EncryptionParameters& next_parms = next_context_data->parms();
        if (!is_scale_within_bounds(encrypted.scale(), next_context_data)) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_internal] Scale out of bounds.");
        }
        
        size_t encrypted_size = encrypted.polynomial_count();
        size_t coeff_count = next_parms.poly_modulus_degree();
        size_t next_coeff_modulus_size = next_parms.coeff_modulus().size();

        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace();
        else destination.to_host_inplace();

        destination.resize(this->context(), next_context_data->parms_id(), encrypted_size);
        for (size_t i = 0; i < encrypted_size; i++) {
            destination.poly(i).copy_from_slice(encrypted.poly(i).const_slice(0, coeff_count * next_coeff_modulus_size));
        }

        destination.is_ntt_form() = encrypted.is_ntt_form();
        destination.scale() = encrypted.scale();
        destination.correction_factor() = encrypted.correction_factor();
    }

    void Evaluator::mod_switch_drop_to_next_plain_inplace_internal(Plaintext& plain) const {
        if (!plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_plain_inplace_internal] Plaintext is not in NTT form.");
        }
        ParmsID parms_id = plain.parms_id();
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_drop_to_next_plain_inplace_internal]", parms_id);
        
        if (!context_data->next_context_data().has_value()) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_internal] Next context data is not set.");
        }
        ContextDataPointer next_context_data = context_data->next_context_data().value();

        const EncryptionParameters& next_parms = next_context_data->parms();
        if (!is_scale_within_bounds(plain.scale(), next_context_data)) {
            throw std::invalid_argument("[Evaluator::mod_switch_drop_to_next_internal] Scale out of bounds.");
        }

        size_t coeff_count = next_parms.poly_modulus_degree();
        size_t next_coeff_modulus_size = next_parms.coeff_modulus().size();
        size_t dest_size = coeff_count * next_coeff_modulus_size;
        plain.parms_id() = parms_id_zero;
        plain.resize(dest_size);
        plain.parms_id() = next_context_data->parms_id();
    }

    void Evaluator::mod_switch_to_next(const Ciphertext& encrypted, Ciphertext& destination) const {
        check_no_seed("[Evaluator::mod_switch_to_next]", encrypted);
        if (this->context()->last_parms_id() == encrypted.parms_id()) {
            throw std::invalid_argument("[Evaluator::mod_switch_to_next] End of modulus switching chain reached.");
        }
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: 
                this->mod_switch_scale_to_next_internal(encrypted, destination);
                break;
            case SchemeType::CKKS:
                this->mod_switch_drop_to_next_internal(encrypted, destination);
                break;
            case SchemeType::BGV:
                this->mod_switch_scale_to_next_internal(encrypted, destination);
                break;
            default:
                throw std::logic_error("[Evaluator::mod_switch_to_next] Scheme not implemented.");
        }
    }

    void Evaluator::mod_switch_to_inplace(Ciphertext& encrypted, const ParmsID& parms_id) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_to_inplace]", encrypted.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_to_inplace]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::mod_switch_to_inplace] Cannot switch to a higher level.");
        }
        while (encrypted.parms_id() != parms_id) {
            this->mod_switch_to_next_inplace(encrypted);
        }
    }

    void Evaluator::mod_switch_plain_to_inplace(Plaintext& plain, const ParmsID& parms_id) const {
        if (!plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::mod_switch_plain_to_inplace] Plaintext is not in NTT form.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::mod_switch_plain_to_inplace]", plain.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::mod_switch_plain_to_inplace]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::mod_switch_plain_to_inplace] Cannot switch to a higher level.");
        }
        while (plain.parms_id() != parms_id) {
            this->mod_switch_plain_to_next_inplace(plain);
        }
    }

    void Evaluator::rescale_to_next(const Ciphertext& encrypted, Ciphertext& destination) const {
        check_no_seed("[Evaluator::rescale_to_next]", encrypted);
        if (this->context()->last_parms_id() == encrypted.parms_id()) {
            throw std::invalid_argument("[Evaluator::rescale_to_next] End of modulus switching chain reached.");
        }
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: case SchemeType::BGV:
                throw std::invalid_argument("[Evaluator::rescale_to_next] Cannot rescale BFV/BGV ciphertext.");
                break;
            case SchemeType::CKKS:
                this->mod_switch_scale_to_next_internal(encrypted, destination);
                break;
            default:
                throw std::logic_error("[Evaluator::rescale_to_next] Scheme not implemented.");
        }
    }
    
    void Evaluator::rescale_to(const Ciphertext& encrypted, const ParmsID& parms_id, Ciphertext& destination) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::rescale_to]", encrypted.parms_id());
        ContextDataPointer target_context_data = this->get_context_data("[Evaluator::rescale_to]", parms_id);
        if (context_data->chain_index() < target_context_data->chain_index()) {
            throw std::invalid_argument("[Evaluator::rescale_to] Cannot rescale to a higher level.");
        }
        while (encrypted.parms_id() != parms_id) {
            this->rescale_to_next(encrypted, destination);
        }
    }

    void Evaluator::translate_plain_inplace(Ciphertext& encrypted, const Plaintext& plain, bool subtract) const {
        check_no_seed("[Evaluator::translate_plain_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::translate_plain_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        SchemeType scheme = parms.scheme();
        switch (scheme) {
            case SchemeType::BFV: case SchemeType::BGV: {
                check_is_not_ntt_form("[Evaluator::translate_plain_inplace]", encrypted);
                break;
            }
            case SchemeType::CKKS: {
                check_is_ntt_form("[Evaluator::translate_plain_inplace]", encrypted);
                if (!utils::are_close_double(plain.scale(), encrypted.scale())) {
                    throw std::invalid_argument("[Evaluator::translate_plain_inplace] Plaintext scale is not equal to the scale of the ciphertext.");
                }
                break;
            }
            default: {
                throw std::logic_error("[Evaluator::translate_plain_inplace] Scheme not implemented.");
            }
        }
        if (encrypted.is_ntt_form() != plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::translate_plain_inplace] Plaintext and ciphertext are not in the same NTT form.");
        }
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        switch (scheme) {
            case SchemeType::BFV: {
                if (!subtract) {
                    scaling_variant::multiply_add_plain(plain, context_data, encrypted.poly(0));
                } else {
                    scaling_variant::multiply_sub_plain(plain, context_data, encrypted.poly(0));
                }
                break;
            }
            case SchemeType::CKKS: {
                if (!subtract) {
                    utils::add_inplace_p(encrypted.poly(0), plain.poly(), coeff_count, coeff_modulus);
                } else {
                    utils::sub_inplace_p(encrypted.poly(0), plain.poly(), coeff_count, coeff_modulus);
                }
                break;
            }
            case SchemeType::BGV: {
                Plaintext plain_copy = plain;
                utils::multiply_scalar(plain.poly(), encrypted.correction_factor(), parms.plain_modulus(), plain_copy.poly());
                if (!subtract) {
                    scaling_variant::add_plain(plain_copy, context_data, encrypted.poly(0));
                } else {
                    scaling_variant::sub_plain(plain_copy, context_data, encrypted.poly(0));
                }
                break;
            }
            default: 
                throw std::logic_error("[Evaluator::translate_plain_inplace] Scheme not implemented.");
        }
    }

    __global__ static void kernel_multiply_plain_normal_no_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_modulus_size,
        ConstSlice<uint64_t> plain, 
        Slice<uint64_t> temp, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= plain_coeff_count) return;
        size_t plain_value = plain[i];
        if (plain_value >= plain_upper_half_threshold) {
            utils::add_uint_uint64(plain_upper_half_increment, plain_value, temp.slice(i * coeff_modulus_size, (i + 1) * coeff_modulus_size));
        } else {
            temp[coeff_modulus_size * i] = plain_value;
        }
    }

    static void multiply_plain_normal_no_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_modulus_size,
        ConstSlice<uint64_t> plain, 
        Slice<uint64_t> temp, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        bool device = temp.on_device();
        if (!device) {
            for (size_t i = 0; i < plain_coeff_count; i++) {
                size_t plain_value = plain[i];
                if (plain_value >= plain_upper_half_threshold) {
                    utils::add_uint_uint64(plain_upper_half_increment, plain_value, temp.slice(i * coeff_modulus_size, (i + 1) * coeff_modulus_size));
                } else {
                    temp[coeff_modulus_size * i] = plain_value;
                }
            } 
        } else {
            size_t block_count = utils::ceil_div(plain_coeff_count, utils::KERNEL_THREAD_COUNT);
            kernel_multiply_plain_normal_no_fast_plain_lift<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                plain_coeff_count, coeff_modulus_size,
                plain, temp, plain_upper_half_threshold, plain_upper_half_increment
            );
        }
    }

    __global__ static void kernel_multiply_plain_normal_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_count, size_t coeff_modulus_size,
        ConstSlice<uint64_t> plain, 
        Slice<uint64_t> temp, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= plain_coeff_count * coeff_modulus_size) return;
        size_t i = global_index / plain_coeff_count;
        size_t j = global_index % plain_coeff_count;
        temp[i * coeff_count + j] = (plain[j] >= plain_upper_half_threshold)
            ? plain[j] + plain_upper_half_increment[i]
            : plain[j];
    }

    static void multiply_plain_normal_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_count, size_t coeff_modulus_size,
        ConstSlice<uint64_t> plain, 
        Slice<uint64_t> temp, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        bool device = temp.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_modulus_size; i++) {
                for (size_t j = 0; j < plain_coeff_count; j++) {
                    temp[i * coeff_count + j] = (plain[j] >= plain_upper_half_threshold)
                        ? plain[j] + plain_upper_half_increment[i]
                        : plain[j];
                }
            }
        } else {
            size_t total = plain_coeff_count * coeff_modulus_size;
            size_t block_count = utils::ceil_div(total, utils::KERNEL_THREAD_COUNT);
            kernel_multiply_plain_normal_fast_plain_lift<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                plain_coeff_count, coeff_count, coeff_modulus_size,
                plain, temp, plain_upper_half_threshold, plain_upper_half_increment
            );
        }
    }

    void Evaluator::multiply_plain_normal_inplace(Ciphertext& encrypted, const Plaintext& plain) const {
        check_no_seed("[Evaluator::multiply_plain_normal_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::multiply_plain_normal_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_count = parms.poly_modulus_degree();
        size_t coeff_modulus_size = coeff_modulus.size();

        size_t plain_upper_half_threshold = context_data->plain_upper_half_threshold();
        ConstSlice<uint64_t> plain_upper_half_increment = context_data->plain_upper_half_increment();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();

        size_t encrypted_size = encrypted.polynomial_count();
        size_t plain_coeff_count = plain.coeff_count();

        // Note: the original implementation has an optimization
        // for plaintexts with only one term.
        // But we are reluctant to detect the number of non-zero terms
        // in the plaintext, so we just use the general implementation.
        
        // Generic case: any plaintext polynomial
        // Allocate temporary space for an entire RNS polynomial
        bool device = encrypted.on_device();
        Buffer<uint64_t> temp(coeff_modulus_size, coeff_count, device);
        if (!context_data->qualifiers().using_fast_plain_lift) {
            multiply_plain_normal_no_fast_plain_lift(
                plain_coeff_count, coeff_modulus_size,
                plain.poly(), temp.reference(), plain_upper_half_threshold, plain_upper_half_increment
            );
            context_data->rns_tool().base_q().decompose_array(temp.reference());
        } else {
            // Note that in this case plain_upper_half_increment holds its value in RNS form modulo the coeff_modulus
            // primes.
            multiply_plain_normal_fast_plain_lift(
                plain_coeff_count, coeff_count, coeff_modulus_size,
                plain.poly(), temp.reference(), plain_upper_half_threshold, plain_upper_half_increment
            );
        }

        // Need to multiply each component in encrypted with temp; first step is to transform to NTT form
        // RNSIter temp_iter(temp.get(), coeff_count);
        utils::ntt_negacyclic_harvey_p(temp.reference(), coeff_count, ntt_tables);
        utils::ntt_negacyclic_harvey_lazy_ps(encrypted.polys(0, encrypted_size), encrypted_size, coeff_count, ntt_tables);
        for (size_t i = 0; i < encrypted_size; i++) {
            utils::dyadic_product_inplace_p(encrypted.poly(i), temp.const_reference(), coeff_count, coeff_modulus);
        }
        utils::inverse_ntt_negacyclic_harvey_ps(encrypted.polys(0, encrypted_size), encrypted_size, coeff_count, ntt_tables);

        if (parms.scheme() == SchemeType::CKKS) {
            encrypted.scale() = encrypted.scale() * plain.scale();
            if (!is_scale_within_bounds(encrypted.scale(), context_data)) {
                throw std::invalid_argument("[Evaluator::multiply_plain_normal_inplace] Scale out of bounds.");
            }
        }
    }

    void Evaluator::multiply_plain_ntt_inplace(Ciphertext& encrypted, const Plaintext& plain) const {
        check_no_seed("[Evaluator::multiply_plain_ntt_inplace]", encrypted);
        if (encrypted.parms_id() != plain.parms_id()) {
            throw std::invalid_argument("[Evaluator::multiply_plain_ntt_inplace] Plaintext and ciphertext parameters do not match.");
        }

        ContextDataPointer context_data = this->get_context_data("[Evaluator::multiply_plain_ntt_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_count = parms.poly_modulus_degree();
        size_t encrypted_size = encrypted.polynomial_count();

        for (size_t i = 0; i < encrypted_size; i++) {
            utils::dyadic_product_inplace_p(encrypted.poly(i), plain.poly(), coeff_count, coeff_modulus);
        }

        if (parms.scheme() == SchemeType::CKKS) {
            encrypted.scale() = encrypted.scale() * plain.scale();
            if (!is_scale_within_bounds(encrypted.scale(), context_data)) {
                throw std::invalid_argument("[Evaluator::multiply_plain_normal_inplace] Scale out of bounds.");
            }
        }
    }

    void Evaluator::multiply_plain_inplace(Ciphertext& encrypted, const Plaintext& plain) const {
        if (encrypted.is_ntt_form() != plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::multiply_plain_inplace] Plaintext and ciphertext are not in the same NTT form.");
        }
        if (encrypted.is_ntt_form()) {
            this->multiply_plain_ntt_inplace(encrypted, plain);
        } else {
            this->multiply_plain_normal_inplace(encrypted, plain);
        }
    }

    static void transform_plain_to_ntt_no_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_modulus_size,
        ConstSlice<uint64_t> plain, 
        Slice<uint64_t> temp, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        multiply_plain_normal_no_fast_plain_lift(
            plain_coeff_count, coeff_modulus_size,
            plain, temp, plain_upper_half_threshold, plain_upper_half_increment
        );
    }

    __global__ static void kernel_transform_plain_to_ntt_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_count, size_t coeff_modulus_size,
        Slice<uint64_t> plain, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        size_t global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index >= plain_coeff_count * (coeff_modulus_size - 1)) return;
        size_t i = (global_index / plain_coeff_count) + 1;
        size_t j = global_index % plain_coeff_count;
        size_t plain_index = i * coeff_count + j;
        plain[plain_index] = (plain[j] >= plain_upper_half_threshold)
            ? plain[j] + plain_upper_half_increment[i]
            : plain[j];
        // sync
        __syncthreads();
        if (i == 1) {
            plain[j] = (plain[j] >= plain_upper_half_threshold)
                ? plain[j] + plain_upper_half_increment[0]
                : plain[j];
        }
    }

    static void transform_plain_to_ntt_fast_plain_lift(
        size_t plain_coeff_count, size_t coeff_count, size_t coeff_modulus_size,
        Slice<uint64_t> plain, 
        uint64_t plain_upper_half_threshold,
        ConstSlice<uint64_t> plain_upper_half_increment
    ) {
        bool device = plain.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_modulus_size; i++) {
                for (size_t j = 0; j < plain_coeff_count; j++) {
                    size_t plain_index = (coeff_modulus_size - 1 - i) * coeff_count + j;
                    size_t increment_index = coeff_modulus_size - 1 - i;
                    plain[plain_index] = (plain[j] >= plain_upper_half_threshold)
                        ? plain[j] + plain_upper_half_increment[increment_index]
                        : plain[j];
                }
            }
        } else {
            size_t total = plain_coeff_count * coeff_modulus_size;
            size_t block_count = utils::ceil_div(total, utils::KERNEL_THREAD_COUNT);
            kernel_transform_plain_to_ntt_fast_plain_lift<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                plain_coeff_count, coeff_count, coeff_modulus_size,
                plain, plain_upper_half_threshold, plain_upper_half_increment
            );
        }
    }

    void Evaluator::transform_plain_to_ntt_inplace(Plaintext& plain, const ParmsID& parms_id) const {
        if (plain.is_ntt_form()) {
            throw std::invalid_argument("[Evaluator::transform_plain_to_ntt_inplace] Plaintext is already in NTT form.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::transform_plain_to_ntt_inplace]", parms_id);
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t plain_coeff_count = plain.coeff_count();

        plain.resize(coeff_count * coeff_modulus_size);

        size_t plain_upper_half_threshold = context_data->plain_upper_half_threshold();
        ConstSlice<uint64_t> plain_upper_half_increment = context_data->plain_upper_half_increment();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();

        if (!context_data->qualifiers().using_fast_plain_lift) {
            bool device = plain.on_device();
            Buffer<uint64_t> temp(coeff_modulus_size, coeff_count, device);
            transform_plain_to_ntt_no_fast_plain_lift(
                plain_coeff_count, coeff_modulus_size,
                plain.const_poly(), temp.reference(), plain_upper_half_threshold, plain_upper_half_increment
            );
            context_data->rns_tool().base_q().decompose_array(temp.reference());
            plain.poly().copy_from_slice(temp.const_reference());
        } else {
            // Note that in this case plain_upper_half_increment holds its value in RNS form modulo the coeff_modulus
            // primes.
            transform_plain_to_ntt_fast_plain_lift(
                plain_coeff_count, coeff_count, coeff_modulus_size,
                plain.poly(), plain_upper_half_threshold, plain_upper_half_increment
            );
        }

        utils::ntt_negacyclic_harvey_p(plain.poly(), coeff_count, ntt_tables);
        plain.parms_id() = parms_id;
    }

    void Evaluator::transform_to_ntt_inplace(Ciphertext& encrypted) const {
        check_no_seed("[Evaluator::transform_to_ntt_inplace]", encrypted);
        check_is_not_ntt_form("[Evaluator::transform_to_ntt_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::transform_to_ntt_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        utils::ntt_negacyclic_harvey_ps(
            encrypted.polys(0, encrypted.polynomial_count()), 
            encrypted.polynomial_count(), 
            coeff_count, ntt_tables
        );
        encrypted.is_ntt_form() = true;
    }

    void Evaluator::transform_from_ntt_inplace(Ciphertext& encrypted) const {
        check_no_seed("[Evaluator::transform_to_ntt_inplace]", encrypted);
        check_is_ntt_form("[Evaluator::transform_to_ntt_inplace]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::transform_to_ntt_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        utils::inverse_ntt_negacyclic_harvey_ps(
            encrypted.polys(0, encrypted.polynomial_count()), 
            encrypted.polynomial_count(), 
            coeff_count, ntt_tables
        );
        encrypted.is_ntt_form() = false;
    }
    
    void Evaluator::apply_galois_inplace(Ciphertext& encrypted, size_t galois_element, const GaloisKeys& galois_keys) const {
        check_no_seed("[Evaluator::apply_galois_inplace]", encrypted);
        if (galois_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois keys has incorrect parms id.");
        }
        ContextDataPointer context_data = this->get_context_data("[Evaluator::apply_galois_inplace]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.polynomial_count();
        ContextDataPointer key_context_data = this->context()->key_context_data().value();
        const GaloisTool& galois_tool = key_context_data->galois_tool();

        if (!galois_keys.has_key(galois_element)) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois key not present.");
        }
        size_t m = coeff_count * 2;
        if (galois_element & 1 == 0 || galois_element > m) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois element is not valid.");
        }
        if (encrypted_size > 2) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Ciphertext size must be 2.");
        }

        Array<uint64_t> temp(coeff_count * coeff_modulus_size, encrypted.on_device());
        // DO NOT CHANGE EXECUTION ORDER OF FOLLOWING SECTION
        // BEGIN: Apply Galois for each ciphertext
        // Execution order is sensitive, since apply_galois is not inplace!
        if (!encrypted.is_ntt_form()) {
            galois_tool.apply_p(encrypted.const_poly(0), galois_element, coeff_modulus, temp.reference());
            encrypted.poly(0).copy_from_slice(temp.const_reference());
            galois_tool.apply_p(encrypted.const_poly(1), galois_element, coeff_modulus, temp.reference());
        } else {
            galois_tool.apply_ntt_p(encrypted.const_poly(0), coeff_modulus_size, galois_element, temp.reference());
            encrypted.poly(0).copy_from_slice(temp.const_reference());
            galois_tool.apply_ntt_p(encrypted.const_poly(1), coeff_modulus_size, galois_element, temp.reference());
        }
        encrypted.poly(1).set_zero();

        this->switch_key_inplace_internal(encrypted, temp.const_reference(), galois_keys.as_kswitch_keys(), GaloisKeys::get_index(galois_element));
    }
    
    void Evaluator::apply_galois_plain_inplace(Plaintext& plain, size_t galois_element) const {
        ContextDataPointer context_data = plain.is_ntt_form()
            ? this->get_context_data("[Evaluator::apply_galois_plain_inplace]", plain.parms_id())
            : this->context()->key_context_data().value();
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        ContextDataPointer key_context_data = this->context()->key_context_data().value();
        const GaloisTool& galois_tool = key_context_data->galois_tool();
        
        size_t m = coeff_count * 2;
        if (galois_element & 1 == 0 || galois_element > m) {
            throw std::invalid_argument("[Evaluator::apply_galois_inplace] Galois element is not valid.");
        }

        Array<uint64_t> temp(coeff_count * (plain.is_ntt_form() ? coeff_modulus_size : 1), plain.on_device());
        if (!plain.is_ntt_form()) {
            if (context_data->is_ckks()) {
                galois_tool.apply_p(plain.const_poly(), galois_element, coeff_modulus, temp.reference());
            } else {
                galois_tool.apply(plain.const_poly(), galois_element, context_data->parms().plain_modulus(), temp.reference());
            }
        } else {
            galois_tool.apply_ntt_p(plain.const_poly(), coeff_modulus_size, galois_element, temp.reference());
        }

        ParmsID parms_id = plain.parms_id();
        plain.parms_id() = parms_id_zero;
        plain.resize(temp.size());
        plain.data().copy_from_slice(temp.const_reference());
        plain.parms_id() = parms_id;
    }

    void Evaluator::rotate_inplace_internal(Ciphertext& encrypted, int steps, const GaloisKeys& galois_keys) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::rotate_inplace_internal]", encrypted.parms_id());
        if (!context_data->qualifiers().using_batching) {
            throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Batching must be enabled to use rotate.");
        }
        if (galois_keys.parms_id() != this->context()->key_parms_id()) {
            throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Galois keys has incorrect parms id.");
        }
        if (steps == 0) return;
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        const GaloisTool& galois_tool = context_data->galois_tool();
        if (galois_keys.has_key(galois_tool.get_element_from_step(steps))) {
            size_t element = galois_tool.get_element_from_step(steps);
            this->apply_galois_inplace(encrypted, element, galois_keys);
        } else {
            // Convert the steps to NAF: guarantees using smallest HW
            std::vector<int> naf_steps = utils::naf(steps);
            if (naf_steps.size() == 1) {
                throw std::invalid_argument("[Evaluator::rotate_inplace_internal] Galois key not present.");
            }
            for (int naf_step : naf_steps) {
                this->rotate_inplace_internal(encrypted, naf_step, galois_keys);
            }
        }
    }
    
    void Evaluator::conjugate_inplace_internal(Ciphertext& encrypted, const GaloisKeys& galois_keys) const {
        ContextDataPointer context_data = this->get_context_data("Evaluator::conjugate_inplace_internal", encrypted.parms_id());
        if (!context_data->qualifiers().using_batching) {
            throw std::logic_error("[Evaluator::conjugate_inplace_internal] Batching is not enabled.");
        }
        const GaloisTool& galois_tool = context_data->galois_tool();
        this->apply_galois_inplace(encrypted, galois_tool.get_element_from_step(0), galois_keys);
    }

    void Evaluator::negacyclic_shift(const Ciphertext& encrypted, size_t shift, Ciphertext& destination) const {
        check_no_seed("[Evaluator::negacyclic_shift]", encrypted);
        ContextDataPointer context_data = this->get_context_data("[Evaluator::negacyclic_shift]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();

        destination = encrypted.clone();
        utils::negacyclic_shift_ps(
            encrypted.polys(0, encrypted.polynomial_count()),
            shift, encrypted.polynomial_count(), coeff_count, coeff_modulus, 
            destination.polys(0, destination.polynomial_count())
        );
    }

    __global__ static void kernel_extract_lwe_gather_c0(
        size_t coeff_modulus_size, size_t coeff_count, size_t term,
        ConstSlice<uint64_t> rlwe_c0, Slice<uint64_t> c0
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= coeff_modulus_size) return;
        c0[i] = rlwe_c0[coeff_count * i + term];
    }

    static void extract_lwe_gather_c0(
        size_t coeff_modulus_size, size_t coeff_count, size_t term,
        ConstSlice<uint64_t> rlwe_c0, Slice<uint64_t> c0
    ) {
        bool device = rlwe_c0.on_device();
        if (!device) {
            for (size_t i = 0; i < coeff_modulus_size; i++) {
                c0[i] = rlwe_c0[coeff_count * i + term];
            }
        } else {
            if (coeff_modulus_size >= utils::KERNEL_THREAD_COUNT) {
                size_t block_count = utils::ceil_div(coeff_modulus_size, utils::KERNEL_THREAD_COUNT);
                kernel_extract_lwe_gather_c0<<<block_count, utils::KERNEL_THREAD_COUNT>>>(
                    coeff_modulus_size, coeff_count, term, rlwe_c0, c0
                );
            } else {
                kernel_extract_lwe_gather_c0<<<1, coeff_modulus_size>>>(
                    coeff_modulus_size, coeff_count, term, rlwe_c0, c0
                );
            }
        }
    }
    
    LWECiphertext Evaluator::extract_lwe_new(const Ciphertext& encrypted, size_t term) const {
        check_no_seed("[Evaluator::extract_lwe_new]", encrypted);
        if (encrypted.polynomial_count() != 2) {
            throw std::invalid_argument("[Evaluator::extract_lwe_new] Ciphertext size must be 2.");
        }
        if (encrypted.is_ntt_form()) {
            Ciphertext transformed;
            this->transform_from_ntt(encrypted, transformed);
            return this->extract_lwe_new(transformed, term);
        }
        // else
        ContextDataPointer context_data = this->get_context_data("[Evaluator::extract_lwe_new]", encrypted.parms_id());
        const EncryptionParameters& parms = context_data->parms();
        size_t coeff_count = parms.poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = parms.coeff_modulus().size();

        // gather c1
        size_t shift = (term == 0) ? 0 : (coeff_count * 2 - term);
        bool device = encrypted.on_device();
        utils::DynamicArray<uint64_t> c1(coeff_count * coeff_modulus_size, device);
        utils::negacyclic_shift_p(
            encrypted.const_poly(1), shift, coeff_count, coeff_modulus, c1.reference()
        );

        // gather c0
        utils::DynamicArray<uint64_t> c0(coeff_modulus_size, device);
        extract_lwe_gather_c0(
            coeff_modulus_size, coeff_count, term,
            encrypted.const_poly(0), c0.reference()
        );

        // set lwe
        LWECiphertext ret;
        ret.coeff_modulus_size() = coeff_modulus_size;
        ret.poly_modulus_degree() = coeff_count;
        ret.c0_dyn() = std::move(c0);
        ret.c1_dyn() = std::move(c1);
        ret.parms_id() = encrypted.parms_id();
        ret.scale() = encrypted.scale();
        ret.correction_factor() = encrypted.correction_factor();
        return ret;
    }

    
    void Evaluator::field_trace_inplace(Ciphertext& encrypted, const GaloisKeys& automorphism_keys, size_t logn) const {
        size_t poly_degree = encrypted.poly_modulus_degree();
        Ciphertext temp;
        while (poly_degree > (1 << logn)) {
            size_t galois_element = poly_degree + 1;
            this->apply_galois(encrypted, galois_element, automorphism_keys, temp);
            this->add_inplace(encrypted, temp);
            poly_degree >>= 1;
        }
    }
    
    void Evaluator::divide_by_poly_modulus_degree_inplace(Ciphertext& encrypted, uint64_t mul) const {
        ContextDataPointer context_data = this->get_context_data("[Evaluator::divide_by_poly_modulus_degree_inplace]", encrypted.parms_id());
        size_t size = encrypted.polynomial_count();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        ConstSlice<Modulus> coeff_modulus = context_data->parms().coeff_modulus();
        size_t n = context_data->parms().poly_modulus_degree();
        size_t logn = static_cast<size_t>(utils::get_power_of_two(n));
        utils::ntt_multiply_inv_degree(
            encrypted.polys(0, size), size, logn, ntt_tables
        );
        if (mul != 1) {
            utils::multiply_scalar_ps(encrypted.const_polys(0, size), mul, size, n, coeff_modulus, encrypted.polys(0, size));
        }
    }
    
    Ciphertext Evaluator::pack_lwe_ciphertexts_new(const std::vector<LWECiphertext>& lwes, const GaloisKeys& automorphism_keys) const {
        size_t lwes_count = lwes.size();
        if (lwes_count == 0) {
            throw std::invalid_argument("[Evaluator::pack_lwe_ciphertexts_new] LWE ciphertexts must not be empty.");
        }
        ParmsID lwe_parms_id = lwes[0].parms_id();
        // check all have same parms id
        for (size_t i = 1; i < lwes_count; i++) {
            if (lwes[i].parms_id() != lwe_parms_id) {
                throw std::invalid_argument("[Evaluator::pack_lwe_ciphertexts_new] LWE ciphertexts must have same parms id.");
            }
        }

        ContextDataPointer context_data = this->get_context_data("[Evaluator::pack_lwe_ciphertexts_new]", lwe_parms_id);
        SchemeType scheme = context_data->parms().scheme();
        if (scheme == SchemeType::CKKS) {
            // all should have same scale
            double scale = lwes[0].scale();
            for (size_t i = 1; i < lwes_count; i++) {
                if (!utils::are_close_double(lwes[i].scale(), scale)) {
                    throw std::invalid_argument("[Evaluator::pack_lwe_ciphertexts_new] LWE ciphertexts must have same scale.");
                }
            }
        }
        if (scheme == SchemeType::BGV) {
            // all should have same correction factor
            uint64_t cf = lwes[0].correction_factor();
            for (size_t i = 1; i < lwes_count; i++) {
                if (lwes[i].correction_factor() != cf) {
                    throw std::invalid_argument("[Evaluator::pack_lwe_ciphertexts_new] LWE ciphertexts must have same correction factor.");
                }
            }
        }
        size_t poly_modulus_degree = context_data->parms().poly_modulus_degree();
        ConstSlice<Modulus> coeff_modulus = context_data->parms().coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        if (lwes_count > poly_modulus_degree) {
            throw std::invalid_argument("[Evaluator::pack_lwe_ciphertexts_new] LWE ciphertexts count must be less than poly_modulus_degree.");
        }
        size_t l = 0;
        while ((1 << l) < lwes_count) l += 1;
        std::vector<Ciphertext> rlwes(1 << l);
        Ciphertext zero_rlwe = this->assemble_lwe_new(lwes[0]);
        zero_rlwe.data().reference().set_zero();
        for (size_t i = 0; i < (1<<l); i++) {
            size_t index = static_cast<size_t>(utils::reverse_bits_uint64(static_cast<uint64_t>(i), l));
            if (index < lwes_count) {
                rlwes[i] = this->assemble_lwe_new(lwes[index]);
                this->divide_by_poly_modulus_degree_inplace(rlwes[i]);
            } else {
                rlwes[i] = zero_rlwe;
            }
        }
        Ciphertext temp(std::move(zero_rlwe));
        for (size_t layer = 0; layer < l; layer++) {
            size_t gap = 1 << layer;
            size_t offset = 0;
            size_t shift = poly_modulus_degree >> (layer + 1);
            while (offset < (1 << l)) {
                Ciphertext& even = rlwes[offset];
                Ciphertext& odd = rlwes[offset + gap];
                utils::negacyclic_shift_ps(
                    odd.const_reference(), shift, odd.polynomial_count(), 
                    poly_modulus_degree, coeff_modulus, temp.reference()
                );
                this->sub(even, temp, odd);
                this->add_inplace(even, temp);
                if (scheme == SchemeType::CKKS) {
                    this->transform_to_ntt_inplace(odd);
                }
                this->apply_galois_inplace(odd, (1 << (layer + 1)) + 1, automorphism_keys);
                if (scheme == SchemeType::CKKS) {
                    this->transform_from_ntt_inplace(odd);
                }
                this->add_inplace(even, odd);
                offset += (gap << 1);
            }
        }
        // take the first element
        Ciphertext ret = std::move(rlwes[0]);
        if (scheme == SchemeType::CKKS) {
            this->transform_to_ntt_inplace(ret);
        }
        field_trace_inplace(ret, automorphism_keys, l);
        return ret;
    }
}