#include "hip/hip_runtime.h"
#include "batch_encoder.cuh"
#include "encryption_parameters.cuh"
#include "utils/scaling_variant.cuh"

namespace troy {

    using utils::Array;
    using utils::Slice; 
    using utils::ConstSlice;

    BatchEncoder::BatchEncoder(HeContextPointer context) {
        if (context->on_device()) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Cannot create from device context.");
        }
        if (!context->parameters_set()) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Encryption parameters are not set correctly.");
        }
        
        ContextDataPointer context_data = context->first_context_data().value();
        const EncryptionParameters& parms = context_data->parms();

        if (parms.scheme() != SchemeType::BFV && parms.scheme() != SchemeType::BGV) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Unsupported scheme.");
        }

        size_t slots = parms.poly_modulus_degree();
        Array<uint64_t> roots_of_unity;
        Array<size_t> matrix_reps_index_map;

        if (context_data->qualifiers().using_batching) {
            roots_of_unity = Array<uint64_t>(slots, false, nullptr);
            const Modulus& modulus = *parms.plain_modulus();
            uint64_t root = context_data->plain_ntt_tables()->root();
            uint64_t generator_sq = utils::multiply_uint64_mod(root, root, modulus);
            roots_of_unity[0] = root;
            for (size_t i = 1; i < slots; i++) {
                roots_of_unity[i] = utils::multiply_uint64_mod(roots_of_unity[i - 1], generator_sq, modulus);
            }
            int logn_int = utils::get_power_of_two(static_cast<uint64_t>(slots));
            if (logn_int < 0) {
                throw std::invalid_argument("[BatchEncoder::BatchEncoder] Slots must be a power of two.");
            }
            size_t logn = static_cast<size_t>(logn_int);
            matrix_reps_index_map = Array<size_t>(slots, false, nullptr);
            size_t row_size = slots >> 1;
            size_t m = slots << 1;
            size_t gen = utils::GALOIS_GENERATOR; size_t pos = 1;
            for (size_t i = 0; i < row_size; i++) {
                size_t index1 = (pos - 1) >> 1;
                size_t index2 = (m - pos - 1) >> 1;
                matrix_reps_index_map[i] = utils::reverse_bits_uint64(static_cast<uint64_t>(index1), logn);
                matrix_reps_index_map[i + row_size] = utils::reverse_bits_uint64(static_cast<uint64_t>(index2), logn);
                pos = (pos * gen) & (m - 1);
            }
        }

        this->context_ = context;
        this->matrix_reps_index_map = std::move(matrix_reps_index_map);
        this->slots_ = slots;

    }

    __global__ static
    void kernel_reverse_bits(size_t logn, Slice<uint64_t> input) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < input.size()) {
            size_t j = utils::reverse_bits_uint64(static_cast<uint64_t>(i), logn);
            if (i < j) {
                uint64_t temp = input[i];
                input[i] = input[j];
                input[j] = temp;
            }
        }
    }

    void BatchEncoder::reverse_bits(utils::Slice<uint64_t> input) {
        size_t n = input.size();
        int logn_int = utils::get_power_of_two(static_cast<uint64_t>(n));
        if (logn_int < 0) {
            throw std::invalid_argument("[BatchEncoder::reverse_bits] input size must be a power of two.");
        }
        size_t logn = static_cast<size_t>(logn_int);
        bool device = input.on_device();
        if (!device) {
            for (size_t i = 0; i < n; i++) {
                size_t j = utils::reverse_bits_uint64(static_cast<uint64_t>(i), logn);
                if (i < j) {
                    std::swap(input[i], input[j]);
                }
            }
        } else {
            size_t block_count = utils::ceil_div(n, utils::KERNEL_THREAD_COUNT);
            kernel_reverse_bits<<<block_count, utils::KERNEL_THREAD_COUNT>>>(logn, input);
            hipStreamSynchronize(0);
        }
    }

    __global__ static void kernel_encode_set_values(
        ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < values.size()) {
            destination[index_map[i]] = values[i];
        } else if (i < destination.size()) {
            destination[index_map[i]] = 0;
        }
    }

    static void encode_set_values(ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination) {
        size_t device = index_map.on_device();
        if (!utils::same(device, values.on_device(), destination.on_device())) {
            throw std::invalid_argument("[BatchEncoder::encode_set_values] All inputs must reside on same device.");
        }
        if (!device) {
            for (size_t i = 0; i < values.size(); i++) {
                destination[index_map[i]] = values[i];
            }
            for (size_t i = values.size(); i < destination.size(); i++) {
                destination[index_map[i]] = 0;
            }
        } else {
            size_t block_count = utils::ceil_div(destination.size(), utils::KERNEL_THREAD_COUNT);
            kernel_encode_set_values<<<block_count, utils::KERNEL_THREAD_COUNT>>>(values, index_map, destination);
            hipStreamSynchronize(0);
        }
    }
    
    void BatchEncoder::encode_slice(utils::ConstSlice<uint64_t> values, Plaintext& destination, MemoryPoolHandle pool) const {

        if (!pool_compatible(pool)) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Memory pool is not compatible with device.");
        }

        // if values and this is not on the same device, convert first
        if (this->on_device() && (!values.on_device() || values.device_index() != this->device_index())) {
            Array<uint64_t> values_device = Array<uint64_t>::create_and_copy_from_slice(values, true, pool);
            encode_slice(values_device.const_reference(), destination, pool);
            return;
        } else if (!this->on_device() && values.on_device()) {
            Array<uint64_t> values_host = Array<uint64_t>::create_and_copy_from_slice(values, false, nullptr);
            encode_slice(values_host.const_reference(), destination, pool);
            return;
        }

        // check compatible
        if (!utils::device_compatible(values, *this)) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Values and destination are not compatible.");
        }

        if (this->matrix_reps_index_map.size() == 0) {
            throw std::logic_error("[BatchEncoder::encode_slice] The parameters does not support vector batching.");
        }
        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t value_size = values.size();
        if (value_size > this->slot_count()) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Values has size larger than the number of slots.");
        }
        // Set destination to full size
        size_t slots = this->slot_count();
        bool device = this->on_device();
        if (device) {destination.to_device_inplace(pool);}
        else {destination.to_host_inplace();}
        destination.parms_id() = parms_id_zero;
        destination.resize(slots);
        destination.poly_modulus_degree() = slots;
        destination.coeff_modulus_size() = 1;
        destination.is_ntt_form() = false;
        // First write the values to destination coefficients.
        // Read in top row, then bottom row.
        encode_set_values(
            values,
            this->matrix_reps_index_map.const_reference(), 
            destination.poly()
        );
        // Transform destination using inverse of negacyclic NTT
        // Note: We already performed bit-reversal when reading in the matrix
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Context and destination must reside on same device.");
        }
        utils::inverse_ntt_negacyclic_harvey(
            destination.poly(),
            slots,
            context_data->plain_ntt_tables()
        );
    }

    void BatchEncoder::encode_polynomial_slice(utils::ConstSlice<uint64_t> values, Plaintext& destination, MemoryPoolHandle pool) const {
        
        if (!pool_compatible(pool)) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Memory pool is not compatible with device.");
        }

        // if values and this is not on the same device, convert first
        if (this->on_device() && (!values.on_device() || values.device_index() != this->device_index())) {
            Array<uint64_t> values_device = Array<uint64_t>::create_and_copy_from_slice(values, true, pool);
            encode_polynomial_slice(values_device.const_reference(), destination, pool);
            return;
        } else if (!this->on_device() && values.on_device()) {
            Array<uint64_t> values_host = Array<uint64_t>::create_and_copy_from_slice(values, false, nullptr);
            encode_polynomial_slice(values_host.const_reference(), destination, pool);
            return;
        }

        // check compatible
        if (!utils::device_compatible(values, *this)) {
            throw std::invalid_argument("[BatchEncoder::encode_polynomial_slice] Values and destination are not compatible.");
        }

        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t value_size = values.size();
        if (value_size > this->slot_count()) {
            throw std::invalid_argument("[BatchEncoder::encode_polynomial_slice] Values has size larger than the number of slots.");
        }
        // Set destination to full size
        bool device = this->on_device();
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::encode_polynomial_slice] Context and destination must reside on same device.");
        }
        size_t slots = this->slot_count();
        if (device) {destination.to_device_inplace(pool);}
        else {destination.to_host_inplace();}
        destination.parms_id() = parms_id_zero;
        destination.resize(value_size);
        destination.poly_modulus_degree() = slots;
        destination.coeff_modulus_size() = 1;
        destination.is_ntt_form() = false;
        utils::ConstPointer<Modulus> plain_modulus = context_data->parms().plain_modulus();
        utils::modulo(
            values,
            plain_modulus, destination.poly().slice(0, value_size)
        );
    }

    __global__ static void kernel_decode_set_values(
        ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < values.size()) {
            destination[i] = values[index_map[i]];
        }
    }

    static void decode_set_values(ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination) {
        size_t device = index_map.on_device();
        if (!utils::same(device, values.on_device(), destination.on_device())) {
            throw std::invalid_argument("[BatchEncoder::decode_set_values] All inputs must reside on same device.");
        }
        if (!device) {
            for (size_t i = 0; i < values.size(); i++) {
                destination[i] = values[index_map[i]];
            }
        } else {
            size_t block_count = utils::ceil_div(destination.size(), utils::KERNEL_THREAD_COUNT);
            kernel_decode_set_values<<<block_count, utils::KERNEL_THREAD_COUNT>>>(values, index_map, destination);
            hipStreamSynchronize(0);
        }
    }
    
    void BatchEncoder::decode_slice(const Plaintext& plain, Slice<uint64_t> destination, MemoryPoolHandle pool) const {

        if (!pool_compatible(pool)) {
            throw std::invalid_argument("[BatchEncoder::encode_slice] Memory pool is not compatible with device.");
        }

        // if values and this is not on the same device, convert first
        if (this->on_device() && (!destination.on_device() || destination.device_index() != this->device_index())) {
            Array<uint64_t> destination_device = Array<uint64_t>(destination.size(), true, pool);
            decode_slice(plain, destination_device.reference(), pool);
            destination.copy_from_slice(destination_device.const_reference());
            return;
        } else if (!this->on_device() && destination.on_device()) {
            Array<uint64_t> destination_host = Array<uint64_t>(destination.size(), false, nullptr);
            decode_slice(plain, destination_host.reference(), pool);
            destination.copy_from_slice(destination_host.const_reference());
            return;
        }
        
        // check compatible
        if (!utils::device_compatible(destination, *this)) {
            throw std::invalid_argument("[BatchEncoder::decode_slice] Values and destination are not compatible.");
        }

        if (this->matrix_reps_index_map.size() == 0) {
            throw std::logic_error("[BatchEncoder::encode] The parameters does not support vector batching.");
        }
        if (plain.is_ntt_form()) {
            throw std::invalid_argument("[BatchEncoder::decode] Plaintext is in NTT form.");
        }
        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t slots = this->slot_count();
        if (destination.size() != slots) {
            throw std::invalid_argument("[BatchEncoder::decode] Destination has incorrect size.");
        }
        size_t plain_coeff_count = std::min(plain.coeff_count(), slots);
        Array<uint64_t> temp_dest(slots, plain.on_device(), pool);
        temp_dest.slice(0, plain_coeff_count).copy_from_slice(plain.poly());
        // Transform destination using negacyclic NTT
        bool device = this->on_device();
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::decode] Context and destination must reside on same device.");
        }
        utils::ntt_negacyclic_harvey(
            temp_dest.reference(),
            slots,
            context_data->plain_ntt_tables()
        );
        decode_set_values(
            temp_dest.const_reference(),
            this->matrix_reps_index_map.const_reference(), 
            destination
        );
    }
    

    void BatchEncoder::decode_polynomial_slice(const Plaintext& plaintext, utils::Slice<uint64_t> destination) const {
        if (plaintext.is_ntt_form() || plaintext.parms_id() != parms_id_zero) {
            throw std::invalid_argument("[BatchEncoder::decode_polynomial_slice] Plaintext is not in valid form.");
        }
        if (destination.size() != plaintext.data().size()) {
            throw std::invalid_argument("[BatchEncoder::decode_polynomial_slice] Destination has incorrect size.");
        }
        destination.copy_from_slice(plaintext.data().const_reference());
    }

    Plaintext BatchEncoder::scale_up_new(const Plaintext& plain, std::optional<ParmsID> parms_id, MemoryPoolHandle pool) const {
        if (this->context_->first_context_data().value()->parms().scheme() != SchemeType::BFV) {
            throw std::logic_error("[BatchEncoder::scale_up_new] Only BFV scheme is supported.");
        }
        if (plain.parms_id() != parms_id_zero) {
            throw std::invalid_argument("[BatchEncoder::scale_up_new] Plaintext is already at the desired level.");
        }
        ParmsID pid = parms_id.value_or(this->context_->first_parms_id());
        ContextDataPointer context_data = this->context_->get_context_data(pid).value();
        Plaintext destination;
        if (plain.on_device()) {
            destination.to_device_inplace(pool);
        } else {
            destination.to_host_inplace();
        }
        destination.resize_rns(*this->context_, pid);
        destination.is_ntt_form() = false;
        scaling_variant::scale_up(plain, context_data, destination.reference(), false, false);
        return destination;
    }

    Plaintext BatchEncoder::scale_down_new(const Plaintext& plain, MemoryPoolHandle pool) const {
        if (this->context_->first_context_data().value()->parms().scheme() != SchemeType::BFV) {
            throw std::logic_error("[BatchEncoder::scale_down_new] Only BFV scheme is supported.");
        }
        if (plain.parms_id() == parms_id_zero) {
            throw std::invalid_argument("[BatchEncoder::scale_down_new] Plaintext not in RNS form.");
        }
        if (plain.is_ntt_form()) {
            throw std::invalid_argument("[BatchEncoder::scale_down_new] Plaintext is in NTT form.");
        }
        Plaintext destination;
        if (plain.on_device()) {
            destination.to_device_inplace(pool);
        } else {
            destination.to_host_inplace();
        }
        destination.coeff_modulus_size() = plain.coeff_modulus_size();
        destination.poly_modulus_degree() = plain.poly_modulus_degree();
        destination.parms_id() = parms_id_zero;
        destination.resize(plain.poly_modulus_degree());
        destination.is_ntt_form() = false;
        std::optional<ContextDataPointer> context_data_opt = this->context_->get_context_data(plain.parms_id());
        if (!context_data_opt.has_value()) {
            throw std::invalid_argument("[BatchEncoder::scale_down_new] Could not find context data.");
        }
        ContextDataPointer context_data = context_data_opt.value();
        context_data->rns_tool().decrypt_scale_and_round(plain.const_reference(), destination.reference(), pool);
        return destination;
    }

    Plaintext BatchEncoder::centralize_new(const Plaintext& plain, std::optional<ParmsID> parms_id, MemoryPoolHandle pool) const {
        if (this->context_->first_context_data().value()->parms().scheme() != SchemeType::BFV) {
            throw std::logic_error("[BatchEncoder::scale_up_new] Only BFV scheme is supported.");
        }
        if (plain.parms_id() != parms_id_zero) {
            throw std::invalid_argument("[BatchEncoder::scale_up_new] Plaintext is already at the desired level.");
        }
        ParmsID pid = parms_id.value_or(this->context_->first_parms_id());
        ContextDataPointer context_data = this->context_->get_context_data(pid).value();
        Plaintext destination;
        if (plain.on_device()) {
            destination.to_device_inplace(pool);
        } else {
            destination.to_host_inplace();
        }
        destination.resize_rns(*this->context_, pid);
        destination.is_ntt_form() = false;
        scaling_variant::centralize(plain, context_data, destination.reference(), pool);
        return destination;
    }

}