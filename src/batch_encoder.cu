#include "hip/hip_runtime.h"
#include "batch_encoder.cuh"

namespace troy {

    using utils::Array;
    using utils::Slice; 
    using utils::ConstSlice;

    BatchEncoder::BatchEncoder(HeContextPointer context) {
        if (context->on_device()) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Cannot create from device context.");
        }
        if (!context->parameters_set()) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Encryption parameters are not set correctly.");
        }
        
        ContextDataPointer context_data = context->first_context_data().value();
        const EncryptionParameters& parms = context_data->parms();

        if (parms.scheme() != SchemeType::BFV && parms.scheme() != SchemeType::BGV) {
            throw std::invalid_argument("[BatchEncoder::BatchEncoder] Unsupported scheme.");
        }

        size_t slots = parms.poly_modulus_degree();
        Array<uint64_t> roots_of_unity;
        Array<size_t> matrix_reps_index_map;

        if (context_data->qualifiers().using_batching) {
            roots_of_unity = Array<uint64_t>(slots, false);
            const Modulus& modulus = *parms.plain_modulus();
            uint64_t root = context_data->plain_ntt_tables()->root();
            uint64_t generator_sq = utils::multiply_uint64_mod(root, root, modulus);
            roots_of_unity[0] = root;
            for (size_t i = 1; i < slots; i++) {
                roots_of_unity[i] = utils::multiply_uint64_mod(roots_of_unity[i - 1], generator_sq, modulus);
            }
            int logn_int = utils::get_power_of_two(static_cast<uint64_t>(slots));
            if (logn_int < 0) {
                throw std::invalid_argument("[BatchEncoder::BatchEncoder] Slots must be a power of two.");
            }
            size_t logn = static_cast<size_t>(logn_int);
            matrix_reps_index_map = Array<size_t>(slots, false);
            size_t row_size = slots >> 1;
            size_t m = slots << 1;
            size_t gen = utils::GALOIS_GENERATOR; size_t pos = 1;
            for (size_t i = 0; i < row_size; i++) {
                size_t index1 = (pos - 1) >> 1;
                size_t index2 = (m - pos - 1) >> 1;
                matrix_reps_index_map[i] = utils::reverse_bits_uint64(static_cast<uint64_t>(index1), logn);
                matrix_reps_index_map[i + row_size] = utils::reverse_bits_uint64(static_cast<uint64_t>(index2), logn);
                pos = (pos * gen) & (m - 1);
            }
        }

        this->context_ = context;
        this->matrix_reps_index_map = std::move(matrix_reps_index_map);
        this->slots_ = slots;

    }

    __global__ static
    void kernel_reverse_bits(size_t logn, Slice<uint64_t> input) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < input.size()) {
            size_t j = utils::reverse_bits_uint64(static_cast<uint64_t>(i), logn);
            if (i < j) {
                uint64_t temp = input[i];
                input[i] = input[j];
                input[j] = temp;
            }
        }
    }

    void BatchEncoder::reverse_bits(utils::Slice<uint64_t> input) {
        size_t n = input.size();
        int logn_int = utils::get_power_of_two(static_cast<uint64_t>(n));
        if (logn_int < 0) {
            throw std::invalid_argument("[BatchEncoder::reverse_bits] input size must be a power of two.");
        }
        size_t logn = static_cast<size_t>(logn_int);
        bool device = input.on_device();
        if (!device) {
            for (size_t i = 0; i < n; i++) {
                size_t j = utils::reverse_bits_uint64(static_cast<uint64_t>(i), logn);
                if (i < j) {
                    std::swap(input[i], input[j]);
                }
            }
        } else {
            size_t block_count = utils::ceil_div(n, utils::KERNEL_THREAD_COUNT);
            kernel_reverse_bits<<<block_count, utils::KERNEL_THREAD_COUNT>>>(logn, input);
        }
    }

    __global__ static void kernel_encode_set_values(
        ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < values.size()) {
            destination[index_map[i]] = values[i];
        } else if (i < destination.size()) {
            destination[index_map[i]] = 0;
        }
    }

    static void encode_set_values(ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination) {
        size_t device = index_map.on_device();
        if (!utils::same(device, values.on_device(), destination.on_device())) {
            throw std::invalid_argument("[BatchEncoder::encode_set_values] All inputs must reside on same device.");
        }
        if (!device) {
            for (size_t i = 0; i < values.size(); i++) {
                destination[index_map[i]] = values[i];
            }
            for (size_t i = values.size(); i < destination.size(); i++) {
                destination[index_map[i]] = 0;
            }
        } else {
            size_t block_count = utils::ceil_div(destination.size(), utils::KERNEL_THREAD_COUNT);
            kernel_encode_set_values<<<block_count, utils::KERNEL_THREAD_COUNT>>>(values, index_map, destination);
        }
    }
    
    void BatchEncoder::encode(const std::vector<uint64_t>& values, Plaintext& destination) const {
        if (this->matrix_reps_index_map.size() == 0) {
            throw std::logic_error("[BatchEncoder::encode] The parameters does not support vector batching.");
        }
        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t value_size = values.size();
        if (value_size > this->slot_count()) {
            throw std::invalid_argument("[BatchEncoder::encode] Values has size larger than the number of slots.");
        }
        // Set destination to full size
        size_t slots = this->slot_count();
        bool device = this->on_device();
        if (device) {destination.to_device_inplace();}
        else {destination.to_host_inplace();}
        destination.parms_id() = parms_id_zero;
        destination.resize(slots);
        destination.poly_modulus_degree() = slots;
        destination.coeff_modulus_size() = 1;
        destination.is_ntt_form() = false;
        // First write the values to destination coefficients.
        // Read in top row, then bottom row.
        if (!device) {
            encode_set_values(
                ConstSlice(values.data(), values.size(), false),
                this->matrix_reps_index_map.const_reference(), 
                destination.poly()
            );
        } else {
            Array<uint64_t> values_device(value_size, false);
            for (size_t i = 0; i < value_size; i++) {
                values_device[i] = values[i];
            }
            values_device.to_device_inplace();
            encode_set_values(
                values_device.const_reference(),
                this->matrix_reps_index_map.const_reference(), 
                destination.poly()
            );
        }
        // Transform destination using inverse of negacyclic NTT
        // Note: We already performed bit-reversal when reading in the matrix
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::encode] Context and destination must reside on same device.");
        }
        utils::inverse_ntt_negacyclic_harvey(
            destination.poly(),
            slots,
            context_data->plain_ntt_tables()
        );
    }
    

    void BatchEncoder::encode_polynomial(const std::vector<uint64_t>& values, Plaintext& destination) const {
        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t value_size = values.size();
        if (value_size > this->slot_count()) {
            throw std::invalid_argument("[BatchEncoder::encode] Values has size larger than the number of slots.");
        }
        // Set destination to full size
        bool device = this->on_device();
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::encode] Context and destination must reside on same device.");
        }
        size_t slots = this->slot_count();
        if (device) {destination.to_device_inplace();}
        else {destination.to_host_inplace();}
        destination.parms_id() = parms_id_zero;
        destination.resize(value_size);
        destination.poly_modulus_degree() = slots;
        destination.coeff_modulus_size() = 1;
        destination.is_ntt_form() = false;
        utils::ConstPointer<Modulus> plain_modulus = context_data->parms().plain_modulus();
        if (!device) {
            utils::modulo(
                ConstSlice(values.data(), values.size(), false),
                plain_modulus, destination.poly().slice(0, value_size)
            );
        } else {
            Array<uint64_t> values_device(value_size, true);
            values_device.copy_from_slice(ConstSlice(values.data(), values.size(), false));
            values_device.to_device_inplace();
            utils::modulo(
                values_device.const_reference(),
                plain_modulus, destination.poly().slice(0, value_size)
            );
        }
    }

    __global__ static void kernel_decode_set_values(
        ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination
    ) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < values.size()) {
            destination[i] = values[index_map[i]];
        }
    }

    static void decode_set_values(ConstSlice<uint64_t> values, ConstSlice<size_t> index_map, Slice<uint64_t> destination) {
        size_t device = index_map.on_device();
        if (!utils::same(device, values.on_device(), destination.on_device())) {
            throw std::invalid_argument("[BatchEncoder::decode_set_values] All inputs must reside on same device.");
        }
        if (!device) {
            for (size_t i = 0; i < values.size(); i++) {
                destination[i] = values[index_map[i]];
            }
        } else {
            size_t block_count = utils::ceil_div(destination.size(), utils::KERNEL_THREAD_COUNT);
            kernel_decode_set_values<<<block_count, utils::KERNEL_THREAD_COUNT>>>(values, index_map, destination);
        }
    }
    
    void BatchEncoder::decode(const Plaintext& plain, std::vector<uint64_t>& destination) const {
        if (this->matrix_reps_index_map.size() == 0) {
            throw std::logic_error("[BatchEncoder::encode] The parameters does not support vector batching.");
        }
        if (plain.is_ntt_form()) {
            throw std::invalid_argument("[BatchEncoder::decode] Plaintext is in NTT form.");
        }
        ContextDataPointer context_data = this->context()->first_context_data().value();
        size_t slots = this->slot_count();
        destination.resize(slots);
        size_t plain_coeff_count = std::min(plain.coeff_count(), slots);
        Array<uint64_t> temp_dest(slots, plain.on_device());
        temp_dest.slice(0, plain_coeff_count).copy_from_slice(plain.poly());
        // Transform destination using negacyclic NTT
        bool device = this->on_device();
        if (device != context_data->on_device()) {
            throw std::invalid_argument("[BatchEncoder::decode] Context and destination must reside on same device.");
        }
        utils::ntt_negacyclic_harvey(
            temp_dest.reference(),
            slots,
            context_data->plain_ntt_tables()
        );
        // Read in top row, then bottom row.
        if (!device) {
            decode_set_values(
                temp_dest.const_reference(),
                this->matrix_reps_index_map.const_reference(), 
                Slice<uint64_t>(destination.data(), destination.size(), false)
            );
        } else {
            Array<uint64_t> temp_dest_host(slots, true);
            decode_set_values(
                temp_dest.const_reference(),
                this->matrix_reps_index_map.const_reference(), 
                temp_dest_host.reference()
            );
            Slice<uint64_t>(destination.data(), destination.size(), false).copy_from_slice(temp_dest_host.const_reference());
        }
    }

    void BatchEncoder::decode_polynomial(const Plaintext& plaintext, std::vector<uint64_t>& destination) const {
        destination.resize(plaintext.data().size());
        Slice<uint64_t>(destination.data(), destination.size(), false)
            .copy_from_slice(plaintext.data().const_reference());
    }
}