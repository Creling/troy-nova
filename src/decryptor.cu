#include "hip/hip_runtime.h"
#include "decryptor.h"
#include "encryption_parameters.h"
#include "utils/constants.h"
#include "utils/scaling_variant.h"

namespace troy {

    using utils::ConstSlice;
    using utils::NTTTables;
    using utils::Array;

    Decryptor::Decryptor(HeContextPointer context, const SecretKey& secret_key, MemoryPoolHandle pool) :
        context_(context) 
    {
        ContextDataPointer key_context_data = context->key_context_data().value();
        const EncryptionParameters& parms = key_context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_count = parms.poly_modulus_degree();
        size_t coeff_modulus_size = coeff_modulus.size();
        if (secret_key.data().size() != coeff_count * coeff_modulus_size)
            throw std::invalid_argument("[Decryptor::Decryptor] secret_key is not valid for encryption parameters");
        this->secret_key_array_ = secret_key.data().clone(pool);
    }

    void Decryptor::dot_product_ct_sk_array(const Ciphertext& encrypted, utils::Slice<uint64_t> destination, MemoryPoolHandle pool) const {
        if (!utils::same(this->on_device(), encrypted.on_device(), destination.on_device())) {
            throw std::invalid_argument("[Decryptor::dot_product_ct_sk_array] Arguments are not on the same device.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        size_t encrypted_size = encrypted.polynomial_count();
        size_t key_coeff_modulus_size = this->context()->key_context_data().value()->parms().coeff_modulus().size();
        ConstSlice<NTTTables> ntt_tables = context_data->small_ntt_tables();
        bool is_ntt_form = encrypted.is_ntt_form();

        // Make sure we have enough secret key powers computed
        if (this->secret_key_array_.size() < (encrypted_size - 1) * coeff_count * key_coeff_modulus_size) {
            std::unique_lock<std::shared_mutex> lock(this->secret_key_array_mutex);
            KeyGenerator::compute_secret_key_powers(this->context(), encrypted_size - 1, this->secret_key_array_);
            lock.unlock();
        }

        // acquire read lock
        std::shared_lock<std::shared_mutex> lock(this->secret_key_array_mutex);
        if (encrypted_size == 2) {
            ConstSlice<uint64_t> c0 = encrypted.poly(0);
            ConstSlice<uint64_t> c1 = encrypted.poly(1);
            ConstSlice<uint64_t> s = this->secret_key_array_.const_slice(0, c0.size());
            if (is_ntt_form) {
                // put < c_1 * s > mod q in destination
                utils::dyadic_product_p(c1, s, coeff_count, coeff_modulus, destination);
                // add c_0 to the result; note that destination should be in the same (NTT) form as encrypted
                utils::add_inplace_p(destination, c0, coeff_count, coeff_modulus);
            } else {
                destination.copy_from_slice(c1);
                utils::ntt_negacyclic_harvey_p(destination, coeff_count, ntt_tables);
                utils::dyadic_product_inplace_p(destination, s, coeff_count, coeff_modulus);
                utils::inverse_ntt_negacyclic_harvey_p(destination, coeff_count, ntt_tables);
                utils::add_inplace_p(destination, c0, coeff_count, coeff_modulus);
            }
        } else {
            size_t poly_coeff_count = coeff_count * coeff_modulus_size;
            size_t key_poly_coeff_count = coeff_count * key_coeff_modulus_size;
            Array<uint64_t> encrypted_copy = Array<uint64_t>::create_and_copy_from_slice(encrypted.data().const_slice(poly_coeff_count, encrypted_size * poly_coeff_count), pool);
            if (!is_ntt_form) {
                utils::ntt_negacyclic_harvey_ps(encrypted_copy.reference(), encrypted_size - 1, coeff_count, ntt_tables);
            }
            for (size_t i = 0; i < encrypted_size - 1; i++) {
                utils::dyadic_product_inplace_p(
                    encrypted_copy.slice(i*poly_coeff_count, (i+1)*poly_coeff_count), 
                    this->secret_key_array_.const_slice(i*key_poly_coeff_count, i*key_poly_coeff_count+poly_coeff_count), 
                    coeff_count, coeff_modulus);
            }
            destination.set_zero();
            for (size_t i = 0; i < encrypted_size - 1; i++) {
                utils::add_inplace_p(
                    destination, 
                    encrypted_copy.const_slice(i*poly_coeff_count, (i+1)*poly_coeff_count),
                    coeff_count, coeff_modulus);
            }
            if (!is_ntt_form) {
                utils::inverse_ntt_negacyclic_harvey_p(destination, coeff_count, ntt_tables);
            }
            utils::add_inplace_p(destination, encrypted.poly(0), coeff_count, coeff_modulus);
        }

        // release read lock
        lock.unlock();
    }

    void Decryptor::decrypt(const Ciphertext& encrypted, Plaintext& destination, MemoryPoolHandle pool) const {
        // sanity check
        if (encrypted.contains_seed()) {
            throw std::invalid_argument("[Decryptor::decrypt] Seed should be expanded first.");
        }
        if (encrypted.polynomial_count() < utils::HE_CIPHERTEXT_SIZE_MIN) {
            throw std::invalid_argument("[Decryptor::decrypt] Ciphertext is empty.");
        }
        if (encrypted.on_device()) destination.to_device_inplace(pool);
        else destination.to_host_inplace();
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        switch (scheme) {
            case SchemeType::BFV: this->bfv_decrypt(encrypted, destination, pool); break;
            case SchemeType::CKKS: this->ckks_decrypt(encrypted, destination, pool); break;
            case SchemeType::BGV: this->bgv_decrypt(encrypted, destination, pool); break;
            default: throw std::invalid_argument("[Decryptor::decrypt] Unsupported scheme.");
        }
    }

    void Decryptor::bfv_decrypt_without_scaling_down(const Ciphertext& encrypted, Plaintext& destination, MemoryPoolHandle pool) const {
        if (encrypted.is_ntt_form()) {
            throw std::invalid_argument("[Decryptor::bfv_decrypt] Ciphertext is in NTT form.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        
        // Firstly find c_0 + c_1 *s + ... + c_{count-1} * s^{count-1} mod q
        // This is equal to Delta m + v where ||v|| < Delta/2.

        // Make a temp destination for all the arithmetic mod qi before calling FastBConverse
        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();
        
        destination.resize_rns(*this->context_, encrypted.parms_id());

        // put < (c_1 , c_2, ... , c_{count-1}) , (s,s^2,...,s^{count-1}) > mod q in destination
        // Now do the dot product of encrypted_copy and the secret key array using NTT.
        // The secret key powers are already NTT transformed.
        this->dot_product_ct_sk_array(encrypted, destination.reference(), pool);
    }

    void Decryptor::bfv_decrypt(const Ciphertext& encrypted, Plaintext& destination, MemoryPoolHandle pool) const {
        if (encrypted.is_ntt_form()) {
            throw std::invalid_argument("[Decryptor::bfv_decrypt] Ciphertext is in NTT form.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();

        Plaintext temp;
        this->bfv_decrypt_without_scaling_down(encrypted, temp, pool);
        
        // Add Delta / 2 and now we have something which is Delta * (m + epsilon) where epsilon < 1
        // Therefore, we can (integer) divide by Delta and the answer will round down to m.

        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();

        // Allocate a full size destination to write to
        destination.parms_id() = parms_id_zero;
        destination.resize(coeff_count);

        // Divide scaling variant using BEHZ FullRNS techniques
        context_data->rns_tool().decrypt_scale_and_round(
            temp.const_reference(), coeff_count, destination.poly(), pool
        );
        destination.is_ntt_form() = false;
        destination.coeff_modulus_size() = coeff_modulus_size;
        destination.poly_modulus_degree() = coeff_count;
    }

    void Decryptor::ckks_decrypt(const Ciphertext& encrypted, Plaintext& destination, MemoryPoolHandle pool) const {

        if (!encrypted.is_ntt_form()) {
            throw std::invalid_argument("[Decryptor::ckks_decrypt] Ciphertext is not in NTT form.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;
        
        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();

        // Decryption consists in finding
        // c_0 + c_1 *s + ... + c_{count-1} * s^{count-1} mod q_1 * q_2 * q_3
        // as long as ||m + v|| < q_1 * q_2 * q_3.
        // This is equal to m + v where ||v|| is small enough.
        
        // Since we overwrite destination, we zeroize destination parameters
        // This is necessary, otherwise resize will throw an exception.
        destination.parms_id() = parms_id_zero;
        // Resize destination to appropriate size
        destination.resize(rns_poly_uint64_count);
        
        // Do the dot product of encrypted and the secret key array using NTT.
        this->dot_product_ct_sk_array(encrypted, destination.poly(), pool);

        // Set destination parameters as in encrypted
        destination.parms_id() = encrypted.parms_id();
        destination.scale() = encrypted.scale();
        destination.is_ntt_form() = true;
        destination.coeff_modulus_size() = coeff_modulus_size;
        destination.poly_modulus_degree() = coeff_count;
    }

    void Decryptor::bgv_decrypt(const Ciphertext& encrypted, Plaintext& destination, MemoryPoolHandle pool) const {
        if (!encrypted.is_ntt_form()) {
            throw std::invalid_argument("[Decryptor::bgv_decrypt] Ciphertext is not in NTT form.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        
        // Make a temp destination for all the arithmetic mod qi before calling FastBConverse
        bool device = encrypted.on_device();
        if (device) destination.to_device_inplace(pool);
        else destination.to_host_inplace();
        Plaintext tmp_dest_modq; if (encrypted.on_device()) tmp_dest_modq.to_device_inplace(pool);
        tmp_dest_modq.resize_rns(*this->context_, encrypted.parms_id());

        this->dot_product_ct_sk_array(encrypted, tmp_dest_modq.reference(), pool);

        // Allocate a full size destination to write to
        destination.parms_id() = parms_id_zero;
        destination.resize(coeff_count);

        utils::inverse_ntt_negacyclic_harvey_p(tmp_dest_modq.reference(), coeff_count, context_data->small_ntt_tables());

        scaling_variant::decentralize(tmp_dest_modq, context_data, destination.poly(), encrypted.correction_factor(), pool);

        destination.is_ntt_form() = false;
        destination.coeff_modulus_size() = coeff_modulus_size;
        destination.poly_modulus_degree() = coeff_count;
    }

    static void poly_infty_norm(ConstSlice<uint64_t> poly, size_t coeff_uint64_count, ConstSlice<uint64_t> modulus, utils::Slice<uint64_t> result) {
        if (modulus.size() != coeff_uint64_count) {
            throw std::invalid_argument("[poly_infty_norm] Modulus is not valid.");
        }
        bool device = poly.on_device();
        if (device) {
            throw std::invalid_argument("[poly_infty_norm] Poly is on device.");
        }
        // Construct negative threshold: (modulus + 1) / 2
        Array<uint64_t> modulus_neg_threshold(modulus.size(), false, nullptr);
        utils::half_round_up_uint(modulus, modulus_neg_threshold.reference());
        // Mod out the poly coefficients and choose a symmetric representative from [-modulus,modulus)
        result.set_zero();
        Array<uint64_t> coeff_abs_value(coeff_uint64_count, false, nullptr);
        coeff_abs_value.set_zero();
        size_t coeff_count = poly.size() / coeff_uint64_count;
        for (size_t i = 0; i < coeff_count; i++) {
            ConstSlice<uint64_t> poly_i = poly.const_slice(i * coeff_uint64_count, (i + 1) * coeff_uint64_count);
            if (utils::is_greater_or_equal_uint(poly_i, modulus_neg_threshold.const_reference())) {
                utils::sub_uint(modulus, poly_i, coeff_abs_value.reference());
            } else {
                coeff_abs_value.copy_from_slice(poly_i);
            }
            if (utils::is_greater_than_uint(coeff_abs_value.const_reference(), result.as_const())) {
                result.copy_from_slice(coeff_abs_value.const_reference());
            }
        }
    }

    size_t Decryptor::invariant_noise_budget(const Ciphertext& encrypted, MemoryPoolHandle pool) const {
        if (encrypted.polynomial_count() < utils::HE_CIPHERTEXT_SIZE_MIN) {
            throw std::invalid_argument("[Decryptor::invariant_noise_budget] Ciphertext is invalid.");
        }
        SchemeType scheme = this->context()->first_context_data().value()->parms().scheme();
        if (scheme != SchemeType::BFV && scheme != SchemeType::BGV) {
            throw std::invalid_argument("[Decryptor::invariant_noise_budget] Unsupported scheme.");
        }
        ContextDataPointer context_data = this->context()->get_context_data(encrypted.parms_id()).value();
        const EncryptionParameters& parms = context_data->parms();
        ConstSlice<Modulus> coeff_modulus = parms.coeff_modulus();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t coeff_count = parms.poly_modulus_degree();
        const Modulus& plain_modulus = parms.plain_modulus_host();

        // Now need to compute c(s) - Delta*m (mod q)
        // Firstly find c_0 + c_1 *s + ... + c_{count-1} * s^{count-1} mod q
        // This is equal to Delta m + v where ||v|| < Delta/2.
        // put < (c_1 , c_2, ... , c_{count-1}) , (s,s^2,...,s^{count-1}) > mod q
        // in destination_poly.
        // Now do the dot product of encrypted_copy and the secret key array using NTT.
        // The secret key powers are already NTT transformed.
        Array<uint64_t> noise_poly(coeff_count * coeff_modulus_size, encrypted.on_device(), pool);
        this->dot_product_ct_sk_array(encrypted, noise_poly.reference(), pool);

        if (encrypted.is_ntt_form()) {
            // In the case of NTT form, we need to transform the noise to normal form
            utils::inverse_ntt_negacyclic_harvey_p(noise_poly.reference(), coeff_count, context_data->small_ntt_tables());
        }

        // Multiply by plain_modulus and reduce mod coeff_modulus to get
        // coeffModulus()*noise.
        if (scheme == SchemeType::BFV) {
            utils::multiply_scalar_inplace_p(
                noise_poly.reference(), plain_modulus.value(), coeff_count, coeff_modulus
            );
        }

        // CRT-compose the noise
        context_data->rns_tool().base_q().compose_array(noise_poly.reference(), pool);

        // Next we compute the infinity norm mod parms.coeffModulus()
        Array<uint64_t> norm(coeff_modulus_size, false, nullptr);
        noise_poly.to_host_inplace();
        Array<uint64_t> total_coeff_modulus = Array<uint64_t>::create_and_copy_from_slice(context_data->total_coeff_modulus(), false, nullptr);
        poly_infty_norm(noise_poly.const_reference(), coeff_modulus_size, total_coeff_modulus.const_reference(), norm.reference());

        // The -1 accounts for scaling the invariant noise by 2;
        // note that we already took plain_modulus into account in compose
        // so no need to subtract log(plain_modulus) from this
        int64_t bit_count_diff = 
            static_cast<int64_t>(context_data->total_coeff_modulus_bit_count()) 
            - static_cast<int64_t>(utils::get_significant_bit_count_uint(norm.const_reference())) 
            - 1;
        if (bit_count_diff < 0) {
            return 0;
        } else {
            return static_cast<size_t>(bit_count_diff);
        }
    }

}