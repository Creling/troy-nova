#include <gtest/gtest.h>
#include "../test.cuh"
#include "../../src/utils/rns_base.cuh"
#include <vector>

using namespace std;
using namespace troy;
using namespace troy::utils;

namespace rns_base {

    bool test_compose_decompose_single(const RNSBase& base, vector<uint64_t> input, vector<uint64_t> output, bool device) {

        Array<uint64_t> x(input.size(), false, nullptr);
        x.copy_from_slice(ConstSlice<uint64_t>(input.data(), input.size(), false, nullptr));

        if (device) x.to_device_inplace(MemoryPool::GlobalPool());

        base.decompose_single(x.reference());

        if (device) x.to_host_inplace();
        for (size_t i = 0; i < input.size(); i++) {
            if (x[i] != output[i]) return false;
        }

        if (device) x.to_device_inplace();
        base.compose_single(x.reference());

        if (device) x.to_host_inplace();
        for (size_t i = 0; i < input.size(); i++) {
            if (x[i] != input[i]) return false;
        }

        return true;
    }

    TEST(RNSBase, ComposeDecomposeSingle) {

        bool device = false;
        
        {
            Array<Modulus> moduli(1, false);
            moduli[0] = Modulus(2);
            RNSBase base(moduli.const_reference());
            ASSERT_TRUE(test_compose_decompose_single(base, {0}, {0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {1}, {1}, device));
        }
        
        {
            Array<Modulus> moduli(1, false);
            moduli[0] = Modulus(5);
            RNSBase base(moduli.const_reference());
            ASSERT_TRUE(test_compose_decompose_single(base, {0}, {0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {1}, {1}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {4}, {4}, device));
        }

        {
            Array<Modulus> moduli(2, false);
            moduli[0] = Modulus(3);
            moduli[1] = Modulus(5);
            RNSBase base(moduli.const_reference());
            ASSERT_TRUE(test_compose_decompose_single(base, {0, 0}, {0, 0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {1, 0}, {1, 1}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {2, 0}, {2, 2}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {3, 0}, {0, 3}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {4, 0}, {1, 4}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {8, 0}, {2, 3}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {12, 0}, {0, 2}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {14, 0}, {2, 4}, device));
        }

        {
            Array<Modulus> moduli(3, false);
            moduli[0] = Modulus(2);
            moduli[1] = Modulus(3);
            moduli[2] = Modulus(5);
            RNSBase base(moduli.const_reference());
            ASSERT_TRUE(test_compose_decompose_single(base, {0, 0, 0}, {0, 0, 0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {1, 0, 0}, {1, 1, 1}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {2, 0, 0}, {0, 2, 2}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {3, 0, 0}, {1, 0, 3}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {4, 0, 0}, {0, 1, 4}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {10, 0, 0}, {0, 1, 0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {29, 0, 0}, {1, 2, 4}, device));
        }

        {
            Array<Modulus> moduli(4, false);
            moduli[0] = Modulus(13);
            moduli[1] = Modulus(37);
            moduli[2] = Modulus(53);
            moduli[3] = Modulus(97);
            RNSBase base(moduli.const_reference());
            ASSERT_TRUE(test_compose_decompose_single(base, {  0, 0, 0, 0}, {0, 0, 0, 0}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {  1, 0, 0, 0}, {1, 1, 1, 1}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {  2, 0, 0, 0}, {2, 2, 2, 2}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, { 12, 0, 0, 0}, {12, 12, 12, 12}, device));
            ASSERT_TRUE(test_compose_decompose_single(base, {321, 0, 0, 0}, {9, 25, 3, 30}, device));
        }

        {
            Array<Modulus> moduli = Array<Modulus>::from_vector(
                utils::get_primes(2048, 60, 4)
            );
            RNSBase base(moduli.const_reference());
            Array<uint64_t> input = Array<uint64_t>::from_vector({
                0xAAAAAAAAAAA, 0xBBBBBBBBBB, 0xCCCCCCCCCC, 0xDDDDDDDDDD
            });
            Array<uint64_t> output = Array<uint64_t>::from_vector({
                utils::modulo_uint(input.const_reference(), moduli[0]),
                utils::modulo_uint(input.const_reference(), moduli[1]),
                utils::modulo_uint(input.const_reference(), moduli[2]),
                utils::modulo_uint(input.const_reference(), moduli[3])
            });
            ASSERT_TRUE(test_compose_decompose_single(base, input.to_vector(), output.to_vector(), device));
        }

    }

    bool test_compose_decompose_array(const RNSBase& base, vector<uint64_t> input, vector<uint64_t> output, bool device) {

        Array<uint64_t> x(input.size(), false);
        x.copy_from_slice(ConstSlice<uint64_t>(input.data(), input.size(), false, nullptr));

        if (device) x.to_device_inplace();

        base.decompose_array(x.reference());

        if (device) x.to_host_inplace();
        for (size_t i = 0; i < input.size(); i++) {
            if (x[i] != output[i]) return false;
        }

        if (device) x.to_device_inplace();
        base.compose_array(x.reference());

        if (device) x.to_host_inplace();
        for (size_t i = 0; i < input.size(); i++) {
            if (x[i] != input[i]) return false;
        }

        return true;
    }

    void test_body_compose_decompose_array(bool device) {
        
        {
            Array<Modulus> moduli(1, false);
            moduli[0] = Modulus(2);
            RNSBase base(moduli.const_reference());
            if (device) base.to_device_inplace();
            ASSERT_TRUE(test_compose_decompose_array(base, {0}, {0}, device));
            ASSERT_TRUE(test_compose_decompose_array(base, {1}, {1}, device));
        }
        
        {
            Array<Modulus> moduli(1, false);
            moduli[0] = Modulus(5);
            RNSBase base(moduli.const_reference());
            if (device) base.to_device_inplace();
            ASSERT_TRUE(test_compose_decompose_array(base, {0, 1, 2}, {0, 1, 2}, device));
        }

        {
            Array<Modulus> moduli(2, false);
            moduli[0] = Modulus(3);
            moduli[1] = Modulus(5);
            RNSBase base(moduli.const_reference());
            if (device) base.to_device_inplace();
            ASSERT_TRUE(test_compose_decompose_array(base, {7, 0}, {1, 2}, device));
            ASSERT_TRUE(test_compose_decompose_array(base, {7, 0, 8, 0}, {1, 2, 2, 3}, device));
        }

        {
            Array<Modulus> moduli(3, false);
            moduli[0] = Modulus(3);
            moduli[1] = Modulus(5);
            moduli[2] = Modulus(7);
            RNSBase base(moduli.const_reference());
            if (device) base.to_device_inplace();
            ASSERT_TRUE(test_compose_decompose_array(base, {7, 0, 0}, {1, 2, 0}, device));
            ASSERT_TRUE(test_compose_decompose_array(base, {7, 0, 0, 8, 0, 0}, {1, 2, 2, 3, 0, 1}, device));
            ASSERT_TRUE(test_compose_decompose_array(base, {7, 0, 0, 8, 0, 0, 9, 0, 0}, {1, 2, 0, 2, 3, 4, 0, 1, 2}, device));
        }

        {
            Array<Modulus> moduli = Array<Modulus>::from_vector(
                utils::get_primes(2048, 60, 2)
            );
            RNSBase base(moduli.const_reference());
            if (device) base.to_device_inplace();
            Array<uint64_t> input = Array<uint64_t>::from_vector({
                0xAAAAAAAAAAA, 0xBBBBBBBBBB, 0xCCCCCCCCCC, 0xDDDDDDDDDD,
                0xEEEEEEEEEE, 0xFFFFFFFFFF
            });
            Array<uint64_t> output = Array<uint64_t>::from_vector({
                utils::modulo_uint(input.const_slice(0, 2), moduli[0]),
                utils::modulo_uint(input.const_slice(2, 4), moduli[0]),
                utils::modulo_uint(input.const_slice(4, 6), moduli[0]),
                utils::modulo_uint(input.const_slice(0, 2), moduli[1]),
                utils::modulo_uint(input.const_slice(2, 4), moduli[1]),
                utils::modulo_uint(input.const_slice(4, 6), moduli[1])
            });
            ASSERT_TRUE(test_compose_decompose_array(base, input.to_vector(), output.to_vector(), device));
        }

    }

    TEST(RNSBase, HostComposeDecomposeArray) {
        test_body_compose_decompose_array(false);
    }

    TEST(RNSBase, DeviceComposeDecomposeArray) {
        test_body_compose_decompose_array(true);
        MemoryPool::Destroy();
    }

    bool test_fast_convert_array(vector<uint64_t> imod, vector<uint64_t> omod, vector<uint64_t> input, vector<uint64_t> output, bool device) {
        // create rns base
        vector<Modulus> imv; imv.reserve(imod.size());
        for (size_t i = 0; i < imod.size(); i++) imv.push_back(Modulus(imod[i]));
        Array<Modulus> im = Array<Modulus>::from_vector(std::move(imv));
        RNSBase ibase(im.const_reference());
        vector<Modulus> omv; omv.reserve(omod.size());
        for (size_t i = 0; i < omod.size(); i++) omv.push_back(Modulus(omod[i]));
        Array<Modulus> om = Array<Modulus>::from_vector(std::move(omv));
        RNSBase obase(om.const_reference());
        // create base converter
        BaseConverter converter(ibase, obase);
        if (device) converter.to_device_inplace();
        // create input array
        Array<uint64_t> x = Array<uint64_t>::from_vector(std::move(input));
        if (device) x.to_device_inplace(MemoryPool::GlobalPool());
        // convert
        Array<uint64_t> y(output.size(), x.on_device(), x.pool());
        converter.fast_convert_array(x.const_reference(), y.reference());
        if (device) y.to_host_inplace();
        // check
        for (size_t i = 0; i < output.size(); i++) {
            if (y[i] != output[i]) return false;
        }
        return true;
    }

    void test_body_fast_convert_array(bool device) {
        test_fast_convert_array({3}, {2}, {0, 1, 2}, {0, 1, 0}, device);
        test_fast_convert_array({2, 3}, {2}, {0, 1, 0, 0, 1, 2}, {0, 1, 0}, device);
        test_fast_convert_array({2, 3}, {2, 3}, {1, 1, 0, 1, 2, 2}, {1, 1, 0, 1, 2, 2}, device);
        test_fast_convert_array({2, 3}, {3, 4, 5}, {0, 1, 1, 0, 1, 2}, {0, 1, 2, 0, 3, 1, 0, 2, 0}, device);
    }

    TEST(RNSBase, HostFastConvertArray) {
        test_body_fast_convert_array(false);
    }

    TEST(RNSBase, DeviceFastConvertArray) {
        test_body_fast_convert_array(true);
        MemoryPool::Destroy();
    }

}